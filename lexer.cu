#include "llvm/ADT/APFloat.h"
#include "llvm/ADT/STLExtras.h"
#include "llvm/IR/BasicBlock.h"
#include "llvm/IR/Constants.h"
#include "llvm/IR/DerivedTypes.h"
#include "llvm/IR/Function.h"
#include "llvm/IR/IRBuilder.h"
#include "llvm/IR/LLVMContext.h"
#include "llvm/IR/Module.h"
#include "llvm/IR/Type.h"
#include "llvm/IR/Verifier.h"
#include <algorithm>
#include <cctype>
#include <cstdio>
#include <cstdlib>
#include <map>
#include <memory>
#include <string>
#include <vector>

using namespace llvm;


// The lexer returns tokens [0-255] if it is an unknown character, otherwise one
// of these for known things.
enum Token {
  tok_eof = -1,

  // commands
  tok_def = -2,
  tok_extern = -3,

  // primary
  tok_identifier = -4,
  tok_number = -5,
};

static std::string IdentifierStr; // Filled in if tok_identifier
static float NumVal;             // Filled in if tok_number


/// gettok - Return the next token from standard input.
static int gettok() {
  static int LastChar = ' ';

  // Skip whitespace
  while (isspace(LastChar))
    LastChar = getchar();

  if (isalpha(LastChar)) { // identifier: [a-zA-Z][a-zA-Z0-9]*
    IdentifierStr = LastChar;
    while (isalnum((LastChar = getchar())))
        IdentifierStr += LastChar;

    // Reserved words
    if (IdentifierStr == "def")
        return tok_def;
    if (IdentifierStr == "extern")
        return tok_extern;

    return tok_identifier;
  }

  if (isdigit(LastChar) || LastChar == '.') {   // Number: [0-9.]+
    std::string NumStr;
    do {
        NumStr += LastChar;
        LastChar = getchar();
    } while (isdigit(LastChar) || LastChar == '.');

    NumVal = strtod(NumStr.c_str(), 0);
    return tok_number;
    // TODO: it will incorrectly read 1.23.45.67 and handle it as if you typed in 1.23.
  }

  if (LastChar == '#') { // Comments
    do
        LastChar = getchar();
    while (LastChar != EOF && LastChar != '\n' && LastChar != '\r');

    if (LastChar != EOF)
        return gettok();
  }

  // Check EOF
  if (LastChar == EOF)
    return tok_eof;

  // Otherwise, just return the character as its ascii value.
  int ThisChar = LastChar;
  LastChar = getchar();
  return ThisChar;
}




/// Base class for all Expression Nodes.
class ExprAST {
  public:
    virtual ~ExprAST() = default;
};

/// Expression for numeric literals like "1.0".
class NumberExprAST : public ExprAST {
  float Val;

  public:
    NumberExprAST(float Val) : Val(Val) {}
};


/// Expression for referencing a variable, like "a".
class VariableExprAST : public ExprAST {
  std::string Name;

  public:
    VariableExprAST(const std::string &Name) : Name(Name) {}
};

/// Expression for a binary operator.
class BinaryExprAST : public ExprAST {
  char Op;
  std::unique_ptr<ExprAST> LHS, RHS;

  public:
    BinaryExprAST(char Op, std::unique_ptr<ExprAST> LHS,
                  std::unique_ptr<ExprAST> RHS)
      : Op(Op), LHS(std::move(LHS)), RHS(std::move(RHS)) {}
};

/// Expression for function calls.
class CallExprAST : public ExprAST {
  std::string Callee;
  std::vector<std::unique_ptr<ExprAST>> Args;

  public:
    CallExprAST(const std::string &Callee,
                std::vector<std::unique_ptr<ExprAST>> Args)
      : Callee(Callee), Args(std::move(Args)) {}
};




/// PrototypeAST - This class represents the "prototype" for a function,
/// which captures its name, and its argument names (thus implicitly the number
/// of arguments the function takes).
class PrototypeAST {
  std::string Name;
  std::vector<std::string> Args;

  public:
    PrototypeAST(const std::string &Name, std::vector<std::string> Args)
      : Name(Name), Args(std::move(Args)) {}

  const std::string &getName() const { return Name; }
};

/// FunctionAST - This class represents a function definition itself.
class FunctionAST {
  std::unique_ptr<PrototypeAST> Proto;
  std::unique_ptr<ExprAST> Body;

  public:
    FunctionAST(std::unique_ptr<PrototypeAST> Proto,
                std::unique_ptr<ExprAST> Body)
      : Proto(std::move(Proto)), Body(std::move(Body)) {}
};


// Token Buffer
static int CurTok; // Token Parser
static int getNextToken() {
  return CurTok = gettok();
}
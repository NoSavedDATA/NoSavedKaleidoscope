#include "llvm/ADT/APFloat.h"
#include "llvm/ADT/STLExtras.h"
#include "llvm/IR/BasicBlock.h"
#include "llvm/IR/Constants.h"
#include "llvm/IR/DerivedTypes.h"
#include "llvm/IR/Function.h"
#include "llvm/IR/IRBuilder.h"
#include "llvm/IR/LLVMContext.h"
#include "llvm/IR/Module.h"
#include "llvm/IR/Type.h"
#include "llvm/IR/Verifier.h"
#include <algorithm>
#include <cctype>
#include <cstdio>
#include <cstdlib>
#include <map>
#include <memory>
#include <string>
#include <vector>

using namespace llvm;


// The lexer returns tokens [0-255] if it is an unknown character, otherwise one
// of these for known things.
enum Token {
  tok_eof = -1,

  // commands
  tok_def = -2,
  tok_extern = -3,

  // primary
  tok_identifier = -4,
  tok_number = -5,
};

static std::string IdentifierStr; // Filled in if tok_identifier
static float NumVal;             // Filled in if tok_number


/// get_token - Return the next token from standard input.
static int get_token() {
  static int LastChar = ' ';

  // Skip whitespace, ' '
  while (LastChar==32)
    LastChar = getchar();


  if (isalpha(LastChar)) { // identifier: [a-zA-Z][a-zA-Z0-9]*
    IdentifierStr = LastChar;
    while (isalnum((LastChar = getchar())))
        IdentifierStr += LastChar;

    // Reserved words
    if (IdentifierStr == "def")
        return tok_def;
    if (IdentifierStr == "extern")
        return tok_extern;

    return tok_identifier;
  }


  if (isdigit(LastChar) || LastChar == '.') {   // Number: [0-9.]+
    std::string NumStr;
    do {
        NumStr += LastChar;
        LastChar = getchar();
    } while (isdigit(LastChar) || LastChar == '.');

    NumVal = strtof(NumStr.c_str(), 0);
    return tok_number;
    // TODO: it will incorrectly read 1.23.45.67 and handle it as if you typed in 1.23.
  }


  if (LastChar == '#') { // Comments
    do
        LastChar = getchar();
    while (LastChar != EOF && LastChar != '\n' && LastChar != '\r');

    if (LastChar != EOF)
        return get_token();
  }


  // Check EOF
  if (LastChar == EOF)
    return tok_eof;


  // Otherwise, just return the character as its ascii value.
  int ThisChar = LastChar;
  LastChar = getchar();
  int otherChar = LastChar;

  if((ThisChar==47)&&(otherChar == 47)){
    LastChar = getchar();
    return 77; //
  }
  return ThisChar;
}


/********************************************************************************************************************************************************/


/// Base class for all Expression Nodes.
class ExprAST {
  public:
    virtual ~ExprAST() = default;
    virtual Value *codegen() = 0;
    // Value can only be changed when the instruction executes, or re-executes
};

/// Expression for numeric literals like "1.0".
class NumberExprAST : public ExprAST {
  float Val;

  public:
    NumberExprAST(float Val) : Val(Val) {}
    Value *codegen() override;
};


/// Expression for referencing a variable, like "a".
class VariableExprAST : public ExprAST {
  std::string Name;

  public:
    VariableExprAST(const std::string &Name) : Name(Name) {}
    Value *codegen() override;
};

/// Expression for a binary operator.
class BinaryExprAST : public ExprAST {
  char Op;
  std::unique_ptr<ExprAST> LHS, RHS;

  public:
    BinaryExprAST(char Op, std::unique_ptr<ExprAST> LHS,
                  std::unique_ptr<ExprAST> RHS)
      : Op(Op), LHS(std::move(LHS)), RHS(std::move(RHS)) {}
    Value *codegen() override;
};

/// Expression for function calls.
class CallExprAST : public ExprAST {
  std::string Callee;
  std::vector<std::unique_ptr<ExprAST>> Args;

  public:
    CallExprAST(const std::string &Callee,
                std::vector<std::unique_ptr<ExprAST>> Args)
      : Callee(Callee), Args(std::move(Args)) {}
    Value *codegen() override;
};


/// PrototypeAST - This class represents the "prototype" for a function,
/// which captures its name, and its argument names (thus implicitly the number
/// of arguments the function takes).
class PrototypeAST {
  std::string Name;
  std::vector<std::string> Args;

  public:
    PrototypeAST(const std::string &Name, std::vector<std::string> Args)
      : Name(Name), Args(std::move(Args)) {}
  Function *codegen();

  const std::string &getName() const { return Name; }
};

/// FunctionAST - This class represents a function definition itself.
class FunctionAST {
  std::unique_ptr<PrototypeAST> Proto;
  std::unique_ptr<ExprAST> Body;

  public:
    FunctionAST(std::unique_ptr<PrototypeAST> Proto,
                std::unique_ptr<ExprAST> Body)
      : Proto(std::move(Proto)), Body(std::move(Body)) {}
  Function *codegen();
};


/********************************************************************************************************************************************************/


// Token Buffer
static int CurTok; // Token Parser
static int getNextToken() {
  return CurTok = get_token();
}


/********************************************************************************************************************************************************/



std::unique_ptr<ExprAST> LogError(const char *Str) {
  fprintf(stderr, "Error: %s\n", Str);
  return nullptr;
}

std::unique_ptr<PrototypeAST> LogErrorP(const char *Str) {
  LogError(Str);
  return nullptr;
}

static std::unique_ptr<LLVMContext> TheContext;
static std::unique_ptr<IRBuilder<>> Builder(TheContext);
static std::unique_ptr<Module> TheModule;
static std::map<std::string, Value *> NamedValues;

// TheContext is an opaque object that owns a lot of core LLVM data structures, such as the type and constant value table.
// Instances of the IRBuilder class template keep track of the current place to insert instructions and has methods to create new instructions.
// TheModule is an LLVM construct that contains functions and global variables. In many ways, it is the top-level structure that the LLVM IR uses to contain code.
//It will own the memory for all of the IR that we generate.
// The NamedValues map keeps track of which values are defined in the current scope and what their LLVM representation is. (In other words, it is a symbol table for the code).
//In this form of Kaleidoscope, the only things that can be referenced are function parameters. As such, function parameters will be in this map when generating code for their function body.


Value *LogErrorV(const char *Str) {
  LogError(Str);
  return nullptr;
}


/********************************************************************************************************************************************************/



static std::unique_ptr<ExprAST> ParseNumberExpr() {
  auto Result = std::make_unique<NumberExprAST>(NumVal);
  getNextToken(); // consume the number
  return std::move(Result);
}

static std::unique_ptr<ExprAST> ParseExpression();


static std::unique_ptr<ExprAST> ParseParenExpr() {
  getNextToken(); // eat (.
  auto V = ParseExpression();
  if (!V)
    return nullptr;

  if (CurTok != ')')
    return LogError("expected ')'");
  getNextToken(); // eat ).
  return V;
}



///   ::= identifier
///   ::= identifier '(' expression* ')'
static std::unique_ptr<ExprAST> ParseIdentifierExpr() {
  std::string IdName = IdentifierStr;

  getNextToken();  // eat identifier.

  if (CurTok != '(') // lookahead for function calls
    return std::make_unique<VariableExprAST>(IdName);

  // Call.
  getNextToken();  // eat (
  std::vector<std::unique_ptr<ExprAST>> Args;
  if (CurTok != ')') {
    while (true) {
      if (auto Arg = ParseExpression())
        Args.push_back(std::move(Arg));
      else
        return nullptr;

      if (CurTok == ')')
        break;

      if (CurTok != ',')
        return LogError("Expected ')' or ',' in argument list");
      getNextToken();
    }
  }

  // Eat the ')'.
  getNextToken();

  return std::make_unique<CallExprAST>(IdName, std::move(Args));
}


static std::map<char, int> BinopPrecedence;

/// GetTokPrecedence - Get the precedence of the pending binary operator token.
static int GetTokPrecedence() {
  if (!isascii(CurTok))
    return -1;

  // Make sure it's a declared binop.
  int TokPrec = BinopPrecedence[CurTok];
  if (TokPrec <= 0) return -1;
  return TokPrec;
}


static std::unique_ptr<ExprAST> ParsePrimary() {
  switch (CurTok) {
  default:
    return LogError("Unknown token parsed");
  case tok_identifier:
    return ParseIdentifierExpr();
  case tok_number:
    return ParseNumberExpr();
  case '(':
    return ParseParenExpr();
  }
}



/// Mount AST with binary operators
static std::unique_ptr<ExprAST> ParseBinOpRHS(int ExprPrec,
                                              std::unique_ptr<ExprAST> LHS) {
  // If this is a binop, find its precedence.
  while (true) {
    int TokPrec = GetTokPrecedence();

    // If this is a binop that binds at least as tightly as the current binop,
    // consume it, otherwise we are done.
    if (TokPrec < ExprPrec)
      return LHS;

    // Okay, we know this is a binop.
    int BinOp = CurTok;
    getNextToken();  // eat binop

    // Parse the primary expression after the binary operator.
    auto RHS = ParsePrimary();
    if (!RHS)
      return LogError("expected any kind value after the operator");
      //return nullptr;

    // If BinOp binds less tightly with RHS than the operator after RHS, let
    // the pending operator take RHS as its LHS.
    int NextPrec = GetTokPrecedence();
    if (TokPrec < NextPrec) {
      RHS = ParseBinOpRHS(TokPrec+1, std::move(RHS));
      if (!RHS)
        return nullptr;
    }
    // Merge LHS/RHS.
    LHS = std::make_unique<BinaryExprAST>(BinOp, std::move(LHS),
                                           std::move(RHS));
  }
}


static std::unique_ptr<ExprAST> ParseExpression() {
  auto LHS = ParsePrimary();
  if (!LHS)
    return nullptr;

  return ParseBinOpRHS(0, std::move(LHS));
}

static std::unique_ptr<PrototypeAST> ParsePrototype() {
  if (CurTok != tok_identifier)
    return LogErrorP("Expected function name");

  std::string FnName = IdentifierStr;
  getNextToken();

  if (CurTok != '(')
    return LogErrorP("Expected '(' before declaring the function args");

  // Read the list of argument names.
  std::vector<std::string> ArgNames;
  while (getNextToken() == tok_identifier)
    ArgNames.push_back(IdentifierStr);
  if (CurTok != ')')
    return LogErrorP("Expected ')' after declaring the function args");

  // success.
  getNextToken();  // eat ')'.

  return std::make_unique<PrototypeAST>(FnName, std::move(ArgNames));
}

static std::unique_ptr<PrototypeAST> ParseExtern() {
  getNextToken();  // eat extern.
  return ParsePrototype();
}


static std::unique_ptr<FunctionAST> ParseTopLevelExpr() {
  if (auto E = ParseExpression()) {
    // Make an anonymous proto.
    auto Proto = std::make_unique<PrototypeAST>("", std::vector<std::string>());
    return std::make_unique<FunctionAST>(std::move(Proto), std::move(E));
  }
  return nullptr;
}


/********************************************************************************************************************************************************/


Value *NumberExprAST::codegen() {
  return ConstantFP::get(*TheContext, APFloat(Val));
  // Arbitrary Precision Float
}

Value *VariableExprAST::codegen() {
  // Look this variable up in the function.
  Value *V = NamedValues[Name];
  // Only function names, loop induction variables and variables are contained at the NamedValues map
  if (!V)
    LogErrorV("Unknown variable name");
  return V;
}

Value *BinaryExprAST::codegen() {
  Value *L = LHS->codegen();
  Value *R = RHS->codegen();
  if (!L || !R)
    return nullptr;
  // Left and Right operators must be of the same type.
  switch (Op) {
  case '+':
    return Builder->CreateFAdd(L, R, "add instruct");
  case '-':
    return Builder->CreateFSub(L, R, "sub instruct");
  case '*':
    return Builder->CreateFMul(L, R, "mul instruct");
  case '/':
    return Builder->CreateFDiv(L, R, "div instruct");
  case 77:
    return LogErrorV("GOTCHA");
  case '<':
    L = Builder->CreateFCmpULT(L, R, "cmp instruct");
    // Convert bool 0/1 to double 0.0 or 1.0
    return Builder->CreateUIToFP(L, Type::getFloatTy(TheContext),
                                 "bool instruct");
  case '>':
    L = Builder->CreateFCmpULT(R, L, "cmp instruct");
    // Convert bool 0/1 to double 0.0 or 1.0
    return Builder->CreateUIToFP(L, Type::getFloatTy(TheContext),
                                 "bool instruct");
  default:
    return LogErrorV("Invalid operator.");
  }
  // TODO: implement ==
}


Value *CallExprAST::codegen() {
  // Look up the name in the symbol table.
  Function *CalleeF = TheModule->getFunction(Callee);
  if (!CalleeF)
    return LogErrorV("Unknown function referenced");

  // If argument mismatch error.
  if (CalleeF->arg_size() != Args.size())
    return LogErrorV("Incorrect # arguments passed");

  std::vector<Value *> ArgsV;
  for (unsigned i = 0, e = Args.size(); i != e; ++i) {
    ArgsV.push_back(Args[i]->codegen());
    if (!ArgsV.back())
      return nullptr;
  }

  return Builder->CreateCall(CalleeF, ArgsV, "calltmp");
}


Function *PrototypeAST::codegen() {
  // Make the function type:  double(double,double) etc.
  std::vector<Type*> Floats(Args.size(),
                             Type::getFloatTy(*TheContext));
  FunctionType *FT =
    FunctionType::get(Type::getFloatTy(*TheContext), Floats, false);

  // Function name is registered at TheModule
  Function *F =
    Function::Create(FT, Function::ExternalLinkage, Name, TheModule.get());
  

  // Set names for all arguments.
  unsigned Idx = 0;
  for (auto &Arg : F->args())
    Arg.setName(Args[Idx++]);

  return F;
}


Function *FunctionAST::codegen() {
  // First, check for an existing function from a previous 'extern' declaration.
  Function *TheFunction = TheModule->getFunction(Proto->getName());


  if (!TheFunction) // Only create the function proto if it has not been created already.
    TheFunction = Proto->codegen();

  if (!TheFunction)
    return nullptr;

  if (!TheFunction->empty())
    return (Function*)LogErrorV("Function cannot be redefined.");

  // Create a new basic block to start insertion into.
  BasicBlock *BB = BasicBlock::Create(*TheContext, "entry", TheFunction);
  Builder->SetInsertPoint(BB);

  // Record the function args at NamedValues map.
  NamedValues.clear();
  for (auto &Arg : TheFunction->args())
    NamedValues[std::string(Arg.getName())] = &Arg;

  if (Value *RetVal = Body->codegen()) {
    // Finish the function.
    Builder->CreateRet(RetVal);

    // Validate the generated code, checking for consistency.
    verifyFunction(*TheFunction);

    return TheFunction;
  }

  // Error reading body, remove function.
  TheFunction->eraseFromParent();
  return nullptr;

}


/********************************************************************************************************************************************************/


static void InitializeModule() {
  // Open a new context and module.
  TheContext = std::make_unique<LLVMContext>();
  TheModule = std::make_unique<Module>("my cool jit", *TheContext);

  // Create a new builder for the module.
  Builder = std::make_unique<IRBuilder<>>(*TheContext);
}


/// top ::= definition | external | expression | ';'
static void MainLoop() {
  while (true) {
    fprintf(stderr, "ready> ");
    switch (CurTok) {
    case tok_eof:
      return;
    case ';': // ignore top-level semicolons.
      getNextToken();
      break;
    case 10: // ignore top-level semicolons.
      getNextToken();
      break;
    case tok_def:
      HandleDefinition();
      break;
    case tok_extern:
      HandleExtern();
      break;
    default:
      HandleTopLevelExpression();
      break;
    }
  }
}


int main() {
  // Install standard binary operators.
  // 1 is lowest precedence.
  BinopPrecedence['<'] = 10;
  BinopPrecedence['+'] = 20;
  BinopPrecedence['-'] = 20;
  BinopPrecedence['/'] = 39;
  BinopPrecedence['*'] = 40;  // highest.
  BinopPrecedence['^'] = 50;
  BinopPrecedence['@'] = 60;

  // Prime the first token.
  fprintf(stderr, "ready> ");
  getNextToken();

  // Make the module, which holds all the code.
  InitializeModule();

  // Run the main "interpreter loop" now.
  MainLoop();

  // Print out all of the generated code.
  TheModule->print(errs(), nullptr);

  return 0;

}


#include "hip/hip_runtime.h"
// JIT
#include "llvm/ADT/APFloat.h"
#include "llvm/ADT/STLExtras.h"
#include "llvm/IR/BasicBlock.h"
#include "llvm/IR/Constants.h"
#include "llvm/IR/DerivedTypes.h"
#include "llvm/IR/Function.h"
#include "llvm/IR/Instructions.h"
#include "llvm/IR/IRBuilder.h"
#include "llvm/IR/LLVMContext.h"
#include "llvm/IR/Module.h"
#include "llvm/IR/Type.h"
#include "llvm/IR/Verifier.h"
#include "llvm/Support/TargetSelect.h"
#include "llvm/Target/TargetMachine.h"

#include "include/KaleidoscopeJIT.h"


#include <algorithm>
#include <cstdarg>
#include <cassert>
#include <cctype>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <map>
#include <memory>
#include <string>
#include <iostream>
#include <numeric>
#include <string>
#include <utility>
#include <vector>
#include <iomanip>
#include <math.h>
#include <fenv.h>
#include <tuple>
#include <glob.h>


// Cuda
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <omp.h>

#include "include/cu_commons.h"


// Files
#define STB_IMAGE_IMPLEMENTATION
#include "include/stb/stb_image.h"


static hipblasHandle_t cublas_handle;
static hipblasLtHandle_t cublaslt_handle;

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
static size_t cublaslt_workspace_size = 32 * 1024 * 1024; // 32 MB
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;


hipblasComputeType_t cublas_compute = HIPBLAS_COMPUTE_32F;

#define CUBLAS_LOWP HIP_R_32F
#define PRECISION_MODE PRECISION_FP32



using namespace llvm;
using namespace llvm::orc;

#if __CUDA_ARCH__ == 800 || __CUDA_ARCH__ >= 900
#define MAX_1024_THREADS_BLOCKS 2
#else
#define MAX_1024_THREADS_BLOCKS 1
#endif

// Error Colors

// \033[0m default
// \033[31m red
// \033[33m yellow
// \033[95m purple


bool ends_with(std::string str_input, std::string str_end)
{
  return str_input.size() >= str_end.size() && str_input.compare(str_input.size() - str_end.size(), str_end.size(), str_end) == 0;
}

bool starts_with(const char* str, const char* sub) {
  return strncmp(str, sub, strlen(sub)) == 0;
}

std::vector<std::string> split_str(const std::string& str, char delimiter) {
  std::vector<std::string> tokens;
  std::string token;
  std::istringstream stream(str);

  while (std::getline(stream, token, delimiter)) {
    tokens.push_back(token);
  }

  return tokens;
}

std::vector<std::string> split(const char* input, const std::string& delimiter) {
    std::vector<std::string> tokens;
    size_t start = 0, end = 0;
    while ((end = std::string(input + start).find(delimiter)) != std::string::npos) {
        tokens.push_back(std::string(input + start, end));
        start += end + delimiter.length();
    }
    tokens.push_back(std::string(input + start));
    return tokens;
}




bool in_str(std::string str, std::vector<std::string> list) {
    return std::find(list.begin(), list.end(), str) != list.end();
}


std::vector<std::string> tensor_methods = {"view","permute", "onehot", "mean", "sum", "max", "min"};
std::vector<std::string> vararg_methods = {"view", "Datasetyield"};
std::vector<std::string> tensor_inits = {"randint", "randu", "zeros", "ones", "xavu", "xavn"};


//===----------------------------------------------------------------------===//
// Lexer
//===----------------------------------------------------------------------===//


// The lexer returns tokens [0-255] if it is an unknown character, otherwise one
// of these for known things.
enum Token {
  tok_eof = -1,

  // functions/classes
  tok_def = -2,
  tok_class = -77,
  tok_self = -78,
  tok_class_attr = -79,
  tok_extern = -3,

  // primary
  tok_identifier = -4,
  tok_number = -5,
  tok_str = -40,

  // control
  tok_if = -6,
  tok_then = -7,
  tok_else = -8,
  tok_for = -9,
  tok_while = -10,
  tok_tab = 9,

  // operators
  tok_binary = -11,
  tok_unary = -12,


  tok_space = -14,

  
  // var definition
  tok_var = -15,
  tok_tensor = -16,
  tok_var_str = -17,
  tok_attr_var = -18,
  tok_attr_tensor = -19,

  // function ops
  tok_log = -30
};

static std::string IdentifierStr; // Filled in if tok_identifier
static float NumVal;             // Filled in if tok_number
int LineCounter;

/// get_token - Return the next token from standard input.
static int get_token() {
  static int LastChar = ' ';

  // Skip any whitespace and backspace.
  //while (LastChar==32 || LastChar==tok_tab)
  while (LastChar==32 || LastChar==tok_tab)
    LastChar = getchar();
  //while (isspace(LastChar))
    
  if (LastChar=='"')
  {

    LastChar = getchar();
    IdentifierStr = LastChar;

    bool name_ok=true;
    while (name_ok)
    {
      LastChar = getchar();
      
      if(LastChar!='"')
        IdentifierStr += LastChar;
      else
        name_ok = false;

    }
    LastChar = getchar();
    
    return tok_str;
  }

  if (isalpha(LastChar) || LastChar=='_') { // identifier: [a-zA-Z][a-zA-Z0-9]*
    IdentifierStr = LastChar;
    bool name_ok=true;
    while (name_ok)
    {
      LastChar = getchar();
      
      if(isalnum(LastChar) || LastChar=='_')
        IdentifierStr += LastChar;
      else
        name_ok = false;

      if (IdentifierStr == "tensor" && LastChar=='[')
      {
        LastChar = getchar();
        return tok_tensor;
      }
      if (LastChar=='.')
      {
        LastChar = getchar();
        if (IdentifierStr == "self")
          return tok_self;
        return tok_class_attr;
      }
    }

    if (IdentifierStr == "def")
      return tok_def;
    if (IdentifierStr == "class")
      return tok_class;
    if (IdentifierStr == "extern")
      return tok_extern;
    if (IdentifierStr == "if")
      return tok_if;
    if (IdentifierStr == "then")
      return tok_then;
    if (IdentifierStr == "else")
      return tok_else;
    if (IdentifierStr == "for")
      return tok_for;
    if (IdentifierStr == "while")
      return tok_while;
    if (IdentifierStr == "binary")
      return tok_binary;
    if (IdentifierStr == "unary")
      return tok_unary;
    if (IdentifierStr == "var")
      return tok_var;
    if (IdentifierStr == "log")
      return tok_log;
    if (IdentifierStr == "glob")
      IdentifierStr = "_glob_b_";
    if (IdentifierStr == "str")
      return tok_var_str;
    
    return tok_identifier;
  }

  if (isdigit(LastChar) || LastChar == '.') { // Number: [-.]+[0-9.]+
    std::string NumStr;
    if (LastChar == '-') { // Check for optional minus sign
      NumStr += LastChar;
      LastChar = getchar();
    }
    do {
      NumStr += LastChar;
      LastChar = getchar();
    } while (isdigit(LastChar) || LastChar == '.');

    NumVal = strtod(NumStr.c_str(), nullptr);
    return tok_number;
  }

  if (LastChar == '#') {
    // Comment until end of line.
    do
      LastChar = getchar();
    while (LastChar != EOF && LastChar != '\n' && LastChar != tok_space && LastChar != '\r');

    if (LastChar != EOF)
      return get_token();
  }

  // Check for end of file.  Don't eat the EOF.
  if (LastChar == EOF)
    return tok_eof;

  // Otherwise, just return the character as its ascii value.
  int ThisChar = LastChar;
  LastChar = getchar();
  int otherChar = LastChar;


  
  if(ThisChar==10)
  {
    LineCounter += 1;
    return tok_space;
  }

  if((ThisChar==47)&&(otherChar == 47)){
    LastChar = getchar();
    return 77; //
  }
  return ThisChar;
}

//===----------------------------------------------------------------------===//
// Abstract Syntax Tree (aka Parse Tree)
//===----------------------------------------------------------------------===//

/// ExprAST - Base class for all expression nodes.
class ExprAST {
public:
  virtual ~ExprAST() = default;
  std::vector<float> Dims = {-1.0f};
  std::string Type = "None";
  std::string Name = "Unnamed";
  std::string isSelf = "false";

  virtual Value *codegen() = 0;
  virtual void SetType(std::string Type) {
    this->Type=Type;
  }
  virtual std::string GetType() {
    return Type;
  }
  virtual void SetSelf(std::string Self) {
    this->isSelf=Self;
  }
  virtual std::string GetSelf() {
    return isSelf;
  }
  virtual std::string GetName() {
    return Name;
  }
  virtual void SetName(std::string Name) {
    this->Name=Name;
  }
  virtual std::vector<float> GetDims() {
    return Dims;
  }
  virtual void SetDims(std::vector<float> Dims) {
    this->Dims=Dims;
  }
};

/// NumberExprAST - Expression class for numeric literals like "1.0".
class NumberExprAST : public ExprAST {
  float Val;

  public:
    NumberExprAST(float Val) : Val(Val) {} //{std::cout << "number created";}
  std::string Type = "num";

  Value *codegen() override;
};



class StringExprAST : public ExprAST {
  std::string Val;

  public:
    StringExprAST(std::string Val) : Val(Val) {} //{std::cout << "number created";}
  std::string Type = "str";

  Value *codegen() override;
};



/// VariableExprAST - Expression class for referencing a variable, like "a".
class VariableExprAST : public ExprAST {
  std::string Name;

  public:
    VariableExprAST(const std::string &Name) : Name(Name) {}

    Value *codegen() override;
    const std::string &getName() const { return Name; }
    std::string GetName() override {
    return Name;
  }
};


/// VarExprAST - Expression class for var/in
class VarExprAST : public ExprAST {

  public:
    std::vector<std::pair<std::string, std::unique_ptr<ExprAST>>> VarNames;
    std::unique_ptr<ExprAST> Body;
    std::string Type;
    VarExprAST(
        std::vector<std::pair<std::string, std::unique_ptr<ExprAST>>> VarNames,
        std::unique_ptr<ExprAST> Body,
        std::string Type)
        : VarNames(std::move(VarNames)), Body(std::move(Body)), Type(Type) {}

  Value *codegen() override;
};

class StrExprAST : public ExprAST {

  public:
    std::vector<std::pair<std::string, std::unique_ptr<ExprAST>>> VarNames;
    std::unique_ptr<ExprAST> Body;
    std::string Type;
    StrExprAST(
        std::vector<std::pair<std::string, std::unique_ptr<ExprAST>>> VarNames,
        std::unique_ptr<ExprAST> Body,
        std::string Type)
        : VarNames(std::move(VarNames)), Body(std::move(Body)), Type(Type) {}

  Value *codegen() override;
};

class TensorExprAST : public VarExprAST {
  public:
    std::vector<std::unique_ptr<ExprAST>> V_Dims;
    std::string TensorInit;

    TensorExprAST(
      std::vector<std::pair<std::string, std::unique_ptr<ExprAST>>> VarNames,
      std::unique_ptr<ExprAST> Body,
      std::string Type,
      std::vector<std::unique_ptr<ExprAST>> V_Dims,
      const std::string &TensorInit)
      : VarExprAST(std::move(VarNames), std::move(Body), std::move(Type)),
                   V_Dims(std::move(V_Dims)), TensorInit(TensorInit) {}

  Value *codegen() override;
};

class LogExprAST : public ExprAST {
  std::string Name;

  public:
    LogExprAST(const std::string &Name) : Name(Name) {}

    Value *codegen() override;
    std::string GetName() override {
      return Name;
    }
};

class LossBackwardExprAST : public ExprAST {
  public:
    LossBackwardExprAST() {}
    Value *codegen() override;
};


/// UnaryExprAST - Expression class for a unary operator.
class UnaryExprAST : public ExprAST {
  char Opcode;
  std::unique_ptr<ExprAST> Operand;

public:
  UnaryExprAST(char Opcode, std::unique_ptr<ExprAST> Operand)
      : Opcode(Opcode), Operand(std::move(Operand)) {}

  Value *codegen() override;
};



/// BinaryExprAST - Expression class for a binary operator.
class BinaryExprAST : public ExprAST {
  char Op;
  std::unique_ptr<ExprAST> LHS, RHS;

public:
  BinaryExprAST(char Op, std::unique_ptr<ExprAST> LHS,
                std::unique_ptr<ExprAST> RHS)
      : Op(Op), LHS(std::move(LHS)), RHS(std::move(RHS)) {}

  Value *codegen() override;
};


class BinaryTensorScalarExprAST : public ExprAST {
  char Op;
  std::unique_ptr<ExprAST> LHS, RHS;

public:
  BinaryTensorScalarExprAST(char Op, std::unique_ptr<ExprAST> LHS,
                std::unique_ptr<ExprAST> RHS)
      : Op(Op), LHS(std::move(LHS)), RHS(std::move(RHS)) {}

  Value *codegen() override;
};


class BinaryTensorTensorExprAST : public ExprAST {
  char Op;
  std::unique_ptr<ExprAST> LHS, RHS;

public:
  BinaryTensorTensorExprAST(char Op, std::unique_ptr<ExprAST> LHS,
                std::unique_ptr<ExprAST> RHS)
      : Op(Op), LHS(std::move(LHS)), RHS(std::move(RHS)) {}

  Value *codegen() override;
};


/// CallExprAST - Expression class for function calls.
class CallExprAST : public ExprAST {
  std::string Callee;
  std::vector<std::unique_ptr<ExprAST>> Args;
  std::string Class;
  std::string Pre_dot;

  public:
    CallExprAST(const std::string &Callee,
                std::vector<std::unique_ptr<ExprAST>> Args,
                const std::string &Class,
                const std::string &Pre_dot)
        : Callee(Callee), Args(std::move(Args)), Class(Class), Pre_dot(Pre_dot) {}

  Value *codegen() override;
};

/// IfExprAST - Expression class for if/then/else.
class IfExprAST : public ExprAST {
  std::unique_ptr<ExprAST> Cond, Then, Else;

  public:
    IfExprAST(std::unique_ptr<ExprAST> Cond, std::unique_ptr<ExprAST> Then,
              std::unique_ptr<ExprAST> Else)
        : Cond(std::move(Cond)), Then(std::move(Then)), Else(std::move(Else)) {}

  Value *codegen() override;
};

/// ForExprAST - Expression class for for.
class ForExprAST : public ExprAST {
  std::string VarName;
  std::unique_ptr<ExprAST> Start, End, Step, Body;

  public:
    ForExprAST(const std::string &VarName, std::unique_ptr<ExprAST> Start,
              std::unique_ptr<ExprAST> End, std::unique_ptr<ExprAST> Step,
              std::unique_ptr<ExprAST> Body)
        : VarName(VarName), Start(std::move(Start)), End(std::move(End)),
          Step(std::move(Step)), Body(std::move(Body)) {}

  Value *codegen() override;
};

/// WhileExprAST - Expression class for while.
class WhileExprAST : public ExprAST {
	std::unique_ptr<ExprAST> Cond, Body;

  public:
    WhileExprAST(std::unique_ptr<ExprAST> Cond, std::unique_ptr<ExprAST> Body)
      : Cond(std::move(Cond)), Body(std::move(Body)) {}

	Value* codegen() override;
};


/// PrototypeAST - This class represents the "prototype" for a function,
/// which captures its name, and its argument names (thus implicitly the number
/// of arguments the function takes), as well as if it is an operator.
class PrototypeAST {
  std::string Name;
  std::vector<std::string> Args;
  std::vector<std::string> Types;
  bool IsOperator;
  unsigned Precedence; // Precedence if a binary op.

  public:
    PrototypeAST(const std::string &Name, std::vector<std::string> Args,
                std::vector<std::string> Types,
                bool IsOperator = false, unsigned Prec = 0)
        : Name(Name), Args(std::move(Args)), Types(std::move(Types)),
          IsOperator(IsOperator), Precedence(Prec) {}

  Function *codegen();
  const std::string &getName() const { return Name; }

  bool isUnaryOp() const { return IsOperator && Args.size() == 1; }
  bool isBinaryOp() const { return IsOperator && Args.size() == 2; }

  char getOperatorName() const {
    assert(isUnaryOp() || isBinaryOp());
    return Name[Name.size() - 1];
  }

  unsigned getBinaryPrecedence() const { return Precedence; }
};


class ClassAST : public ExprAST {
  std::vector<std::unique_ptr<FunctionAST>> Functions;

  public:
    ClassAST(std::vector<std::unique_ptr<FunctionAST>> Functions)
        : Functions(std::move(Functions)) {}
  
  const PrototypeAST& getProto(int i) const;
  const std::string& getName(int i) const;
  
  Value *codegen();
};

//===----------------------------------------------------------------------===//
// Parser
//===----------------------------------------------------------------------===//

/// CurTok/getNextToken - Provide a simple token buffer.  CurTok is the current
/// token the parser is looking at.  getNextToken reads another token from the
/// lexer and updates CurTok with its results.
static int CurTok;
static int getNextToken() { return CurTok = get_token(); }

/// BinopPrecedence - This holds the precedence for each binary operator that is
/// defined.
static std::map<char, int> BinopPrecedence;

/// get_tokenPrecedence - Get the precedence of the pending binary operator token.
static int get_tokenPrecedence() {
  if (CurTok==tok_space)
    return 1;

  if (!isascii(CurTok))
    return -1;

  // Make sure it's a declared binop.
  int TokPrec = BinopPrecedence[CurTok];
  if (TokPrec <= 0)
    return -1;
  return TokPrec;
}



/// LogError* - These are little helper functions for error handling.
//std::unique_ptr<ExprAST> LogError(const char *Str) {
std::unique_ptr<ExprAST> LogErrorS(std::string Str) {
  //fprintf(stderr, "\033[31m Erro: \033[0m%s\n", Str);
  if (Str!=" ")
    std::cout << "\nLinha: " << LineCounter << "\n   \033[31m Erro: \033[0m " << Str << "\n\n";
  
  
  return nullptr;
}

std::unique_ptr<ExprAST> LogError(std::string Str) {
  //fprintf(stderr, "\033[31m Erro: \033[0m%s\n", Str);
  LogErrorS(Str);

  while(CurTok!=tok_space && CurTok!=';' && CurTok!=',' && CurTok!=')')
    getNextToken();
  
  return nullptr;
}

std::unique_ptr<ExprAST> LogErrorBreakLine(std::string Str) {
  //fprintf(stderr, "\033[31m Erro: \033[0m%s\n", Str);
  LogErrorS(Str);

  while(CurTok!=tok_space && CurTok!=';')
    getNextToken();
  
  return nullptr;
}


void LogWarning(const char *Str) {
  std::cout << "\nLinha: " << LineCounter << "\n   \033[33m Aviso: \033[0m " << Str << "\n\n";
}

// Modified LogError function with token parameter
std::unique_ptr<ExprAST> LogErrorT(int CurTok) {
  //char buf[100];
  //snprintf(buf, sizeof(buf), "token %d inesperado.", CurTok);
  //fprintf(stderr, "\033[31mErro: \033[0m%s\n", buf);
  std::cout << "\nLinha: " << LineCounter << "\n   \033[31m Erro: \033[0mtoken " << IdentifierStr << " inesperado. Esperava-se uma expressão.\n\n";
  return nullptr;
}


std::unique_ptr<PrototypeAST> LogErrorP(const char *Str) {
  LogError(Str);
  while(CurTok!=tok_space && CurTok!=';')
    getNextToken();
  return nullptr;
}


std::unique_ptr<PrototypeAST> LogErrorP_to_comma(const char *Str) {
  LogError(Str);
  while(CurTok!=tok_space && CurTok!=';' && CurTok!=',' && CurTok!=')')
  {
    std::cout << "LogErrorP: " << IdentifierStr << "\n";
    
    getNextToken();
    }
  return nullptr;
}

Value *LogErrorV(std::string Str) {
  LogError(Str);
  return nullptr;
}

static std::unique_ptr<ExprAST> ParseExpression(int tabcount=0);

/// numberexpr ::= number
static std::unique_ptr<ExprAST> ParseNumberExpr(int tabcount=0) {
  auto Result = std::make_unique<NumberExprAST>(NumVal);
  getNextToken(); // consume the number
  return std::move(Result);
}

static std::unique_ptr<ExprAST> ParseStringExpr(int tabcount=0) {
  auto Result = std::make_unique<StringExprAST>(IdentifierStr);
  getNextToken(); // consume the "
  return std::move(Result);
}

/// parenexpr ::= '(' expression ')'
static std::unique_ptr<ExprAST> ParseParenExpr() {
  getNextToken(); // eat (.
  auto V = ParseExpression();
  if (!V)
    return nullptr;

  if (CurTok != ')')
    return LogError("Esperado ')' na expressão em paranteses");
  
  std::cout << "Close brackets\n";
  getNextToken(); // eat ).
  return V;
}

std::vector<std::string> tensorVars;



static std::vector<std::string> Classes;
static std::map<std::string, std::string> Object_toClass;


/// identifierexpr
///   ::= identifier
///   ::= identifier '(' expression* ')'
static std::unique_ptr<ExprAST> ParseIdentifierExpr(int tabcount=0) {
  
  for(int i=0; i<Classes.size(); i++)
    if(IdentifierStr==Classes[i]) 
    {
      getNextToken();
      std::cout << "Object name: " << IdentifierStr << " and Class: " << Classes[i]<< "\n";
      Object_toClass[IdentifierStr] = Classes[i];
      
      getNextToken();
      return std::move(std::make_unique<NumberExprAST>(0.0f));
    }

  std::string IdName = IdentifierStr;

  getNextToken(); // eat identifier.
  
  if (CurTok != '(') // Simple variable ref.
  {
    auto aux = std::make_unique<VariableExprAST>(IdName);
    if (std::find(tensorVars.begin(), tensorVars.end(), IdentifierStr) != tensorVars.end())
      aux->SetType("tensor");
    //std::cout << "call arg identifier type: " << aux->GetType() <<  "\n";
    return aux;
  }

  

  // Call.
  getNextToken(); // eat (
  std::vector<std::unique_ptr<ExprAST>> Args;
  if (CurTok != ')') {
    while (true) {
      
      if (auto Arg = ParseExpression())
        Args.push_back(std::move(Arg));
      else
        return nullptr;
      

      if (CurTok == ')')
        break;

      if (CurTok != ',')
        return LogError("Esperado ')' ou ',' na lista de argumentos");
      getNextToken();
    }
  }

  
  

  // Eat the ')'.
  getNextToken();

  return std::make_unique<CallExprAST>(IdName, std::move(Args), "None", "None");
}

/// ifexpr ::= 'if' expression 'then' expression 'else' expression
static std::unique_ptr<ExprAST> ParseIfExpr(int tabcount=1) {
  
  //std::cout << tabcount << " " << CurTok << "token if atual\n";
  if(CurTok==tok_space)
    getNextToken();

  getNextToken(); // eat the if.
  //CurTok = '(';
  //std::cout << CurTok << "token if posterior\n";
  

  //std::cout << CurTok << " Cond token \n";
  // condition.
  auto Cond = ParseExpression(tabcount+1);
  if (!Cond)
    return nullptr;

  if(CurTok==tok_space)
    getNextToken();


  //std::cout << "If then token " << CurTok << "\n";
  auto Then = ParseExpression(tabcount+1);
  //std::cout << "Then finished \n";
  if (!Then)
  {
    //std::cout << "Then is null \n";
    return nullptr;
  }
  
  //std::cout << "If else token " << CurTok << "\n";
  
  if(CurTok==tok_space)
    getNextToken();


  if (CurTok != tok_else){
    auto Else = std::make_unique<NumberExprAST>(0);
    getNextToken();
    return std::make_unique<IfExprAST>(std::move(Cond), std::move(Then),
                                      std::move(Else));
  }
  else {
    getNextToken();

    auto Else = ParseExpression(tabcount+1);
    if (!Else)
      return nullptr;
    
    return std::make_unique<IfExprAST>(std::move(Cond), std::move(Then),
                                      std::move(Else));
  }
}

/// forexpr ::= 'for' identifier '=' expr ',' expr (',' expr)? 'in' expression
static std::unique_ptr<ExprAST> ParseForExpr() {
  getNextToken(); // eat the for.

  if (CurTok != tok_identifier)
    return LogError("identificador da variável de controle esperado depois do for.");

  std::string IdName = IdentifierStr;
  getNextToken(); // eat identifier.

  if (CurTok != '=')
    return LogError("Esperada atribuição do valor inicial do for.");
  getNextToken(); // eat '='.

  auto Start = ParseExpression(0);
  if (!Start)
    return nullptr;
  if (CurTok != ',')
    return LogError("Esperado ',' depois de atribuir valor inicial do for.");
  getNextToken();

  auto End = ParseExpression(0);
  if (!End)
    return nullptr;

  std::unique_ptr<ExprAST> Step = std::make_unique<NumberExprAST>(1.0);
  if (CurTok == ',') { // The step value is optional.
    getNextToken();
    auto aux = ParseExpression(0);
    if (aux)
      Step = std::move(aux);
  }

  
  auto Body = ParseExpression();
  if (!Body)
    return nullptr;

  return std::make_unique<ForExprAST>(IdName, std::move(Start), std::move(End),
                                       std::move(Step), std::move(Body));
}


/// whileexpr ::= 'while' identifier '=' expr ',' expr (',' expr)? 'in' expression
static std::unique_ptr<ExprAST> ParseWhileExpr() {
  getNextToken(); // eat the while.

  if (CurTok != tok_identifier)
    return LogError("Identificador da variável de controle esperado depois do while.");


  auto Cond = ParseExpression(0);
  if (!Cond)
    return nullptr;
  
  auto Body = ParseExpression();
  if (!Body)
    return nullptr;

  return std::make_unique<WhileExprAST>(std::move(Cond), std::move(Body));
}


/// varexpr ::= 'var' identifier ('=' expression)?
//                    (',' identifier ('=' expression)?)* 'in' expression
static std::unique_ptr<ExprAST> ParseVarExpr() {
  getNextToken(); // eat the var.
  

  // mem2reg is alloca-driven: it looks for allocas and if it can handle them, it promotes them. It DOES NOT APPLY TO GLOBAL variables or heap allocations.
  // mem2reg only promotes allocas whose uses are direct loads and stores. If the address of the stack object is passed to a function,
  //or if any funny pointer arithmetic is involved, the alloca will not be promoted.

  std::vector<std::pair<std::string, std::unique_ptr<ExprAST>>> VarNames;

  // At least one variable name is required.
  if (CurTok != tok_identifier)
    return LogError("Esperado identificador após var.");

  while (true) {
    std::string Name = IdentifierStr;
    getNextToken(); // eat identifier.

    // Read the optional initializer.
    std::unique_ptr<ExprAST> Init = nullptr;
    if (CurTok == '=') {
      getNextToken(); // eat the '='.

      Init = ParseExpression();
      if (!Init)
        return nullptr;
    }

    VarNames.push_back(std::make_pair(Name, std::move(Init)));

    // End of var list, exit loop.
    if (CurTok != ',')
      break;
    getNextToken(); // eat the ','.

    if (CurTok != tok_identifier)
      return LogError("Esperado um ou mais identificadores após var.");
  }

  auto Body = ParseExpression();
  if (!Body)
    return nullptr;

  return std::make_unique<VarExprAST>(std::move(VarNames), std::move(Body), "var");
}


static std::unique_ptr<ExprAST> ParseStrExpr() {
  getNextToken(); // eat the var.
  

  // mem2reg is alloca-driven: it looks for allocas and if it can handle them, it promotes them. It DOES NOT APPLY TO GLOBAL variables or heap allocations.
  // mem2reg only promotes allocas whose uses are direct loads and stores. If the address of the stack object is passed to a function,
  //or if any funny pointer arithmetic is involved, the alloca will not be promoted.

  std::vector<std::pair<std::string, std::unique_ptr<ExprAST>>> VarNames;

  // At least one variable name is required.
  if (CurTok != tok_identifier)
    return LogError("Esperado identificador após var.");

  while (true) {
    std::string Name = IdentifierStr;
    getNextToken(); // eat identifier.

    // Read the optional initializer.
    std::unique_ptr<ExprAST> Init = nullptr;
    if (CurTok == '=') {
      getNextToken(); // eat the '='.

      Init = ParseStringExpr();
      if (!Init)
        return nullptr;
    }

    VarNames.push_back(std::make_pair(Name, std::move(Init)));

    // End of var list, exit loop.
    if (CurTok != ',')
      break;
    getNextToken(); // eat the ','.

    if (CurTok != tok_identifier)
      return LogError("Esperado um ou mais identificadores após var.");
  }

  auto Body = ParseExpression();
  if (!Body)
    return nullptr;

  return std::make_unique<StrExprAST>(std::move(VarNames), std::move(Body), "str");
}




int used_cuda = 0;
unsigned char* current_data_attr;
std::vector<float> current_data_attr_dims;


extern "C" float * load_img(char *img_name)
{
  used_cuda=0;
  int width, height, channels;
  
  unsigned char* image_data = stbi_load(img_name, &width, &height, &channels, 0);

  if (image_data) {
    
    current_data_attr_dims.clear();
    current_data_attr_dims.push_back((float)width);
    current_data_attr_dims.push_back((float)height);
    current_data_attr_dims.push_back((float)channels);

    /*
    std::cout << "Width: " << width << " pixels\n";
    std::cout << "Height: " << height << " pixels\n";
    std::cout << "Channels: " << channels << "\n";
    */

    //stbi_image_free(image_data);

    float *image_data_float = new float[width * height * channels];
  
    // Loop through each pixel and convert to float between 0.0 and 1.0
    for (int y = 0; y < height; ++y) {
      for (int x = 0; x < width; ++x) {
        for (int c = 0; c < channels; ++c) {
          // Assuming unsigned char has 8 bits, scale by 1/255.0 to get a float value between 0.0 and 1.0
          image_data_float[(y * width + x) * channels + c] = (float)image_data[(y * width + x) * channels + c] / 255.0f;
        }
      }
    }

    return image_data_float;
    
  } else {
    std::string img_n = img_name;
    std::string _error = "Falha ao abrir a imagem: " + img_n + ".";
    LogErrorS(_error);
  }

  return nullptr;
}


extern "C" float * split_str_to_float(char *in_string)
{
  std::cout << "split_str_to_float CALLED\n";
  std::vector<std::string> splitted = split_str(in_string,'/');

  float * ret = new float[1];
  ret[0] = std::stof(splitted[splitted.size()-2]);
  std::cout << "Split retrieval: " << ret[0] << "\n";
  return ret;
}




std::map<std::string, std::function<float *(char*)>> preprocessings;




static std::unique_ptr<ExprAST> ParseSelfExpr() {

  std::string pre_dot = IdentifierStr;
  std::string object_class;
  bool is_class_attr=false;

  //std::cout << "CLASS ATTR IS: " << IdentifierStr << "\n";

  if (CurTok!=tok_self)
  {
    is_class_attr = true;
    pre_dot="";

    
    while (CurTok==tok_class_attr)
    {
      object_class=IdentifierStr;
      pre_dot+=IdentifierStr;
      getNextToken();
    }
    
    //std::cout << "Search object method: " << IdentifierStr <<  "\n";
    if (Object_toClass.find(object_class) != Object_toClass.end())
    {
      //std::cout << "Found object to class for\n";
      object_class = Object_toClass[object_class]; 
    }
  } else
    getNextToken(); // eat object or self token.
  
  
  //std::cout << "Pre-dot: " << pre_dot << " Post-dot: " << IdentifierStr  << "\n";


  std::string IdName = IdentifierStr;

  getNextToken(); // eat identifier.

  
  if (pre_dot=="loss" and IdName=="backward")
  {
    if (CurTok != '(')
      LogErrorBreakLine("Precisa do '(' na chamada do backward");
    getNextToken();
    if (CurTok != ')')
      LogErrorBreakLine("Precisa do ')' na chamada do backward");
    getNextToken();
    return std::make_unique<LossBackwardExprAST>();
  }

  if (CurTok != '(') // Simple variable ref.
  {
    auto aux = std::make_unique<VariableExprAST>(IdName);
    if (std::find(tensorVars.begin(), tensorVars.end(), IdentifierStr) != tensorVars.end())
      aux->SetType("tensor");
    if (is_class_attr)
      aux->SetSelf(pre_dot);
    if (pre_dot=="self")
      aux->SetSelf("true");
    
    if (starts_with(IdName.c_str(), "preprocess_") && pre_dot=="self")
    {
      getNextToken(); // eat =
      std::cout << "\nPARSED PREPROCESS: " << IdName << "\n\n";
      
      std::vector<std::string> preprocessings_vec = split_str(IdentifierStr, ',');
      for (int i=0; i<preprocessings_vec.size(); i++)
      {
        std::cout << "Cur identifier:" << IdentifierStr << "\n";
        if (IdentifierStr=="load_img")
        {
          preprocessings[IdName] = load_img;
        }
        if (starts_with(IdentifierStr.c_str(), "split_str_to_float"))
        {
          std::cout << "SPLIT\n";
          preprocessings[IdName] = split_str_to_float;
        }
      }

      getNextToken();
    }
    
    return aux;
  }

  // Call.
  getNextToken(); // eat (
  std::vector<std::unique_ptr<ExprAST>> Args;
  if (CurTok != ')') {
    while (true) {
      if (auto Arg = ParseExpression())
        Args.push_back(std::move(Arg));
      else
        return nullptr;

      if (CurTok == ')')
        break;

      if (CurTok != ',')
        return LogError("Esperado ')' ou ',' na lista de argumentos");
      getNextToken();
    }
  }

  if (IdName=="view")
    Args.push_back(std::make_unique<NumberExprAST>(-2.0f));
  if (ends_with(IdName, "yield"))
    Args.push_back(std::make_unique<StringExprAST>("-2"));

  // Eat the ')'.
  getNextToken();


  return std::make_unique<CallExprAST>(IdName, std::move(Args), object_class, pre_dot);
}


static std::unique_ptr<ExprAST> ParseTensorExpr() {
  // TODO: Allow finishing line with ";"
  getNextToken(); // eat the tensor.
  
  std::vector<std::unique_ptr<ExprAST>> dims;
  std::vector<std::pair<std::string, std::unique_ptr<ExprAST>>> VarNames;
  std::string init = "randu";
  //std::make_unique<NumberExprAST>(NumVal)
  
  while (true) {
    if (CurTok != tok_number && CurTok != tok_identifier && CurTok != tok_self)
      return LogError("Esperado número da dimensão do tensor.");
    
    if (CurTok==tok_number)
    {
      if (std::fmod(NumVal, 1.0) != 0)
        LogWarning("A dimensão do tensor precisa ser int. Não pode ser float ou double.");
    
      dims.push_back(std::make_unique<NumberExprAST>( (float)((int)round(NumVal)) ));
      getNextToken();
    } else if (CurTok==tok_identifier)
      if (in_str(IdentifierStr, tensor_inits))
      {
        init = IdentifierStr;
        getNextToken();
      } else
        dims.push_back(std::move(ParseIdentifierExpr()));
    else {
      dims.push_back(std::move(ParseSelfExpr()));
    }

    
    if (CurTok != ',')
      break;
    getNextToken(); // eat the ','.
  }

  
  if (CurTok != ']')
    return LogError("Faltou fechar ].");
    getNextToken();


  std::string pre_dot = "false";
  if (CurTok == tok_self)
  {
    pre_dot = "true";
    getNextToken();
  }
  if (CurTok == tok_class_attr)
  {
    pre_dot = IdentifierStr;
    std::cout << "Obj attr tensor: " << pre_dot << "\n";
    getNextToken();
  }

  if (CurTok != tok_identifier)
    return LogError("Esperado identificador após var.");

  while (true) {
    std::string Name = IdentifierStr;
    tensorVars.push_back(IdentifierStr);
    getNextToken(); // eat identifier.

    
    std::unique_ptr<ExprAST> Init = nullptr;
    VarNames.push_back(std::make_pair(Name, std::move(Init)));

    // End of var list, exit loop.
    if (CurTok != ',')
      break;
    getNextToken(); // eat the ','.

    if (CurTok != tok_identifier)
      return LogError("Esperado um ou mais identificadores após var.");
  }


  std::unique_ptr<ExprAST> Body;
  if (CurTok==';')
    Body = std::make_unique<NumberExprAST>(0.0f);
  else {  
    Body = ParseExpression();
    if (!Body)
      return nullptr;
  }

  auto aux = std::make_unique<TensorExprAST>(std::move(VarNames), std::move(Body), "tensor",
                                             std::move(dims), init);
  aux->SetSelf(pre_dot);
  
  return aux;
}



static std::unique_ptr<ExprAST> ParseLogExpr() {
  getNextToken(); // eat the log.
  

  if (CurTok != '(')
    return LogError("Esperado ( após a função log.");
  getNextToken();
  

  if (CurTok != tok_identifier)
    return LogError("Esperado tensor à função log.");
  
  std::string Name = IdentifierStr;
  getNextToken();
  

  if (CurTok != ')')
    return LogError("Esperado ) na função log.");
  getNextToken();

  auto aux = std::make_unique<LogExprAST>(std::move(Name));
  aux->SetType("tensor");
  return aux;
}



/// primary
///   ::= identifierexpr
///   ::= numberexpr
///   ::= parenexpr
///   ::= ifexpr
///   ::= forexpr
///   ::= varexpr
static std::unique_ptr<ExprAST> ParsePrimary(int tabcount=0) {
  while(CurTok==tok_tab)
    getNextToken();
  switch (CurTok) {
  default:
    std::cout << CurTok << " token atual de erro esperando expressão\n";
    return LogErrorT(CurTok);
  case tok_identifier:
    return ParseIdentifierExpr(tabcount);
  case tok_class_attr:
    return ParseSelfExpr();
  case tok_self:
    return ParseSelfExpr();
  case tok_number:
    return ParseNumberExpr(tabcount);
  case tok_str:
    return ParseStringExpr();
  case tok_var_str:
    return ParseStrExpr();
  case '(':
    return ParseParenExpr();
  case tok_if:
    return ParseIfExpr();
  case tok_for:
    return ParseForExpr();
  case tok_while:
    return ParseWhileExpr();
  case tok_var:
    return ParseVarExpr();
  case tok_tensor:
    return ParseTensorExpr();
  case tok_log:
    return ParseLogExpr();
  case tok_tab:
    getNextToken();
    return ParsePrimary();
  case tok_space:
    getNextToken();
    return ParsePrimary();
  }
}

/// unary
///   ::= primary
///   ::= '!' unary
static std::unique_ptr<ExprAST> ParseUnary(int tabcount=0) {
  //std::cout <<"Parse unary\n";
  while((CurTok==tok_tab)||(CurTok==tok_space))
    getNextToken();
  // If the current token is not an operator, it must be a primary expr.
  
  //std::cout << "Unary current token " << CurTok << "\n";
  if (!isascii(CurTok) || CurTok == '(' || CurTok == ',')
  {
    //std::cout << "Returning, non-ascii found.\n";
    return ParsePrimary(tabcount);
  }
  
  
  // If this is a unary operator, read it.
  int Opc = CurTok;
  
  //std::cout << "Unary expr\n";
  getNextToken();
  if (auto Operand = ParseUnary(tabcount))
    return std::make_unique<UnaryExprAST>(Opc, std::move(Operand));
  return nullptr;
}

/// binoprhs
///   ::= ('+' unary)*
static std::tuple<std::unique_ptr<ExprAST>, int> ParseBinOpRHS(int ExprPrec,
                                              std::unique_ptr<ExprAST> LHS,
                                              int tabcount=0) {
  
  // If this is a binop, find its precedence.
  int RhsTok = 0;
  int LhsTok = 0;

  int L_cuda = 0;
  int R_cuda = 0;

  std::string LName, RName;
  if (LHS->GetType()=="tensor")
    L_cuda = 1;

  while (true) {
    
    
    int TokPrec = get_tokenPrecedence();

    // If this is a binop that binds at least as tightly as the current binop,
    // consume it, otherwise we are done.
    

    if (TokPrec==BinopPrecedence[':'])
    {
      getNextToken();
      return std::make_tuple(std::move(LHS),L_cuda);
    }
    if (TokPrec < ExprPrec)
      return std::make_tuple(std::move(LHS),L_cuda);

    
    int BinOp = CurTok;

    if(CurTok==':')
    {
      getNextToken();
      return std::make_tuple(std::move(LHS),L_cuda);
    }

    if (CurTok==')')
      return std::make_tuple(std::move(LHS),L_cuda);

    
    getNextToken(); // eat binop
    if (CurTok==tok_number)
      RName = std::to_string(NumVal);
    else
      RName = IdentifierStr;

    // Get the Right Hand Side token

    int seen_tabs = 0;
    
    
    while(CurTok==tok_tab)
    {
      getNextToken();
      seen_tabs+=1;
    }

    

    
    RhsTok = CurTok;


    //std::cout << "Before RHS " << LhsTok << " " << BinOp << " " << CurTok << " " << seen_tabs << "/" << tabcount << " " << RName << " \n";

    
    if((BinOp==tok_space) && (!( CurTok==tok_identifier || CurTok==tok_number || CurTok==tok_self || CurTok==tok_class_attr || CurTok==tok_var || CurTok==tok_tensor)))
    {
      
      std::cout << "SPACE WITHOUT NUMBER OR VAR " << CurTok << " " << IdentifierStr << "\n";
      return std::make_tuple(std::move(LHS),L_cuda);
    }
    
    


    auto RHS = ParseUnary(); // Returns an identifier, number or expression result
    if (RHS->GetType()=="tensor")
      R_cuda=1;
    
    /*
    if(BinOp==tok_space)
    {
      std::cout << "FOUND SPACE HEREEE\n\n";
      return RHS;
    }
    */
    if (!RHS)
    {
      //std::cout << "RETURNING NULL Parse Unary \n";
      return std::make_tuple(nullptr,0);
    }

    
    if ((CurTok==tok_space)&&(seen_tabs<tabcount)&&(seen_tabs>0))
    {
      //std::cout << "DIMNISHING IJFNASEJHFBEAIUYSBFESABHFGIYBUEASFBEIAUSBFYEASUIBFYAEUSB\n";
      //LHS = std::move(RHS); //RETORNA O LADO DIREITO COMO O PRÓPRIO ELSE
      
      //LHS = ParseBinOpRHS(TokPrec + 1, std::move(LHS), tabcount);

      //RHS = ParseBinOpRHS(TokPrec + 1, std::move(RHS), tabcount);
      //LHS = std::make_unique<BinaryExprAST>(tok_space, std::move(LHS), std::move(RHS));
      
      //LHS = std::make_unique<BinaryExprAST>(BinOp, std::move(LHS), std::move(RHS));
      
      //return LHS;// RETORNA A VARIÁVEL COM ERRO DE INDEX
      return std::make_tuple(std::move(RHS),R_cuda);
    } else {


      // If BinOp binds less tightly with RHS than the operator after RHS, let
      // the pending operator take RHS as its LHS.
      int NextPrec = get_tokenPrecedence();
        
      // || ((seen_tabs<tabcount)&&(seen_tabs>0))
      if (TokPrec < NextPrec){
        //std::cout << NextPrec << " Next Prec\n";
        
        auto tuple = ParseBinOpRHS(TokPrec + 1, std::move(RHS), tabcount);
        RHS = std::move(std::get<0>(tuple));
        R_cuda = std::get<1>(tuple);

        //std::cout << "Error after RHS parse \n";
        if (!RHS)
        {
          //std::cout << "RETURNING NULL Recursive Bin Op \n";
          return std::make_tuple(nullptr,0);
        }
      }

      
      //std::cout << LhsTok << " " << BinOp << " " << RhsTok << "\n" << CurTok <<  " " << RName << "\n\n";
      
      //if(BinOp==64) // @

      

      if (L_cuda==1 && R_cuda==0)
      {
        LHS = std::make_unique<BinaryTensorScalarExprAST>(BinOp,
                                                      std::move(LHS), std::move(RHS));
        
      }
      else if (L_cuda==0 && R_cuda==1 )
      {
        std::cout << "Reverse LHS and RHS\n";
        //std::cout << "Bin op: " << BinOp << "\n";

        /*
        if (BinOp==tok_space)
        {
          std::cout << "Changing BinOp\n";
          BinOp = ':';
        }
        if (BinOp==':')
          BinOp = tok_space;
        */

        if (BinOp==47)
          return std::make_tuple(LogError("Divisão de escalar por tensor."),0);

        if (BinOp==45) // inversion of 1 - tensor
        {
          RHS = std::make_unique<BinaryTensorScalarExprAST>(42,
                                                    std::move(RHS),
                                                    std::move(std::make_unique<NumberExprAST>(-1.0f)));
                                                    //std::move(LHS)
                                                    
          LHS = std::make_unique<BinaryTensorScalarExprAST>(43,
                                                    std::move(RHS), std::move(LHS));
        } else {
          if (BinOp!=tok_space && BinOp!=':') // Avoid codegen reversing
            LHS = std::make_unique<BinaryTensorScalarExprAST>(BinOp,
                                                    std::move(RHS), std::move(LHS));
          else
            LHS = std::make_unique<BinaryTensorScalarExprAST>(BinOp,
                                                    std::move(LHS), std::move(RHS));
        }
          
        L_cuda=1;
        R_cuda=0;
      }
      else if (L_cuda==1 && R_cuda==1)
      { 
        LHS = std::make_unique<BinaryTensorTensorExprAST>(BinOp,
                                                      std::move(LHS), std::move(RHS));
        R_cuda=0;
      }
      else
        LHS = std::make_unique<BinaryExprAST>(BinOp, std::move(LHS), std::move(RHS));

      LhsTok = RhsTok;    
  }
}
}


/// expression
///   ::= unary binoprhs
///
static std::unique_ptr<ExprAST> ParseExpression(int tabcount) {
  //std::cout << "Parse Expression tabcount " << tabcount << "\n";
  //std::cout << "Parse Expression\n";
  
  auto LHS = ParseUnary(tabcount);
  if (!LHS)
    return nullptr;

  return std::get<0>(ParseBinOpRHS(0, std::move(LHS), tabcount));
}

/// prototype
///   ::= id '(' id* ')'
///   ::= binary LETTER number? (id, id)
///   ::= unary LETTER (id)
static std::unique_ptr<PrototypeAST> ParsePrototype(std::string ClassName="") {
  std::string FnName = ClassName;

  unsigned Kind = 0; // 0 = identifier, 1 = unary, 2 = binary.
  unsigned BinaryPrecedence = 30;

  switch (CurTok) {
  default:
    return LogErrorP("Esperado nome da função no protótipo");
  case tok_identifier:
    FnName += IdentifierStr;
    Kind = 0;
    getNextToken();
    break;
  case tok_unary:
    getNextToken();
    if (!isascii(CurTok))
      return LogErrorP("Esperado operador unário");
    FnName += "unary";
    FnName += (char)CurTok;
    Kind = 1;
    getNextToken();
    break;
  case tok_binary:
    getNextToken();
    if (!isascii(CurTok))
      return LogErrorP("Esperado operador binário");
    FnName += "binary";
    FnName += (char)CurTok;
    Kind = 2;
    getNextToken();

    // Read the precedence if present.
    if (CurTok == tok_number) {
      if (NumVal < 1 || NumVal > 100)
        return LogErrorP("Precedência inválida: deve ser entre 1 e 100");
      BinaryPrecedence = (unsigned)NumVal;
      getNextToken();
    }
    break;
  }

  if (CurTok != '(')
    return LogErrorP("Esperado '(' no protótipo");

  getNextToken();

  bool is_tensor=false;
  std::vector<std::string> ArgNames, Types;
  while (CurTok != ')')
  {
    Types.push_back(IdentifierStr);
    if (IdentifierStr=="t")
      is_tensor=true;
    if (IdentifierStr!="t" && IdentifierStr!="f" && IdentifierStr!="s")
      LogErrorP_to_comma("Tipo da variável no protótipo precisa ser t ou f.");
    else {
      getNextToken();

      ArgNames.push_back(IdentifierStr);
      if (is_tensor)
        tensorVars.push_back(IdentifierStr);
      
      getNextToken();
    }
    is_tensor=false;


    if (CurTok == ')')
        break;
      
    if (CurTok != ',')
    {
      std::cout << "comma Cur Tok " << IdentifierStr << "\n";
      return LogErrorP("Esperado ')' ou ',' na lista de argumentos do protótipo.");
    }
    getNextToken();
  }

  // success.
  getNextToken(); // eat ')'.

  // Verify right number of names for operator.
  if (Kind && ArgNames.size() != Kind)
    return LogErrorP("Número inválido de operandos para o operador");

  return std::make_unique<PrototypeAST>(FnName, ArgNames, Types, Kind != 0,
                                         BinaryPrecedence);
}


/// definition ::= 'def' prototype expression
static std::unique_ptr<FunctionAST> ParseDefinition(std::string ClassName="") {
  getNextToken(); // eat def.
  auto Proto = ParsePrototype(ClassName);
  if (!Proto)
    return nullptr;

  if (auto E = ParseExpression())
    return std::make_unique<FunctionAST>(std::move(Proto), std::move(E));
  return nullptr;
}

/// toplevelexpr ::= expression
static std::unique_ptr<FunctionAST> ParseTopLevelExpr() {
  //std::cout << "Top Level Expression\n";
  if (auto E = ParseExpression()) {
    // Make an anonymous proto.
    auto Proto = std::make_unique<PrototypeAST>("__anon_expr",
                                                std::vector<std::string>(),
                                                std::vector<std::string>());
    return std::make_unique<FunctionAST>(std::move(Proto), std::move(E));
  }
  return nullptr;
}

/// external ::= 'extern' prototype
static std::unique_ptr<PrototypeAST> ParseExtern() {
  getNextToken(); // eat extern.
  return ParsePrototype();
}







//===----------------------------------------------------------------------===//
// Code Generation
//===----------------------------------------------------------------------===//

static std::unique_ptr<KaleidoscopeJIT> TheJIT;
static std::unique_ptr<LLVMContext> TheContext;
static std::unique_ptr<LLVMContext> GlobalContext = std::make_unique<LLVMContext>();


static std::unique_ptr<IRBuilder<>> Builder;
static std::unique_ptr<Module> TheModule;


static std::map<std::string, std::unique_ptr<PrototypeAST>> FunctionProtos;
static ExitOnError ExitOnErr;


// Vars
static std::map<std::string, AllocaInst *> NamedValues;
static std::map<std::string, AllocaInst *> NamedStrs;
static std::map<std::string, Value *> NamedClassValues;
static std::map<std::string, float> StoredValues;


// Tensors
static std::map<std::string, float *> NamedTensors;
static std::map<std::string, std::vector<float>> NamedDims;

// Current Cuda Result
float *currentCudaResult;
std::vector<float> currentDims;

// Cuda Parallellism
constexpr int num_parallel_streams = 2;
hipStream_t parallel_streams[num_parallel_streams];
hipEvent_t parallel_events[num_parallel_streams];

// Optimizer
static std::map<std::string, float *> NamedParamGrads;


// File Handling
std::vector<char *> glob_str_files;




// Handle Class self with phantom argument
std::string FirstArg;





static std::unique_ptr<ExprAST> ParseClass() {
  getNextToken(); // eat class.

  if (CurTok != tok_identifier)
    return LogError("Esperado o nome da Classe.");
  std::string Name = IdentifierStr;

  Classes.push_back(Name);

  getNextToken();

  while(CurTok==tok_tab || CurTok==tok_space)
    getNextToken();
  

  if (CurTok!=tok_def)
    return LogError("Definição de uma Classe requer suas respectivas funções.");

  int i=0;
  while(CurTok==tok_def)
  {
    
    auto Func = ParseDefinition(Name);
    if (!Func)
      return nullptr;
      //return LogError("Falha no parsing da função da Classe.");
    if (!ends_with(Func->getProto().getName(),"__init__") && i==0)
      return LogError("Classe requer método init");
    
    //std::cout << "THE FUNCTION WAS CREATED AS: " << Func->getProto().getName() << "\n";

    FunctionProtos[Func->getProto().getName()] =
      std::make_unique<PrototypeAST>(Func->getProto());
    ExitOnErr(TheJIT->addAST(std::move(Func)));
    if(CurTok==';')
      getNextToken();
    while(CurTok==tok_space || CurTok==tok_tab)
      getNextToken();

    i+=1;
  }
  //if (auto E = ParseExpression())
  //  return std::make_unique<ClassAST>(std::move(Proto), std::move(E));
  return nullptr;
}



int dimsProd(std::vector<float> dims)
{
  float aux=1;
  for (int i = 0; i < dims.size(); i++)
    aux = aux*dims[i];
  return (int)aux;
}



std::vector<float> format_LinearLayer_Dims(std::vector<float> dims)
{
  std::vector<float> new_dims;
  int aux=1;
  for (int i = 0; i < dims.size()-1; i++)
    aux *= dims[i];
  new_dims.push_back(aux);
  new_dims.push_back(dims[dims.size()-1]);
  return new_dims;
}


void PrintDims(std::vector<float> dims)
{
  std::cout << "dims: [";
  for (int i=0; i<dims.size();i++)
  {
    std::cout << (int)dims[i];
    if (i==dims.size()-1)
      std::cout << "]";
    else
      std::cout << ", ";
  }
  std::cout  << "\n";
}

int resultingDimsProdOnMult(std::vector<float> Ldims, std::vector<float> Rdims)
{
  float aux=1;
  for (int i = 0; i < Ldims.size()-1; i++)
    aux = aux * Ldims[i];
  aux = aux * Rdims[0];
  return (int)aux;
}


std::vector<float> newDimsOnMult(std::vector<float> Ldims, std::vector<float> Rdims)
{
  std::vector<float> new_dims;
  if (Ldims[Ldims.size()-1]!=Rdims[Rdims.size()-1])
  {
    LogError("A última dimensão dos tensors multiplicados precisa ser igual.");
    std::cout << "Dim LHS: ";
    PrintDims(Ldims);
    std::cout << "Dim RHS: ";
    PrintDims(Rdims);
    return new_dims; 
  }
  for (int i = 0; i < Ldims.size()-1; i++)
    new_dims.push_back(Ldims[i]);
  new_dims.push_back(Rdims[0]);
  
  return new_dims;
}

extern "C" float PrintStr(char* value){
  
  std::cout << "Str: " << value << "\n";
  return 0;
}

extern "C" char * shuffle_str(char *string_list)
{

  std::ostringstream oss;

  std::vector<std::string> splitted = split(string_list, "|||");


  std::random_shuffle(splitted.begin(), splitted.end());

  for (int i=0; i<splitted.size(); i++)
  {
    if (i>0)
      oss << "|||";
    oss << splitted[i];
  }

  std::string result = oss.str();

  char * cstr = new char [result.length()+1];
  std::strcpy (cstr, result.c_str());
    
  return cstr;
}





extern "C" float PrintTensor(char* tensorName){
  std::cout << "Called print tensor\n";
  

  std::vector<float> dims = NamedDims[tensorName];
  int arr_size = dimsProd(dims);


  float *tensor_cuda = NamedTensors[tensorName];
  float *tensor = new float[arr_size];
  //std::cout << "Printing Tensor " << arr_size << "\n";
  
  hipDeviceSynchronize();
  cudaCheck(hipMemcpy(tensor, tensor_cuda, arr_size*sizeof(float), hipMemcpyDeviceToHost));


  std::cout << "\nTensor \033[95m" << tensorName << "\033[0m:\n";
  PrintDims(dims);
  std::cout << "\n";
  std::vector<float> ends;


  for (int i = 0; i < dims.size(); i++) {
    int prod=1;
    for (int j = 0; j <= i; j++)
      prod = prod*dims[dims.size()-1-j];
    ends.push_back(prod);
  }


  int line = 1;
  bool line_changed = true;
  for (int i = 0; i < arr_size; i++) {

    int to_prints = 0;

    for (int e = 0; e < ends.size(); e++)
    {
      if (fmod((arr_size-i),(int)ends[e]) == 0.0f)
        to_prints+=1;
    }

    if(to_prints>0)
    {
      for (int j=0; j<(dims.size()-to_prints); j++)
        std::cout << " ";
        
      for (int j=0; j<to_prints; j++)
        std::cout << "[";
    }
    

    //std::cout << "LAST SIZE " << dims[dims.size()-1] << " Mod: " << fmod(i, 1+dims[dims.size()-1]) << "\n";
    int precision;
    if (tensor[i]>=0)
      precision=4;
    else
      precision=3;
    std::cout << std::fixed  << std::setprecision(precision) << tensor[i];


    for (int e = 0; e < ends.size(); e++)
      if (fmod((i+1),(int)ends[e]) == 0.0f)
        std::cout << "]";
    

    if (i!=(arr_size-1))
    {
      if (fmod(i+1, dims[dims.size()-1]) == 0.0f)
      {
        line+=1;
        line_changed=true;
        std::cout << "\n";
      }
      else
        std::cout << "  ";
    }

    if(fmod(i+1, ends[1]) == 0.0f)
      std::cout << "\n";


  }
  std::cout << "\n";
  PrintDims(dims);
  std::cout << "\n";

  return 0;
}

float PrintTensorF(float *cuda_tensor, int d1, int d2){
  

  std::vector<float> dims;
  dims.push_back(d1);
  dims.push_back(d2);

  int arr_size = dimsProd(dims);


  float *tensor = new float[arr_size];
  //std::cout << "Printing Tensor " << arr_size << "\n";
  
  hipDeviceSynchronize();
  cudaCheck(hipMemcpy(tensor, cuda_tensor, arr_size*sizeof(float), hipMemcpyDeviceToHost));


  
  PrintDims(dims);
  std::cout << "\n";
  std::vector<float> ends;


  for (int i = 0; i < dims.size(); i++) {
    int prod=1;
    for (int j = 0; j <= i; j++)
      prod = prod*dims[dims.size()-1-j];
    ends.push_back(prod);
  }


  int line = 1;
  bool line_changed = true;
  for (int i = 0; i < arr_size; i++) {

    int to_prints = 0;

    for (int e = 0; e < ends.size(); e++)
    {
      if (fmod((arr_size-i),(int)ends[e]) == 0.0f)
        to_prints+=1;
    }

    if(to_prints>0)
    {
      for (int j=0; j<(dims.size()-to_prints); j++)
        std::cout << " ";
        
      for (int j=0; j<to_prints; j++)
        std::cout << "[";
    }
    

    //std::cout << "LAST SIZE " << dims[dims.size()-1] << " Mod: " << fmod(i, 1+dims[dims.size()-1]) << "\n";
    int precision;
    if (tensor[i]>=0)
      precision=4;
    else
      precision=3;
    std::cout << std::fixed  << std::setprecision(precision) << tensor[i];


    for (int e = 0; e < ends.size(); e++)
      if (fmod((i+1),(int)ends[e]) == 0.0f)
        std::cout << "]";
    

    if (i!=(arr_size-1))
    {
      if (fmod(i+1, dims[dims.size()-1]) == 0.0f)
      {
        line+=1;
        line_changed=true;
        std::cout << "\n";
      }
      else
        std::cout << "  ";
    }

    if(fmod(i+1, ends[1]) == 0.0f)
      std::cout << "\n";


  }
  std::cout << "\n";

  return 0;
}




Function *getFunction(std::string Name) {
  // First, see if the function has already been added to the current module.
  if (auto *F = TheModule->getFunction(Name))
    return F;

  // If not, check whether we can codegen the declaration from some existing
  // prototype.
  auto FI = FunctionProtos.find(Name);
  if (FI != FunctionProtos.end())
    return FI->second->codegen();

  // If no existing prototype exists, return null.
  return nullptr;
}

/// CreateEntryBlockAlloca - Create an alloca instruction in the entry block of
/// the function.  This is used for mutable variables etc.
static AllocaInst *CreateEntryBlockAlloca(Function *TheFunction,
                                          StringRef VarName) {
  IRBuilder<> TmpB(&TheFunction->getEntryBlock(),
                   TheFunction->getEntryBlock().begin());
  return TmpB.CreateAlloca(Type::getFloatTy(*TheContext), nullptr, VarName);
}

Value *NumberExprAST::codegen() {
  //std::cout << "Codegen for Number: " << Val << "\n";
  return ConstantFP::get(*TheContext, APFloat(Val));
}

Value *StringExprAST::codegen() {
  SetName(Val);
  return Builder->CreateGlobalString(Val);
}


//===----------------------------------------------------------------------===//
// Dataset
//===----------------------------------------------------------------------===//



extern "C" char * _glob_b_(char *pattern) {
    // TODO: make var of type string vector to hold this result.

    glob_t glob_result;
    //std::vector<char *> glob_str_files;

    std::ostringstream  oss;
    oss << "";

    if (glob(pattern, GLOB_TILDE, NULL, &glob_result) == 0) {
        for (size_t i = 0; i < glob_result.gl_pathc; ++i) {
            //result = result + "|||" + glob_result.gl_pathv[i];
            if (i>0)
              oss << "|||";
            oss << glob_result.gl_pathv[i];

            glob_str_files.push_back(strdup(glob_result.gl_pathv[i]));
        }
        globfree(&glob_result);
    }

    int i=0;

    if (glob_str_files.size()<1)
      LogErrorS("Glob falhou ao encontrar arquivos.");
    
    
    std::string result = oss.str();
    //std::cout << result << "\n";

    char * cstr = new char [result.length()+1];
    std::strcpy (cstr, result.c_str());
    
    return cstr;
}



float *current_data;
float *current_labels;
extern "C" float Datasetinit_dataset(float batch_size)
{
  std::cout << "Executing init dataset\n";
  std::cout << "Fist arg: " << FirstArg << "\n";
  std::random_shuffle(glob_str_files.begin(), glob_str_files.end());
  load_img(glob_str_files[0]);

  
  int dims_prod = dimsProd(current_data_attr_dims);

  current_data = new float[batch_size*dims_prod];
  current_labels = new float[batch_size];

  // Using CUDA CPU pinned memory for faster PCI Express transfers to GPU
  // See: https://developer.nvidia.com/blog/how-optimize-data-transfers-cuda-cc/
  cudaCheck(hipHostMalloc(&current_data, batch_size*dims_prod*sizeof(float)));
  cudaCheck(hipHostMalloc(&current_labels, batch_size*sizeof(float)));
  return 0;
}

extern "C" float Datasetgetitem_1(float idx, char *tensor_name)
{
  //std::cout << "EXECUTING GETITEM\n";
  return 12321;
}

extern "C" float* Datasetgetitem_2(float idx);




int yield_pointer = 0;
extern "C" float Datasetyield(float batch_size, char * x_name, ...)
{
  //std::cout << "Executing yield\n";
  //std::cout << "Fist arg: " << FirstArg << "\n";

  std::vector<char *> tensor_names;
  tensor_names.push_back(x_name);


  //std::cout << "X name: " << x_name << "\n";
  va_list args;
  va_start(args, x_name);

  for (int i=0; i<10; i++)
  {
    //std::cout << "Vararg for: " << i << "\n";
    char * name = va_arg(args, char *);
    //std::cout << "Name: " << name << "\n\n";
    
    if (starts_with(name, "-2"))
      break;

    tensor_names.push_back(name);
  }
  va_end(args);

  char * y_name = tensor_names[1];

  //std::cout << "Finished vararg\n";

  
  int b=0;

  int dims_prod, y_dims_prod;

  float *cur_float_img, *y_aux;

  while (b < batch_size)
  {

    std::cout << "\n";
    //cur_float_img = load_img(glob_str_files[yield_pointer]);

    std::string preprocessing = "preprocess_";
    preprocessing += (const char *)x_name;
    std::cout << "Preprocessing: " << preprocessing << "\n";

    cur_float_img = preprocessings[preprocessing](glob_str_files[yield_pointer]);
    //dims_prod = dimsProd(current_data_attr_dims);
    dims_prod = 28*28;
    for (int j = 0; j < dims_prod; ++j)
      current_data[b * dims_prod + j] = cur_float_img[j];
    
    
    //std::vector<std::string> splitted = split_str(glob_str_files[yield_pointer],'/');
    preprocessing = "preprocess_";
    preprocessing += (const char *)y_name;
    std::cout << "Preprocessing: " << preprocessing << "\n";
    y_aux = preprocessings[preprocessing](glob_str_files[yield_pointer]);
    y_dims_prod=1;
    for (int j = 0; j < y_dims_prod; ++j)
      current_labels[b * y_dims_prod + j] = y_aux[j];

    

    
    b+=1;
    
    yield_pointer+=1;
    // Drop last batch and reset idx
    if(yield_pointer>(glob_str_files.size()-batch_size-batch_size))
    { 
      std::random_shuffle(glob_str_files.begin(), glob_str_files.end());
      yield_pointer=0;
    }
  }

  float *x, *y;
  hipMalloc(&x, batch_size*dims_prod*sizeof(float));
  hipMalloc(&y, batch_size*sizeof(float));

  // todo - inputs is copied on default stream so this synchronises CPU/GPU for now
  /*
  hipMemcpyAsync(x, current_data, batch_size*dims_prod*sizeof(float), hipMemcpyHostToDevice,0);
  // memcpy targets in parallel then wait for them before fused_classifier
  hipMemcpyAsync(y, current_labels, batch_size*sizeof(float), hipMemcpyHostToDevice, parallel_streams[0]);
  hipEventRecord(parallel_events[0], parallel_streams[0]);
  */
  
  hipMemcpy(x, current_data, batch_size*dims_prod*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y, current_labels, batch_size*sizeof(float), hipMemcpyHostToDevice);

  std::vector<float> dims_x, dims_y;
  dims_x.push_back(batch_size);
  dims_y.push_back(batch_size);
  for(int i=0; i<current_data_attr_dims.size(); i++)
    dims_x.push_back(current_data_attr_dims[i]);

  hipFree(NamedTensors[x_name]);
  hipFree(NamedTensors[y_name]);

  NamedTensors[x_name] = x;
  NamedDims[x_name] = dims_x;
  NamedTensors[y_name] = y;
  NamedDims[y_name] = dims_y;

  return 0;
}



extern "C" float load_preprocess_img(char *tensor_name, char *img_name)
{
  float *img;
  img = load_img(img_name); 
    
  
  int dims_prod = 28*28;


  current_data = new float[dims_prod];
  cudaCheck(hipHostMalloc(&current_data, dims_prod*sizeof(float)));


  for (int j = 0; j < dims_prod; ++j)
    current_data[j] = img[j];


  float *x;
  hipMalloc(&x, dims_prod*sizeof(float));
  cudaCheck(hipMemcpy(x, current_data, dims_prod*sizeof(float), hipMemcpyHostToDevice));

  NamedTensors[tensor_name] = x;
  NamedDims[tensor_name] = current_data_attr_dims;

  return 0;
}


//===----------------------------------------------------------------------===//
// Tensor Functionalities
//===----------------------------------------------------------------------===//


extern "C" float view(float first_dim, ...)
{
  
  std::string tensor_name = FirstArg;
  std::vector<float> new_dims, current_dims;
  current_dims = NamedDims[tensor_name];
  
  va_list args;
  va_start(args, first_dim);
  new_dims.push_back(first_dim);

  for (int i=0; i<10; i++)
  {
    if (i==9)
    {
      LogErrorS("Um tensor com 10 dimensões???");
      return 0;
    }

    float dim = va_arg(args, float);
    if (dim==-2)
      break;
    new_dims.push_back(dim);
  }
  va_end(args);


  int current_dims_prod = dimsProd(current_dims);
  int new_dims_prod = dimsProd(new_dims);

  if (current_dims_prod != new_dims_prod)
  {
    LogErrorS("view das dimensões não é compatível");
    PrintDims(current_dims);
    std::cout << "Produto das dimensões atuais: " << current_dims_prod  << "\n";
    PrintDims(new_dims);
    std::cout << "Produto das novas dimensões: " << new_dims_prod  << "\n";
    return 0;
  }

  NamedDims[tensor_name] = new_dims;
  used_cuda=1;

  return  0;
}



//===----------------------------------------------------------------------===//
// Tensor -- Scalar   Operations
//===----------------------------------------------------------------------===//


__global__ void vec_mult(float a, float* x, float* y) {
  y[threadIdx.x] = x[threadIdx.x] * a;
}
__global__ void vec_div(float a, float* x, float* y) {
  y[threadIdx.x] = x[threadIdx.x] / a;
}
__global__ void vec_add(float a, float* x, float* y) {
  y[threadIdx.x] = x[threadIdx.x] + a;
}
__global__ void vec_sub(float a, float* x, float* y) {
  y[threadIdx.x] = x[threadIdx.x] - a;
}
__global__ void vec_log(float* x, float* y) {
  y[threadIdx.x] = logf(x[threadIdx.x]);
}




extern "C" float CudaScalarMult(char *tensorName, float R, int _used_cuda) {
  
  float * device_x;
  

  if (_used_cuda==1)
    device_x = currentCudaResult;
  else
  {
    device_x = NamedTensors[tensorName];
    currentDims = NamedDims[tensorName];
  }

  int kDataLen = dimsProd(currentDims);


  float* device_y;
  cudaCheck(hipMalloc(&device_y, kDataLen * sizeof(float)));
  


  int grid_size = kDataLen;
  int block_size = 32;
  size_t shared_mem_size = 2 * block_size / 32 * sizeof(float);
  vec_mult<<<grid_size, block_size, shared_mem_size>>>(R, device_x, device_y);

  currentCudaResult = device_y;

  return 0;
}


extern "C" float CudaScalarDiv(char *tensorName, float R, int _used_cuda) {
  
  float * device_x;
  

  if (_used_cuda==1)
    device_x = currentCudaResult;
  else
  {
    device_x = NamedTensors[tensorName];
    currentDims = NamedDims[tensorName];
  }

  int kDataLen = dimsProd(currentDims);


  float* device_y;
  cudaCheck(hipMalloc(&device_y, kDataLen * sizeof(float)));


  // Launch the kernel.
  vec_div<<<1, kDataLen>>>(R, device_x, device_y);

  currentCudaResult = device_y;

  return 0;
}

extern "C" float CudaScalarAdd(char *tensorName, float R, int _used_cuda) {
  float * device_x;

  

  if (_used_cuda==1)
    device_x = currentCudaResult;
  else
  {
    device_x = NamedTensors[tensorName];
    currentDims = NamedDims[tensorName];
  }

  int kDataLen = dimsProd(currentDims);


  float* device_y;
  cudaCheck(hipMalloc(&device_y, kDataLen * sizeof(float)));
  
  // Launch the kernel.
  vec_add<<<1, kDataLen>>>(R, device_x, device_y);
  
  currentCudaResult = device_y;


  return 0;
}

extern "C" float CudaScalarSub(char *tensorName, float R, int _used_cuda) {
  
  float * device_x;

  

  if (_used_cuda==1)
    device_x = currentCudaResult;
  else
  {
    device_x = NamedTensors[tensorName];
    currentDims = NamedDims[tensorName];
  }

  int kDataLen = dimsProd(currentDims);


  float* device_y;
  cudaCheck(hipMalloc(&device_y, kDataLen * sizeof(float)));


  // Launch the kernel.
  vec_sub<<<1, kDataLen>>>(R, device_x, device_y);

  currentCudaResult = device_y;

  return 0;
}


extern "C" float logE(char *tensorName, int _used_cuda) {
  
  float * device_x;

  if (_used_cuda==1)
    device_x = currentCudaResult;
  else
  {
    device_x = NamedTensors[tensorName];
    currentDims = NamedDims[tensorName];
  }

  int kDataLen = dimsProd(currentDims);


  float* device_y;
  cudaCheck(hipMalloc(&device_y, kDataLen * sizeof(float)));


  // Launch the kernel.
  vec_log<<<1, kDataLen>>>(device_x, device_y);

  currentCudaResult = device_y;

  return 0;
}


extern "C" float FirstArgOnDemand(char *pre_dot, int nested_function)
{
  if (nested_function)
    FirstArg = FirstArg+pre_dot;
  else
    FirstArg = pre_dot;
  return 0;
}

extern "C" float DimnishFirstArgOnDemand(char *pre_dot, int nested_function)
{
  if (nested_function)
    if(ends_with(FirstArg, pre_dot))
    {
      size_t pos = FirstArg.find(pre_dot);

      FirstArg.erase(pos, std::strlen(pre_dot));
    }
    
  
  return 0;
}


extern "C" char * ConcatStr(char *lc, char *rc)
{
  std::string l = lc;
  std::string r = rc;

  std::string result_str = l + r;
  char* result_cstr = new char[result_str.length() + 1]; // +1 for null terminator
  std::strcpy(result_cstr, result_str.c_str());
  
  return result_cstr;
}

extern "C" char * ConcatFirstArgToVarName(char *var_name)
{
  //std::cout << "\nConcatFirstArgToVarName: " << FirstArg << "\nVar name: " << var_name <<"\n\n";
  
  std::string l = var_name;

  std::string result_str = FirstArg + l;
  char* result_cstr = new char[result_str.length() + 1]; // +1 for null terminator
  std::strcpy(result_cstr, result_str.c_str());


  return result_cstr;
}


extern "C" float StoreOnDemand(char *object_var_name, float value){
  
//  std::cout << "StoreOnDemand: " << FirstArg << "." << object_var_name << " " << value << "\n";

  NamedClassValues[FirstArg + object_var_name] = ConstantFP::get(*GlobalContext, APFloat(value));
  return 0;
}


extern "C" float StoreStrOnDemand(char *object_var_name, char * value){
  

  
  //NamedClassValues[FirstArg + object_var_name] = ConstantFP::get(*GlobalContext, APFloat(value));
  NamedClassValues[FirstArg + object_var_name] = Builder->CreateGlobalString(value);
  return 0;
}


extern "C" float LoadOnDemand(char *object_var_name) {
  //std::cout << "LoadOnDemand var to load: " << object_var_name << "\n";
    
  Value * class_val = NamedClassValues[object_var_name];

  if (class_val) 
    return (float) cast<ConstantFP>(class_val)->getValueAPF().convertToFloat();
  else
    return 0;
}


bool seen_var_attr = false;
Value *VariableExprAST::codegen() {
  // Look this variable up in the function.

  //std::cout << "Now Loading Var "<< Name <<" to Context" << "  \n";


  Function *TheFunction = Builder->GetInsertBlock()->getParent();
  std::string functionName = TheFunction->getName().str();
  
  

  Value *var_name, *object_name, *object_var_name;
  Value * ret = ConstantFP::get(*TheContext, APFloat(0.0f));
  var_name = Builder->CreateGlobalString(Name);
  
  
      
  std::string pre_dot = GetSelf();
  if (pre_dot!="false")
  {
    // Gets from FirstArg if it is self
    if (pre_dot=="true")
      var_name = Builder->CreateCall(TheModule->getFunction("ConcatFirstArgToVarName"),
                                                      {var_name});
    // Gets from pre_dot if it is a class attribute
    else {
      object_name = Builder->CreateGlobalString(pre_dot);
      var_name = Builder->CreateGlobalString(Name);

      var_name = Builder->CreateCall(TheModule->getFunction("ConcatStr"),
                                                      {object_name, var_name});
    }
      for (const auto &entry : NamedClassValues)
        if (ends_with(entry.first, Name))
          return Builder->CreateCall(TheModule->getFunction("LoadOnDemand"),
                                                      {var_name});        
  }

  if (NamedValues.count(Name)>0) 
  {

    Value *V = NamedValues[Name];
    
    

    return Builder->CreateLoad(Type::getFloatTy(*GlobalContext), V, Name.c_str());

  } else if (NamedTensors.count(Name)>0) {
    //std::cout << "Load Tensor " << Name << " Codegen.\n";
  

    if (!seen_var_attr)
    {
      Builder->CreateCall(TheModule->getFunction("PrintTensor"), {var_name});
    }
    
    return ret;
  } else if (NamedStrs.count(Name)>0) {
    for (const auto &entry : NamedTensors)
      if (ends_with(entry.first, Name))
        return ret;

    Value *V = NamedStrs[Name];
    
    V = Builder->CreateLoad(PointerType::get(Type::getInt8Ty(*TheContext), 0), V, Name.c_str());
    if (!seen_var_attr)
    {
      //std::cout << "Print str call for: " << Name << "\n";
      Builder->CreateCall(TheModule->getFunction("PrintStr"),
                      {V});
    }

    //std::cout << "RETURNING STRING: " << Name << "\n";
    //std::cout << "NamedStrs count:" << NamedStrs.count(Name) << "\n";
    return V;
  }
}



extern "C" float toStoredValues(float Val, char * name_to_store)
{
  StoredValues[name_to_store] = Val;
  return 0;
}


extern "C" float temporaryCudaResult_Attr(char *tensorName)
{
  //std::cout << "Attributing to tensor: " << tensorName << "\n";
  cudaCheck(hipFree(NamedTensors[tensorName]));

  float * tensor = new float[4];
  hipMemcpy(tensor, currentCudaResult, 4, hipMemcpyDeviceToHost);

  NamedTensors[tensorName] = currentCudaResult;
  NamedDims[tensorName] = currentDims;

  return 0;
}





Value *BinaryTensorScalarExprAST::codegen() {

  Value *tensorName = Builder->CreateGlobalString(LHS->GetName());

  std::string pre_dot = LHS->GetSelf();
  if (pre_dot=="true")
    tensorName = Builder->CreateCall(TheModule->getFunction("ConcatFirstArgToVarName"),
                                                      {tensorName});
    // Gets from pre_dot if it is a class attribute
  else if (pre_dot!="false") {
    Value * object_name = Builder->CreateGlobalString(pre_dot);

    tensorName = Builder->CreateCall(TheModule->getFunction("ConcatStr"),
                                                      {object_name, tensorName});
  }



  // Special case '=' because we don't want to emit the LHS as an expression.
  if (Op == '=') {
    seen_var_attr=true;
    // Assignment requires the LHS to be an identifier.
    // This assume we're building without RTTI because LLVM builds that way by
    // default.  If you build LLVM with RTTI this can be changed to a
    // dynamic_cast for automatic error checking.
    VariableExprAST *LHSE = static_cast<VariableExprAST *>(LHS.get());
    if (!LHSE)
      return LogErrorV("Destino do '=' deve ser uma variável.");
    // Codegen the RHS.
    
    Value *Val = RHS->codegen();
    if (!Val)
      return nullptr;

    
    
    std::cout << "1 0 attr\n";
    /*
    float *Variable = NamedTensors[LHSE->getName()];
    if (!Variable)
      return LogErrorV("O nome do tensor/variável é desconhecido.");
    */
    
    if (used_cuda)
    {
      Function *temporaryCudaResult_AttrFn = TheModule->getFunction("temporaryCudaResult_Attr");
      Builder->CreateCall(temporaryCudaResult_AttrFn, {tensorName});        
      
      used_cuda=0;
    }
      
    
    seen_var_attr=false;
    return Val;
  }


  Value *L = LHS->codegen();
  Value *R = RHS->codegen();
  
  if (!L || !R)
    return nullptr;


  Function *CudaFn;

  /*
  std::cout << "\nTensorScalar, LHS is self: " << LHS->GetSelf() << "\n";
  Function *TheFunction = Builder->GetInsertBlock()->getParent();
  std::string functionName = TheFunction->getName().str();
  std::cout << "Fname: " << functionName << "\n\n";
  */
  

  Value *used_cuda_aux = ConstantInt::get(Type::getInt32Ty(*TheContext), used_cuda);
  used_cuda = 1;

  switch (Op)
  {
  case '*':
    CudaFn = TheModule->getFunction("CudaScalarMult");
    return Builder->CreateCall(CudaFn, {tensorName, R, used_cuda_aux}, "cudascalarmult");
  case '/':
    CudaFn = TheModule->getFunction("CudaScalarDiv");
    return Builder->CreateCall(CudaFn, {tensorName, R, used_cuda_aux}, "cudascalardiv");
  case '+':
    CudaFn = TheModule->getFunction("CudaScalarAdd");
    return Builder->CreateCall(CudaFn, {tensorName, R, used_cuda_aux}, "cudascalaradd");
  case '-':
    CudaFn = TheModule->getFunction("CudaScalarSub");
    return Builder->CreateCall(CudaFn, {tensorName, R, used_cuda_aux}, "cudascalarsub");
  case ':':
    return L;
  case tok_space:
    return R;
  default:
    break;
  }
  

  // If it wasn't a builtin binary operator, it must be a user defined one. Emit
  // a call to it.
  Function *F = getFunction(std::string("binary") + Op);
  assert(F && "Operator not found.");

  Value *Ops[] = {L, R};
  return Builder->CreateCall(F, Ops, "binop");
}




/*
void matmul_forward2(float* out,
                     const float* inp, const float* weight, const float* bias,
                     int B, int T, int C, int OC,
                     const int sqrt_block_size) {*/
void matmul_forward2(float* out,
                     const float* inp, const float* weight,
                     int B, int C, int OC) {
                     //const int sqrt_block_size
                     
    
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    //std::cout << "matmul_forward. B: " << B << " C: " << C << " OC: " << OC << "\n";
    
    cublasCheck(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, OC, B, C, &alpha, weight, C, inp, C, &beta, out, OC));


    /* //bias
    if (bias != NULL) {
        int block_size = sqrt_block_size * sqrt_block_size;
        int grid_size = ceil_div(OC * B * T, block_size);
        add_bias<<<grid_size, block_size>>>(out, bias, B, T, OC);
        cudaCheck(hipGetLastError());
    }
    */
}

void matmul_backward(float *inp,  float *weight,
                     int B, int C, int OC,
                     float *dinp, float *dw,
                     float *dout)
{
  //std::cout << "matmul_backward. B: " << B << " C: " << C << " OC: " << OC << "\n";

  float one = 1.0f, zero = 0.0f;
  // backward to input
  cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, C, B, OC, &one,
                             weight, CUBLAS_LOWP, C, dout, CUBLAS_LOWP, OC, &zero,
                             dinp, CUBLAS_LOWP, C, cublas_compute, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  // backward to weight, uses += in the backward pass (accumulate the gradient) by setting alpha=one
  cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, C, OC, B, &one,
                             inp, CUBLAS_LOWP, C, dout, CUBLAS_LOWP, OC, &one,
                             dw, CUBLAS_LOWP, C, cublas_compute, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

  cudaCheck(hipGetLastError());
}



using backward_tuple = std::tuple<int, int, int, float *, float *, float *, std::string, std::string>;
std::vector<backward_tuple> todo_backwards;



extern "C" float CudaMult(char *LtensorName, char *RtensorName, int _used_cuda, int is_forward_func) {
  
  float * device_x;
  float * device_w;
  

  /*
  if (_used_cuda==1)
    device_x = currentCudaResult;
  else
  {
    device_x = NamedTensors[LtensorName];
    currentDims = NamedDims[LtensorName];
  }
  */
  
  device_x = NamedTensors[LtensorName];
  currentDims = NamedDims[LtensorName];
  

  device_w = NamedTensors[RtensorName];

  std::vector<float> Rdims = NamedDims[RtensorName];
  


  std::vector<float> linear_layer_dims = format_LinearLayer_Dims(currentDims);
  int input_dims_prod = dimsProd(linear_layer_dims);
  //int resultingDimsProd = (int)linear_layer_dims[0]*Rdims[0];
  int resultingDimsProd = resultingDimsProdOnMult(linear_layer_dims, Rdims);


  float* device_y;
  cudaCheck(hipMalloc(&device_y, resultingDimsProd * sizeof(float)));

  if (currentDims.size()<2)
    LogErrorS("Tensor de entrada da multiplicação de tensors precisa ter ao menos 2 dimensões.");



  
  //if(currentDims[1]==784)
  //PrintTensorF(device_x, currentDims[0], currentDims[1]);
  //PrintTensorF(device_w, Rdims[0], Rdims[1]);

  matmul_forward2(device_y, device_x, device_w,
                  linear_layer_dims[0], linear_layer_dims[1],
                  Rdims[0]);
                  //64
                  //);


  currentCudaResult = device_y;
  //std::cout << "L tensor: " << LtensorName << " R tensor: " << RtensorName << "\n";
  currentDims = newDimsOnMult(currentDims, Rdims);

  
  if (is_forward_func)
  {
    float *inp, *out;


    //oom
    cudaCheck(hipMalloc(&inp, input_dims_prod * sizeof(float)));
    cudaCheck(hipMalloc(&out, resultingDimsProd * sizeof(float)));
    hipMemcpy(inp, device_x, input_dims_prod * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(out, device_y, resultingDimsProd * sizeof(float), hipMemcpyDeviceToDevice);

    todo_backwards.push_back(std::make_tuple(linear_layer_dims[0], linear_layer_dims[1],
                                           Rdims[0], inp, device_w, out,
                                           "matmul", RtensorName));
  }
  return 0;
}

int num_classes=5;

float eps = 1e-8;

// warp-level reduction for finding the maximum value
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

// warp-level reduction for summing values
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}


// Parallelizes over B, C
__global__ void onehot_kernel(const float* tensor,
                           float* probs,
                           int B, int C) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //int i = threadIdx.x;
    
    
    if (i < B * C) {
        int b = i / (C);
        int v = i % C;

        float* probs_b = probs + b * C;

        int ix = tensor[b];

        //float p = probs_b[v];

        float indicator = (v==ix) ? 1.0f : 0.0f;

        probs_b[v] = indicator;
        
    }
}

extern "C" float onehot(float num_classes)
{
  std::string tensor_name = FirstArg;

  float * tensor = NamedTensors[tensor_name];
  std::vector<float> dims = NamedDims[tensor_name];
  
  int B = dimsProd(dims);
  int C = (int)num_classes;

  float *probs_cpu, *probs;
  probs_cpu = new float[B*C];

  hipMalloc(&probs, B*C*sizeof(float));
  //hipMemcpy(probs, probs_cpu, B*C*sizeof(float), hipMemcpyHostToDevice);


  int grid_size = B;
  int block_size = 32;
  size_t shared_mem_size = 2 * block_size / 32 * sizeof(float);


  
  onehot_kernel<<<grid_size, block_size, shared_mem_size>>>(tensor, probs, B, C);
  //grid_size = ceil_div(B*C, block_size);
  //onehot_kernel<<<grid_size, block_size>>>(tensor, probs, B, C);

  dims.push_back(C);
  NamedDims[tensor_name] = dims;

  hipFree(NamedTensors[tensor_name]);
  NamedTensors[tensor_name] = probs;

  used_cuda=1;

  return 0;
}


//TODO: mean, sum, max at CUDA
extern "C" float mean() 
{
  std::string tensor_name = FirstArg;

  float * tensor = NamedTensors[tensor_name];
  std::vector<float> dims = NamedDims[tensor_name];
  
  int B = dimsProd(dims);


  float *meaned = new float[B];

  hipMemcpy(meaned, tensor, B*sizeof(float), hipMemcpyDeviceToHost);
  
  float tensor_mean=0;
  for(int i=0; i<B; i++)
    tensor_mean += meaned[i];
  tensor_mean = tensor_mean/B;

  std::cout << "Mean: " << tensor_mean << "\n";

  return 0;
}

extern "C" float sum()
{
  std::string tensor_name = FirstArg;

  float * tensor = NamedTensors[tensor_name];
  std::vector<float> dims = NamedDims[tensor_name];
  
  int B = dimsProd(dims);


  float *summed = new float[B];

  hipMemcpy(summed, tensor, B*sizeof(float), hipMemcpyDeviceToHost);
  
  float tensor_sum=0;
  for(int i=0; i<B; i++)
    tensor_sum += summed[i];
  tensor_sum = tensor_sum;

  std::cout << "Sum: " << tensor_sum << "\n";

  return 0;
}

extern "C" float max()
{
  std::string tensor_name = FirstArg;

  float * tensor = NamedTensors[tensor_name];
  std::vector<float> dims = NamedDims[tensor_name];
  
  int B = dimsProd(dims);



  float max=-999;
  float *summed = new float[B];

  hipMemcpy(summed, tensor, B*sizeof(float), hipMemcpyDeviceToHost);
  
  float tensor_sum=0;
  for(int i=0; i<B; i++)
  {
    if(summed[i]>max)
      max = summed[i];
  }

  std::cout << "Max: " << max << "\n";

  return 0;
}


__global__ void softmax_forward_kernel4(const float* inp, float* out, int N, int C) {
    // out is (N, C) just like inp. Each row of inp will get softmaxed.
    // same as kernel3, but can handle any block size (multiple of 32)
    // each row of C elements is handled by block_size threads
    // furthermore, each block_size threads get executed in warps of 32 threads

    // special reduction operations warpReduceMax/warpReduceSum are used for intra-warp reductions
    // shared memory is used for inter-warp reduction
    extern __shared__ float shared[];
    int idx = blockIdx.x;
    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32; // warp index within a block
    int laneId = threadIdx.x % 32; // thread index within a warp

    // the number of warps per block. recall that blockDim.x is block_size
    int warpsPerBlock = blockDim.x / 32;

    // shared[] must be allocated to have 2 * warpsPerBlock elements
    // first half for max values, the second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // one row of inp, i.e. inp[idx, :] of shape (C,)
    const float* x = inp + idx * C;

    // first, thread coarsening by directly accessing global memory in series
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += blockDim.x) {
        maxval = fmaxf(maxval, x[i]);
    }
    // now within-warp reductions for maxval
    maxval = warpReduceMax(maxval);

    // the 0th thread of each warp writes the maxval of that warp to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // now the 0th thread reduces the maxvals in shared memory, i.e. across warps
    if (tid == 0) {
        float val = maxvals[tid];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        // store the final max in the first position
        maxvals[0] = val;
    }
    __syncthreads();
    // broadcast the max to all threads
    float offset = maxvals[0];

    // compute expf and write the result to global memory
    for (int i = tid; i < C; i += blockDim.x) {
        out[idx * C + i] = expf(x[i] - offset);
    }

    // okay now we calculated exp(x - max(x))
    // step 2: sum all the values and divide by the sum

    // thread coarsening for sum
    x = out + idx * C;
    float sumval = 0.0f;
    for (int i = tid; i < C; i += blockDim.x) {
        sumval += x[i];
    }
    // within-warp reduction for sumval
    sumval = warpReduceSum(sumval);

    // write sumval to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // inter-thread reduction of sum
    if (tid == 0) {
        float val = sumvals[tid];
        for (int i = 1; i < warpsPerBlock; ++i) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();
    // broadcast the sum to all threads
    float sum = sumvals[0];

    // divide the whole row by the sum
    for (int i = tid; i < C; i += blockDim.x) {
        out[idx * C + i] = x[i] / sum;
    }
}



extern "C" float softmax(char * tensor_name)
{

  float * tensor = NamedTensors[tensor_name];
  std::vector<float> dims = NamedDims[tensor_name];
  
  dims =  format_LinearLayer_Dims(dims);

  int B = dims[0];
  int C = dims[1];

  int grid_size = B;
  int block_size = 32;
  size_t shared_mem_size = 2 * block_size / 32 * sizeof(float);


  float *probs;
  hipMalloc(&probs, B*C*sizeof(float));

  softmax_forward_kernel4<<<grid_size, block_size, shared_mem_size>>>(tensor, probs, B, C);

  std::cout << "\n\nPROBS ARE:\n\n";
  PrintTensorF(probs, B, C);

  return 0;
}


// Parallelizes over B, C
__global__ void crossentropy_softmax_backward_kernel1(float* dlogits,
                           const float* probs, const float* targets,
                           int B, int C) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //int i = threadIdx.x;
    
    
    if (i < B * C) {
        int b = i / (C);
        int v = i % C;

        float* dlogits_b = dlogits + b * C;
        const float* probs_b = probs + b * C;

        //float ix = targets[v];
        float ix = targets[b * C + v];
        float p = probs_b[v];

        //float indicator = (v==ix) ? 1.0f : 0.0f;
        float indicator = ix;

        dlogits_b[v] += (p - indicator) / B;
        
    }
}


void CrossEntropyBackward(float *y_hat,
                          float *y,
                          int B, int C, int OC,
                          float *dloss)
{

  int grid_size = B;
  int block_size = 32;
  size_t shared_mem_size = 2 * block_size / 32 * sizeof(float);

  float *probs;
  hipMalloc(&probs, B*C*sizeof(float));

  softmax_forward_kernel4<<<grid_size, block_size, shared_mem_size>>>(y_hat, probs, B, C);

  grid_size = ceil_div(B*C, block_size);
  crossentropy_softmax_backward_kernel1<<<grid_size, block_size>>>(dloss, probs, y, B, C);
  hipFree(probs);

  cudaCheck(hipGetLastError());
}



extern "C" float cross_entropy(char *y_hat, char *y)
{
  

  float *device_y_hat = NamedTensors[y_hat];
  float *device_y = NamedTensors[y];
  std::vector<float> y_hat_dims = NamedDims[y_hat];
  std::vector<float> y_dims = NamedDims[y];


  /*
  std::cout << "y_hat: " << y_hat << "\n";
  PrintDims(y_hat_dims);

  std::cout << "y: " << y << "\n";
  PrintDims(y_dims);
  */

  std::vector<float> linear_layer_dims = format_LinearLayer_Dims(y_hat_dims);
  

  todo_backwards.push_back(std::make_tuple(linear_layer_dims[0], linear_layer_dims[1],
                                           y_dims[0], device_y_hat, nullptr, device_y,
                                           "cross_entropy", "none"));

  return 0;
}



extern "C" float Backpropagation()
{
  //float * loss_gradient = ;
  
  int B, C, OC;
  float *inp, *w, *out, *last_inp;
  float *dinp, *device_dinp, *dw, *device_dw, *dout, *device_dout;

  std::string op, param_name;
  
  bool first=true;

  while(todo_backwards.size()>0)
  {
    backward_tuple bt = std::move(todo_backwards.back());
    todo_backwards.pop_back();
    // TODO: remove loss dw grad and dinp grad at the end of backprop

    
    B = std::get<0>(bt);
    C = std::get<1>(bt);
    OC = std::get<2>(bt);
    inp = std::get<3>(bt);
    w = std::get<4>(bt);
    out = std::get<5>(bt);
    op = std::get<6>(bt);
    param_name = std::get<7>(bt);

    dinp = make_zeros_float(B*C);
    dw = make_zeros_float(OC*C);

    float *new_grad_ptr;
    
    
    if (NamedParamGrads[param_name]==nullptr)
    {
      NamedParamGrads[param_name] = new_grad_ptr;
      cudaCheck(hipMalloc(&new_grad_ptr, OC*C*sizeof(float)));
      NamedParamGrads[param_name] = new_grad_ptr;
    } 
    
    device_dw = NamedParamGrads[param_name];
    

    hipMalloc(&device_dinp, B*C*sizeof(float));
    //hipMalloc(&device_dw, OC*C*sizeof(float));
    
    
    hipMemcpy(device_dinp, dinp, B*C*sizeof(float), hipMemcpyHostToDevice);
    cudaCheck(hipMemcpy(device_dw, dw, OC*C*sizeof(float), hipMemcpyHostToDevice));
    
    /*
    std::cout << "B: " << B << "\n";
    std::cout << "C: " << C << "\n";
    std::cout << "OC: " << OC << "\n";
    std::cout << "Op: " << op << "\n";
    */


    // No switch case for std::string
    if (op=="matmul")
      matmul_backward(inp, w, B, C, OC, device_dinp, device_dw, device_dout);
    else if (op=="cross_entropy")
      CrossEntropyBackward(inp, out, B, C, OC, device_dinp);
    else
      LogErrorS("A operação não possui implementação do backward.");

    /*
    std::cout << "\nd inp:\n";
    PrintTensorF(device_dinp, B, C);
    std::cout << "\n";

    std::cout << "d w:\n";
    PrintTensorF(device_dw, OC, C);
    std::cout << "\n\n";
    */
    NamedParamGrads[param_name] = device_dw;


    // Garbage Collector on all lines below
    cudaCheck(hipFree(out));
    if (!first)
    {
      cudaCheck(hipFree(last_inp));
      cudaCheck(hipFree(device_dout));
    }
    device_dout = device_dinp; // backpropagate gradient
    last_inp = inp;

    first = false;
  }
  cudaCheck(hipFree(device_dinp));
  cudaCheck(hipFree(inp));

  return 0;
}


class Optimizer {
public:
  virtual ~Optimizer() = default;

  int timestep = 1;
  float lr = 0.0f;
  //float eps = 1.5e-4;
  float eps = 1e-8;
    
  virtual void init_states(std::string param_name, float params_count) {}
  virtual void step(float *param, float *grad, std::vector<float> dims, std::string param_name) {}
  virtual void count_step() {
    timestep+=1;
  }
};

class AdamW_optim : public Optimizer {
  std::map<std::string, float *> NamedV, NamedM;
  float lr, beta1, beta2, weight_decay;

  public:
    AdamW_optim(float lr, float beta1, float beta2, float weight_decay)
      : lr(lr), beta1(beta1), beta2(beta2), weight_decay(weight_decay) {}
    
  void init_states(std::string param_name, float params_count) override;
  void step(float *param, float *grad, std::vector<float> dims, std::string param_name) override;
};



__device__ inline float lerp(float start, float end, float weight) {
    return fma(weight, end, fma(-weight, start, start));
}

__global__ void adamw_kernel(float* params_memory, const float* grads_memory, float* m_memory, float* v_memory, long num_parameters,
                              float learning_rate, float beta1, float beta2, float beta1_correction, float beta2_correction,
                              float eps, float weight_decay) {

   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i >= num_parameters) return;  // guard
   float grad = grads_memory[i];
   float m = m_memory[i];
   float v = v_memory[i];
   // update the first moment (momentum)
   m = lerp(grad, m, beta1);
   m_memory[i] = m;
   // update the second moment (RMSprop)
   v = lerp(grad * grad, v, beta2);
   v_memory[i] = v;
   m /= beta1_correction;  // m_hat
   v /= beta2_correction;  // v_hat
   params_memory[i] -= learning_rate * (m / (sqrtf(v) + eps) + weight_decay * params_memory[i]);
}


void AdamW_optim::init_states(std::string param_name, float params_count)
{
  

  if (NamedV[param_name]==nullptr)
  {
    std::cout << "init_states for param " << param_name << " with params count: " << params_count << "\n";

    float *v, *m, *device_v, *device_m;
    v = new float[params_count];
    m = new float[params_count];

    v = make_zeros_float(params_count);
    m = make_zeros_float(params_count);


    hipMalloc(&device_v, params_count*sizeof(float));
    hipMalloc(&device_m, params_count*sizeof(float));
    hipMemcpy(device_v, v, params_count*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_m, m, params_count*sizeof(float), hipMemcpyHostToDevice);

    NamedV[param_name] = device_v; 
    NamedM[param_name] = device_m;
  }
}

void AdamW_optim::step(float *param, float *grad, std::vector<float> dims, std::string param_name)
{
  //std::cout  << "Optimizer step called\n";
  

  float *v = NamedV[param_name];
  float *m = NamedM[param_name];

  float beta1_correction = 1.0f - powf(beta1, timestep);
  float beta2_correction = 1.0f - powf(beta2, timestep);

  

  /*
  std::cout << "param pre: \n";
  PrintTensorF(param, dims[0], dims[1]);

  std::cout << "\n\ngrad: \n";
  PrintTensorF(grad, dims[0], dims[1]);
  */

  int params_count = dims[0]*dims[1];
  int block_size = 512;
  int num_blocks = ceil_div(params_count, block_size);

  adamw_kernel<<<num_blocks, block_size>>>(param, grad, m, v, params_count,
                                           lr, beta1, beta2, beta1_correction, beta2_correction,
                                           eps, weight_decay);


  /*
  std::cout << "\n\nparam post: \n";
  PrintTensorF(param, dims[0], dims[1]);
  std::cout << "\n\n";
  */
}




std::unique_ptr<Optimizer> optimizer = nullptr;


extern "C" float AdamW(float lr, float beta1, float beta2, float weight_decay)
{

  if (optimizer==nullptr)
    optimizer = std::make_unique<AdamW_optim>(lr, beta1, beta2, weight_decay);

  for (auto& pair : NamedParamGrads)
  {
    std::string param_name = pair.first;

    if (param_name!="none")
    {
      optimizer->init_states(param_name, dimsProd(NamedDims[param_name]));
      optimizer->step(NamedTensors[param_name], pair.second, NamedDims[param_name], param_name);
    }
  }
  optimizer->count_step();

  used_cuda=0;

  return 0;
}




Value *BinaryTensorTensorExprAST::codegen() {
  Value *LtensorName = Builder->CreateGlobalString(LHS->GetName());
  Value *RtensorName = Builder->CreateGlobalString(RHS->GetName());
  Value * object_name;

  std::string pre_dot = LHS->GetSelf();
  if (pre_dot=="true")
    LtensorName = Builder->CreateCall(TheModule->getFunction("ConcatFirstArgToVarName"),
                                                      {LtensorName});
    // Gets from pre_dot if it is a class attribute
  else if (pre_dot!="false") {
    object_name = Builder->CreateGlobalString(pre_dot);

    LtensorName = Builder->CreateCall(TheModule->getFunction("ConcatStr"),
                                                      {object_name, LtensorName});
  }
  pre_dot = RHS->GetSelf();
  if (pre_dot=="true")
    RtensorName = Builder->CreateCall(TheModule->getFunction("ConcatFirstArgToVarName"),
                                                      {RtensorName});
    // Gets from pre_dot if it is a class attribute
  else if (pre_dot!="false") {
    object_name = Builder->CreateGlobalString(pre_dot);

    RtensorName = Builder->CreateCall(TheModule->getFunction("ConcatStr"),
                                                      {object_name, RtensorName});
  }


  if (Op == '=') {
    seen_var_attr=true;

    VariableExprAST *LHSE = static_cast<VariableExprAST *>(LHS.get());
    if (!LHSE)
      return LogErrorV("Destino do '=' deve ser uma variável.");
    
    
    Value *Val = RHS->codegen();
    std::cout << "1 1 attr\n";
    if (!Val)
      return nullptr;

    //float *Variable = NamedTensors[LHSE->getName()];
    //if (!Variable)
    //  return LogErrorV("O nome do tensor/variável é desconhecido.");

    
    Function *temporaryCudaResult_AttrFn = TheModule->getFunction("temporaryCudaResult_Attr");
    Builder->CreateCall(temporaryCudaResult_AttrFn, {LtensorName});


    used_cuda=0;
      
      
    
    seen_var_attr=false;
    return Val;
  }


  Value *L = LHS->codegen();
  Value *R = RHS->codegen();
  

  std::string functionName = Builder->GetInsertBlock()->getParent()->getName().str();
  std::cout << "Tensor Tensor for function: " << functionName << "\n";
  int forward_func = 0;
  if(ends_with(functionName, "forward"))
    forward_func = 1;
  forward_func = 1; // TODO: Remove this line


  
  if (!L || !R)
    return nullptr;

    Function *CudaFn;

    std::cout << "Tensor tensor: " << LHS->GetName() << " " << RHS->GetName() << "\n";
    

    Value *used_cuda_aux = ConstantInt::get(Type::getInt32Ty(*TheContext), used_cuda);
    Value *is_forward_func = ConstantInt::get(Type::getInt32Ty(*TheContext), forward_func);
    used_cuda = 1;

  switch (Op)
  {
  case '@':
    CudaFn = TheModule->getFunction("CudaMult");
    return Builder->CreateCall(CudaFn,{LtensorName, RtensorName, used_cuda_aux, is_forward_func},
                               "cudamult");
  case '*':
    CudaFn = TheModule->getFunction("CudaMult");
    return Builder->CreateCall(CudaFn,{LtensorName, RtensorName, used_cuda_aux, is_forward_func},
                               "cudamult");
  case '/':
    CudaFn = TheModule->getFunction("CudaDiv");
    return Builder->CreateCall(CudaFn, {LtensorName, RtensorName, used_cuda_aux},
                               "cudadiv");
  case '+':
    CudaFn = TheModule->getFunction("CudaAdd");
    return Builder->CreateCall(CudaFn, {LtensorName, RtensorName, used_cuda_aux},
                               "cudaadd");
  case '-':
    CudaFn = TheModule->getFunction("CudaSub");
    return Builder->CreateCall(CudaFn, {LtensorName, RtensorName, used_cuda_aux},
                               "cudasub");
  case ':':
    return L;
  case tok_space:
    return R;
  default:
    break;
  }
  

  
  Function *F = getFunction(std::string("binary") + Op);
  assert(F && "Operator not found.");

  Value *Ops[] = {L, R};
  return Builder->CreateCall(F, Ops, "binop");
}


Value *LossBackwardExprAST::codegen()
{
  return Builder->CreateCall(TheModule->getFunction("Backpropagation"),
                             {}, "backprop");
}

Value *LogExprAST::codegen() {
  
  Value *used_cuda_aux = ConstantInt::get(Type::getInt32Ty(*TheContext), used_cuda);
  used_cuda=1;

  return Builder->CreateCall(TheModule->getFunction("logE"),
                             {Builder->CreateGlobalString(Name), used_cuda_aux}, "cudalog");
}



Value *BinaryExprAST::codegen() {
  // Special case '=' because we don't want to emit the LHS as an expression.
  if (Op == '=') {
    seen_var_attr=true;
    // Assignment requires the LHS to be an identifier.
    // This assume we're building without RTTI because LLVM builds that way by
    // default.  If you build LLVM with RTTI this can be changed to a
    // dynamic_cast for automatic error checking.
    VariableExprAST *LHSE = static_cast<VariableExprAST *>(LHS.get());
    if (!LHSE)
      return LogErrorV("Destino do '=' deve ser uma variável.");
    // Codegen the RHS.
    
    Value *Val = RHS->codegen();
    if (!Val)
      return nullptr;

    // Look up the name.
    if (NamedValues.count(LHSE->getName()) != 0) {
      
      Value *Variable = NamedValues[LHSE->getName()];


      if(LHS->GetSelf()=="true")
        Builder->CreateCall(TheModule->getFunction("StoreOnDemand"),
                                                  {Builder->CreateGlobalString(LHSE->getName()),
                                                   Val});
      else
        Builder->CreateStore(Val, Variable);
      
    
    } else if (NamedTensors.count(LHSE->getName()) != 0 ) {
      /*
      float *Variable = NamedTensors[LHSE->getName()];
      if (!Variable)
        return LogErrorV("O nome do tensor/variável é desconhecido.");
      */
      std::cout << "Atribuíndo em 0 0\n";
      
      Value *valStr = Builder->CreateGlobalString(LHSE->getName());
      Function *temporaryCudaResult_AttrFn = TheModule->getFunction("temporaryCudaResult_Attr");
      Builder->CreateCall(temporaryCudaResult_AttrFn, {valStr});
        
      used_cuda=0;
      
    } else if (NamedStrs.count(LHSE->getName()) != 0 ) {
      //std::cout << "ATTRIBUTTING TO STRING: " << LHSE->getName() << "\n";
      Value *Variable = NamedStrs[LHSE->getName()];
      
      if(LHS->GetSelf()=="true")
        Builder->CreateCall(TheModule->getFunction("StoreStrOnDemand"),
                                                  {Builder->CreateGlobalString(LHSE->getName()),
                                                   Val});
      else
        Builder->CreateStore(Val, Variable);

    } else {

      return LogErrorV("O nome da variável é desconhecido.");
    }

    seen_var_attr=false;
    return Val;
  }


  

  Value *L = LHS->codegen();
  Value *R = RHS->codegen();
  
  if (!L || !R)
    return nullptr;


    switch (Op) {
    case '+':
      return Builder->CreateFAdd(L, R, "addtmp");
    case ':':
      return L;
    case tok_space:
      return R;
    case '-':
      return Builder->CreateFSub(L, R, "subtmp");
    case '*':
      return Builder->CreateFMul(L, R, "multmp");
    case '/':
      return Builder->CreateFDiv(L, R, "divtmp");
    case 77:
      return LogErrorV("GOTCHA");
    case '<':
      L = Builder->CreateFCmpULT(L, R, "cmptmp");
      // Convert bool 0/1 to float 0.0 or 1.0
      return Builder->CreateUIToFP(L, Type::getFloatTy(*TheContext), "booltmp");
    case '>':
      L = Builder->CreateFCmpULT(R, L, "cmptmp");
      // Convert bool 0/1 to float 0.0 or 1.0
      return Builder->CreateUIToFP(L, Type::getFloatTy(*TheContext), "booltmp");
    default:
      break;
    }
  

  // If it wasn't a builtin binary operator, it must be a user defined one. Emit
  // a call to it.
  Function *F = getFunction(std::string("binary") + Op);
  assert(F && "Operator not found.");

  Value *Ops[] = {L, R};
  return Builder->CreateCall(F, Ops, "binop");
}


Value *UnaryExprAST::codegen() {
  Value *OperandV = Operand->codegen();
  if (!OperandV)
    return nullptr;

  
  
  //std::cout << "unary used_cuda: " << used_cuda << "\n";
  //std::cout << "Operand type: " << Operand->GetType();
  if (Opcode=='-')
  {
    if (Operand->GetType()=="tensor")
    {
      Value *tensorName = Builder->CreateGlobalString(Operand->GetName());
      Value *used_cuda_aux = ConstantInt::get(Type::getInt32Ty(*TheContext), used_cuda);
      Value *R = ConstantFP::get(Type::getFloatTy(*TheContext), -1);
      used_cuda=1;
      return Builder->CreateCall(TheModule->getFunction("CudaScalarMult"),
                                {tensorName, R, used_cuda_aux}, "cudascalarmult");
    }
    return Builder->CreateFMul(ConstantFP::get(Type::getFloatTy(*TheContext), -1),
                              OperandV, "multmp");
  }

  //std::cout << "Opcode: " << Opcode << "\n";

  if (Opcode=';')
    return ConstantFP::get(Type::getFloatTy(*TheContext), 0);
  

  Function *F = getFunction(std::string("unary") + Opcode);
  if (!F)
    return LogErrorV("Operador unário desconhecido.");

  return Builder->CreateCall(F, OperandV, "unop");
}


Value *IfExprAST::codegen() {
  Value *CondV = Cond->codegen();
  if (!CondV)
    return nullptr;

  // Convert condition to a bool by comparing equal to 0.0.
  CondV = Builder->CreateFCmpONE(
      CondV, ConstantFP::get(*TheContext, APFloat(0.0)), "ifcond");

  Function *TheFunction = Builder->GetInsertBlock()->getParent();

  // Create blocks for the then and else cases.  Insert the 'then' block at the
  // end of the function.
  BasicBlock *ThenBB = BasicBlock::Create(*TheContext, "then", TheFunction);
  BasicBlock *ElseBB = BasicBlock::Create(*TheContext, "else");
  BasicBlock *MergeBB = BasicBlock::Create(*TheContext, "ifcont");

  Builder->CreateCondBr(CondV, ThenBB, ElseBB);

  // Emit then value.
  Builder->SetInsertPoint(ThenBB);

  Value *ThenV = Then->codegen();
  if (!ThenV)
    return nullptr;

  Builder->CreateBr(MergeBB);
  // Codegen of 'Then' can change the current block, update ThenBB for the PHI.
  ThenBB = Builder->GetInsertBlock();

  // Emit else block.
  TheFunction->insert(TheFunction->end(), ElseBB);
  Builder->SetInsertPoint(ElseBB);

  Value *ElseV = Else->codegen();

  if (!ElseV)
  {
    return nullptr;
  }
    

  Builder->CreateBr(MergeBB);
  // Codegen of 'Else' can change the current block, update ElseBB for the PHI.
  ElseBB = Builder->GetInsertBlock();

  // Emit merge block.
  TheFunction->insert(TheFunction->end(), MergeBB);
  Builder->SetInsertPoint(MergeBB);
  PHINode *PN = Builder->CreatePHI(Type::getFloatTy(*TheContext), 2, "iftmp");

  PN->addIncoming(ThenV, ThenBB);
  PN->addIncoming(ElseV, ElseBB);
  
  return PN;
}

// Output for-loop as:
//   var = alloca float
//   ...
//   start = startexpr
//   store start -> var
//   goto loop
// loop:
//   ...
//   bodyexpr
//   ...
// loopend:
//   step = stepexpr
//   endcond = endexpr
//
//   curvar = load var
//   nextvar = curvar + step
//   store nextvar -> var
//   br endcond, loop, endloop
// outloop:
Value *WhileExprAST::codegen() {
	Function* TheFunction = Builder->GetInsertBlock()->getParent();

	BasicBlock *entryBB = BasicBlock::Create(*TheContext, "entry_while", TheFunction);
	BasicBlock *LoopBB = BasicBlock::Create(*TheContext, "loop_while", TheFunction);
	BasicBlock *AfterBB = BasicBlock::Create(*TheContext, "end_while", TheFunction);

	
	Builder->CreateBr(entryBB);

	// Handle Cond

	Builder->SetInsertPoint(entryBB);
	Value* condVal = Cond->codegen();
	if (! condVal)
    return nullptr;

	condVal = Builder->CreateFCmpONE(condVal, ConstantFP::get(*TheContext, APFloat(0.0)), "loopcond");
	Builder->CreateCondBr(condVal, LoopBB, AfterBB);
	entryBB = Builder->GetInsertBlock();


	// Handle Loop Body
	
  Builder->SetInsertPoint(LoopBB);
	Value* bodyVal = Body->codegen();
	if (! bodyVal)
    return nullptr;
	Builder->CreateBr(entryBB);


	// Handle Loop End
	
	Builder->SetInsertPoint(AfterBB);

	return Constant::getNullValue(Type::getFloatTy(*TheContext));
}


Value *ForExprAST::codegen() {
  Function *TheFunction = Builder->GetInsertBlock()->getParent();

  // Create an alloca for the variable in the entry block.
  AllocaInst *Alloca = CreateEntryBlockAlloca(TheFunction, VarName);

  // Emit the start code first, without 'variable' in scope.
  Value *StartVal = Start->codegen();
  if (!StartVal)
    return nullptr;

  // Store the value into the alloca.
  Builder->CreateStore(StartVal, Alloca);

  // Make the new basic block for the loop header, inserting after current
  // block.
  BasicBlock *LoopBB = BasicBlock::Create(*TheContext, "loop", TheFunction);

  // Insert an explicit fall through from the current block to the LoopBB.
  Builder->CreateBr(LoopBB);

  
  Builder->SetInsertPoint(LoopBB);

  // Within the loop, the variable is defined equal to the PHI node.  If it
  // shadows an existing variable, we have to restore it outside this scope
  AllocaInst *OldVal = NamedValues[VarName];
  NamedValues[VarName] = Alloca;

  // Emit the body of the loop.  This, like any other expr, can change the
  // current BB.  Note that we ignore the value computed by the body, but don't
  // allow an error.
  if (!Body->codegen())
    return nullptr;

  // Emit the step value.
  Value *StepVal = nullptr;
  if (Step) {
    StepVal = Step->codegen();
    if (!StepVal)
      return nullptr;
  } 

  // Compute the end condition.
  Value *EndCond = End->codegen();
  if (!EndCond)
    return nullptr;

  // Reload, increment, and restore the alloca.  This handles the case where
  // the body of the loop mutates the variable.
  Value *CurVar = Builder->CreateLoad(Type::getFloatTy(*TheContext), Alloca,
                                      VarName.c_str());
  Value *NextVar = Builder->CreateFAdd(CurVar, StepVal, "nextvar"); // Increment
  Builder->CreateStore(NextVar, Alloca);

  // Convert condition to a bool by comparing equal to 0.0.
  EndCond = Builder->CreateFCmpONE(
      EndCond, ConstantFP::get(*TheContext, APFloat(0.0)), "loopcond");

  // Create the "after loop" block and insert it.
  BasicBlock *AfterBB =
      BasicBlock::Create(*TheContext, "afterloop", TheFunction);

  // goto branch
  Builder->CreateCondBr(EndCond, LoopBB, AfterBB);

  // Any new code will be inserted in AfterBB.
  Builder->SetInsertPoint(AfterBB);

  // Restore the unshadowed variable.
  if (OldVal)
    NamedValues[VarName] = OldVal;
  else
    NamedValues.erase(VarName);

  // for expr always returns 0.0.
  return Constant::getNullValue(Type::getFloatTy(*TheContext));
}


// Create Var
Value *VarExprAST::codegen() {
  std::vector<AllocaInst *> OldBindings;

  Function *TheFunction = Builder->GetInsertBlock()->getParent();


  // Register all variables and emit their initializer.
  for (unsigned i = 0, e = VarNames.size(); i != e; ++i) {
    const std::string &VarName = VarNames[i].first;
    ExprAST *Init = VarNames[i].second.get();

    // Emit the initializer before adding the variable to scope, this prevents
    // the initializer from referencing the variable itself, and permits stuff
    // like this:
    //  var a = 1 in
    //    var a = a in ...   # refers to outer 'a'.
    Value *InitVal;
    if (Init) {
      InitVal = Init->codegen();
      if (!InitVal)
        return nullptr;
    } else { // If not specified, use 0.0.
      InitVal = ConstantFP::get(*TheContext, APFloat(0.0));
    }


    AllocaInst *Alloca = CreateEntryBlockAlloca(TheFunction, VarName);
    Builder->CreateStore(InitVal, Alloca);
      
    // Remember the old variable binding so that we can restore the binding when
    // we unrecurse.
    OldBindings.push_back(NamedValues[VarName]);

    // Remember this binding.
    NamedValues[VarName] = Alloca;
    
    
    
  }

  // Codegen the body that is contained by the in expression
  Value *BodyVal = Body->codegen();
  if (!BodyVal)
    return nullptr;

  // Pop all our variables from scope.
  for (unsigned i = 0, e = VarNames.size(); i != e; ++i)
    NamedValues[VarNames[i].first] = OldBindings[i];

  // Return the body computation.
  return BodyVal;
}





Value *StrExprAST::codegen() {
  std::vector<AllocaInst *> OldBindings;

  Function *TheFunction = Builder->GetInsertBlock()->getParent();


  // Register all variables and emit their initializer.
  for (unsigned i = 0, e = VarNames.size(); i != e; ++i) {
    const std::string &VarName = VarNames[i].first;
    ExprAST *Init = VarNames[i].second.get();

    // Emit the initializer before adding the variable to scope, this prevents
    // the initializer from referencing the variable itself, and permits stuff
    // like this:
    //  var a = 1 in
    //    var a = a in ...   # refers to outer 'a'.
    Value *InitVal;
    if (Init) {
      InitVal = Init->codegen();
      if (!InitVal)
        return nullptr;
    } else { // If not specified, use 0.0.
      InitVal = ConstantFP::get(*TheContext, APFloat(0.0));
    }


    AllocaInst *Alloca = CreateEntryBlockAlloca(TheFunction, VarName);
    Builder->CreateStore(InitVal, Alloca);
      
    // Remember the old variable binding so that we can restore the binding when
    // we unrecurse.
    //std::cout << "STRING CODEGEN FOR " << VarName << "\n";
    OldBindings.push_back(NamedStrs[VarName]);

    
    // Remember this binding.
    NamedStrs[VarName] = Alloca;
    
  }

  // Codegen the body that is contained by the in expression
  Value *BodyVal = Body->codegen();
  if (!BodyVal)
    return nullptr;

  // Pop all our variables from scope.
  for (unsigned i = 0, e = VarNames.size(); i != e; ++i)
    NamedStrs[VarNames[i].first] = OldBindings[i];

  // Return the body computation.
  return BodyVal;
}






std::vector<float> cur_dim;

extern "C" float StoreDimsOnDemand(float d)
{
  cur_dim.push_back(d);
  return 0;
}

extern "C" float CreateTensorOnDemand(char *tensorName, int is_obj_attr_or_self, char *init)
{
  
  std::string objectTensorName = tensorName;
  if (is_obj_attr_or_self)
    objectTensorName = FirstArg + tensorName;

  char * cObjectTensorName = new char[objectTensorName.length() + 1];
  std::strcpy(cObjectTensorName, objectTensorName.c_str());


  //float * d = (float *) dims;
  int product = dimsProd(cur_dim);
  float * tensor;
  float * tensor_cpu;


  if (std::strcmp(init, "randu") == 0)
    tensor_cpu = make_random_float(product);
  else if (std::strcmp(init, "zeros") == 0)
    tensor_cpu = make_zeros_float(product);
  else if (std::strcmp(init, "ones") == 0)
    tensor_cpu = make_ones_float(product);
  else if (std::strcmp(init, "xavu") == 0)
    tensor_cpu = make_xavier_uniform_float(product, cur_dim[cur_dim.size()-1], cur_dim[cur_dim.size()-2]);
  else if (std::strcmp(init, "randint") == 0)
    tensor_cpu = make_random_int(product, 10);
  

  

  hipMalloc(&tensor, product*sizeof(float));
  cudaCheck(hipMemcpy(tensor, tensor_cpu, product*sizeof(float), hipMemcpyHostToDevice));
  

  NamedTensors[cObjectTensorName] = tensor;
  NamedDims[cObjectTensorName] = cur_dim;



  //PrintTensor(cObjectTensorName);

  cur_dim.clear();

  return 0;
}

Value *TensorExprAST::codegen() {
  std::vector<AllocaInst *> OldBindings;


  Function *TheFunction = Builder->GetInsertBlock()->getParent();

  // Register all variables and emit their initializer.
  for (unsigned i = 0, e = VarNames.size(); i != e; ++i) {
    const std::string &VarName = VarNames[i].first;
    ExprAST *Init = VarNames[i].second.get();

    // Emit the initializer before adding the variable to scope, this prevents
    // the initializer from referencing the variable itself, and permits stuff
    // like this:
    //  var a = 1 in
    //    var a = a in ...   # refers to outer 'a'.
    Value *InitVal;
    if (Init) {
      InitVal = Init->codegen();
      if (!InitVal)
        return nullptr;
    } else { // If not specified, use 0.0.
      InitVal = ConstantFP::get(*TheContext, APFloat(0.0));
    }


    std::vector<float> dims;
    Value *aux;
    std::vector<Value *> dim_values;


    for (int j=0; j<V_Dims.size(); j++)
    {
      aux = V_Dims[j]->codegen();
      Builder->CreateCall(TheModule->getFunction("StoreDimsOnDemand"),
                                                  {aux});
      //dims.push_back(cast<ConstantFP>(aux)->getValueAPF().convertToFloat());
      //std::cout << "Dim: " << cast<ConstantFP>(aux)->getValueAPF().convertToFloat() << "\n";
    }
    //void * v_dims_ptr = &V_Dims;

    
    int is_obj_attr_or_self = 0;
    if (GetSelf()!="false")
      is_obj_attr_or_self=1;
    
    Builder->CreateCall(TheModule->getFunction("CreateTensorOnDemand"),
                                              {Builder->CreateGlobalString(VarName),
                                               ConstantInt::get(Type::getInt32Ty(*GlobalContext), is_obj_attr_or_self),
                                               Builder->CreateGlobalString(TensorInit)});

    /*
    //SetDims(dims);

    int product = dimsProd(Dims);
    float * tensor_cpu = make_random_float(product);
    float * tensor;

    hipMalloc(&tensor, product*sizeof(float));
    cudaCheck(hipMemcpy(tensor, tensor_cpu, product*sizeof(float), hipMemcpyHostToDevice));

    NamedTensors[VarName] = tensor;
    NamedDims[VarName] = Dims;



    
    
    Builder->CreateCall(TheModule->getFunction("PrintTensor"),
                        {Builder->CreateGlobalString(VarName)});
    */
     
  }

  // Codegen the body that is contained by the in expression

  Value *BodyVal = Body->codegen();
  if (!BodyVal)
    return nullptr;



  // Return the body computation.
  return BodyVal;
}




Value *CallExprAST::codegen() {
  // Look up the name in the global module table.
  std::string tgt_function = Callee;
  

  Function *TheFunction = Builder->GetInsertBlock()->getParent();
  std::string functionName = TheFunction->getName().str();
  std::string tgt_function_name;

  int nested_function;
  if (functionName=="__anon_expr")
    nested_function=0;
  else
    nested_function=1;

  int args_removal = 0;
  if(Class!="None")
  {
    if (!in_str(tgt_function, tensor_methods))
      tgt_function = Class+tgt_function;
    Builder->CreateCall(TheModule->getFunction("FirstArgOnDemand"),
                                                  {Builder->CreateGlobalString(Pre_dot),
                                                   ConstantInt::get(Type::getInt32Ty(*TheContext), nested_function)});
    //args_removal=1;
  }

  Function *CalleeF = getFunction(tgt_function);
  if (!CalleeF)
  {
    std::string _error = "Função referenciada, "+ tgt_function +", ainda não foi declarada";
    return LogErrorV(_error);
  }

  tgt_function_name = CalleeF->getName().str();

  // If argument mismatch error.
  if ((CalleeF->arg_size()-args_removal) != Args.size() && !in_str(tgt_function_name, vararg_methods))
    return LogErrorV("Parâmetros passados incorretos.");

  std::vector<Value *> ArgsV;
  
  //if(Class!="None")
  //  ArgsV.push_back(ConstantFP::get(Type::getFloatTy(*TheContext), APFloat(0.0f)));

  
  for (unsigned i = 0, e = Args.size(); i != e; ++i) {
    //std::cout << "\n\nCallExprAST::codegen for argument n°: " << i << ".\n";

    Value * arg;
    if (Args[i]->GetType()=="tensor")
      arg = Builder->CreateGlobalString(Args[i]->GetName());
    else
      arg = Args[i]->codegen();

    //std::cout << "Args[i]: " << Args[i]->GetName() << "\n";


    ArgsV.push_back(arg);


    if (!ArgsV.back())
      return nullptr;
  }
  
  //std::cout << "\n\n";

  Value * ret = Builder->CreateCall(CalleeF, ArgsV, "calltmp");
  if(Class!="None")
    Builder->CreateCall(TheModule->getFunction("DimnishFirstArgOnDemand"),
                                                  {Builder->CreateGlobalString(Pre_dot),
                                                   ConstantInt::get(Type::getInt32Ty(*TheContext), nested_function)});
  return ret;
}



Function *PrototypeAST::codegen() {
  // Make the function type:  float(float,float) etc.

  std::vector<Type *> types;
  for (auto &type : Types)
  {
    if (type=="s")
      types.push_back(PointerType::get(Type::getInt8Ty(*TheContext), 0));
    else
      types.push_back(Type::getFloatTy(*TheContext));
  }

  //std::vector<Type *> Floats(Args.size(), Type::getFloatTy(*TheContext));
  
  /*
  if (Args.size()>0)
    if (Args[0]=="self")
      Floats[0] = PointerType::get(Type::getInt8Ty(*TheContext), 0);
  */

  FunctionType *FT = FunctionType::get(Type::getFloatTy(*TheContext), types, false);
  

  Function *F =
      Function::Create(FT, Function::ExternalLinkage, Name, TheModule.get());

  // Set names for all arguments.
  unsigned Idx = 0;
  for (auto &Arg : F->args())
    Arg.setName(Args[Idx++]);
  

  return F;
}

const PrototypeAST& FunctionAST::getProto() const {
  return *Proto;
}

const std::string& FunctionAST::getName() const {
  return Proto->getName();
}

Function *FunctionAST::codegen() {
  
  // Transfer ownership of the prototype to the FunctionProtos map, but keep a
  // reference to it for use below.
  auto &P = *Proto;

    

  FunctionProtos[Proto->getName()] = std::move(Proto);
  Function *TheFunction = getFunction(P.getName());
  if (!TheFunction)
    return nullptr;

  // If this is an operator, install it.
  if (P.isBinaryOp())
    BinopPrecedence[P.getOperatorName()] = P.getBinaryPrecedence();

  // Create a new basic block to start insertion into.
  BasicBlock *BB = BasicBlock::Create(*TheContext, "entry", TheFunction);
  Builder->SetInsertPoint(BB);


  


  // Record the function arguments in the NamedValues map.


  //std::cout << "\n\n";

  NamedValues.clear();

  float val;
  int i = 0;
  for (auto &Arg : TheFunction->args()) {
    // Create an alloca for this variable.
    
    //std::cout << "Create Function alloca for: " << Arg.getName().str() << "\n";
    AllocaInst *Alloca = CreateEntryBlockAlloca(TheFunction, Arg.getName());

    // Store the initial value into the alloca.
    Builder->CreateStore(&Arg, Alloca);

    // Add arguments to variable symbol table.
    NamedValues[std::string(Arg.getName())] = Alloca;
    
  }
  //std::cout << "\n\n";


  if (Value *RetVal = Body->codegen()) {
    // Finish off the function.
    Builder->CreateRet(RetVal);

    // Validate the generated code, checking for consistency.
    verifyFunction(*TheFunction);


    return TheFunction;
  }


  // Error reading body, remove function.
  TheFunction->eraseFromParent();

  if (P.isBinaryOp())
    BinopPrecedence.erase(P.getOperatorName());
  return nullptr;
}





const PrototypeAST& ClassAST::getProto(int i) const {
  return Functions[i]->getProto(); 
}

const std::string& ClassAST::getName(int i) const {
  return Functions[i]->getProto().getName();
}

Value *ClassAST::codegen() {
  /*
  // Transfer ownership of the prototype to the FunctionProtos map, but keep a
  // reference to it for use below.
  auto &P = *Proto;
  FunctionProtos[Proto->getName()] = std::move(Proto);
  Function *TheFunction = getFunction(P.getName());
  if (!TheFunction)
    return nullptr;

  // If this is an operator, install it.
  if (P.isBinaryOp())
    BinopPrecedence[P.getOperatorName()] = P.getBinaryPrecedence();

  // Create a new basic block to start insertion into.
  BasicBlock *BB = BasicBlock::Create(*TheContext, "entry", TheFunction);
  Builder->SetInsertPoint(BB);



  // Record the function arguments in the NamedValues map.
  NamedValues.clear();
  float val;
  int i = 0;
  for (auto &Arg : TheFunction->args()) {
    // Create an alloca for this variable.
    AllocaInst *Alloca = CreateEntryBlockAlloca(TheFunction, Arg.getName());

    // Store the initial value into the alloca.
    Builder->CreateStore(&Arg, Alloca);

    // Add arguments to variable symbol table.
    NamedValues[std::string(Arg.getName())] = Alloca;

    
  }


  if (Value *RetVal = Body->codegen()) {
    // Finish off the function.
    Builder->CreateRet(RetVal);

    // Validate the generated code, checking for consistency.
    verifyFunction(*TheFunction);

    return TheFunction;
  }


  // Error reading body, remove function.
  TheFunction->eraseFromParent();

  
  if (P.isBinaryOp())
    BinopPrecedence.erase(P.getOperatorName());
  */
  return nullptr;
}





//===----------------------------------------------------------------------===//
// Top-Level parsing and JIT Driver
//===----------------------------------------------------------------------===//

static void InitializeModule() {
  used_cuda=0;
  // Open a new context and module.
  TheContext = std::make_unique<LLVMContext>();
  TheModule = std::make_unique<Module>("my cool jit", *TheContext);
  TheModule->setDataLayout(TheJIT->getDataLayout());

  //std::cout << "Initialize Module\n";
  // todo: It's creating one initialize for each ";" (top level expression).

  // Create a new builder for the module.
  Builder = std::make_unique<IRBuilder<>>(*TheContext);

  Type *floatPtrType = PointerType::get(Type::getFloatTy(*TheContext), 0);

  //===----------------------------------------------------------------------===//
  // Tensor -- Scalar   Operations
  //===----------------------------------------------------------------------===//

  // char *, float, int
  FunctionType *CudaScalarMultTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getFloatTy(*TheContext), Type::getInt32Ty(*TheContext)}, 
      false // Not vararg
  );

  Function::Create(
    CudaScalarMultTy,
    Function::ExternalLinkage, // Linkage (e.g., external for linking with other modules)
    "CudaScalarMult", // Function name
    TheModule.get() // Module to which the function belongs
  );



  // char *, float, int
  FunctionType *CudaScalarDivTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getFloatTy(*TheContext), Type::getInt32Ty(*TheContext)}, 
      false // Not vararg
  );

  Function::Create(
    CudaScalarDivTy,
    Function::ExternalLinkage, // Linkage (e.g., external for linking with other modules)
    "CudaScalarDiv", // Function name
    TheModule.get() // Module to which the function belongs
  );



  // char *, float, int
  FunctionType *CudaScalarAddTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getFloatTy(*TheContext), Type::getInt32Ty(*TheContext)}, 
      false // Not vararg
  );

  Function::Create(
    CudaScalarAddTy,
    Function::ExternalLinkage, 
    "CudaScalarAdd", 
    TheModule.get() 
  );



  // char *, float, int
  FunctionType *CudaScalarSubTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getFloatTy(*TheContext), Type::getInt32Ty(*TheContext)}, 
      false // Not vararg
  );

  Function::Create(
    CudaScalarSubTy,
    Function::ExternalLinkage, 
    "CudaScalarSub", 
    TheModule.get()
  );


  //===----------------------------------------------------------------------===//
  // Tensor Tensor CUDA Ops
  //===----------------------------------------------------------------------===//


  // char *, char *, int
  FunctionType *CudaMultTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getInt32Ty(*TheContext), Type::getInt32Ty(*TheContext)}, 
      false // Not vararg
  );

  Function::Create(
    CudaMultTy,
    Function::ExternalLinkage, // Linkage (e.g., external for linking with other modules)
    "CudaMult", // Function name
    TheModule.get() // Module to which the function belongs
  );



  //===----------------------------------------------------------------------===//
  // Backward and Optimizers CUDA Ops
  //===----------------------------------------------------------------------===//

  //
  FunctionType *BackpropagationTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {}, 
      false // Not vararg
  );
  Function::Create(
    BackpropagationTy,
    Function::ExternalLinkage,
    "Backpropagation",
    TheModule.get()
  );

  //
  FunctionType *AdamWTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {Type::getFloatTy(*TheContext),
       Type::getFloatTy(*TheContext),
       Type::getFloatTy(*TheContext),
       Type::getFloatTy(*TheContext)}, 
      false // Not vararg
  );
  Function::Create(
    AdamWTy,
    Function::ExternalLinkage,
    "AdamW",
    TheModule.get()
  );

  //===----------------------------------------------------------------------===//
  // Unary CUDA Ops
  //===----------------------------------------------------------------------===//

  // char *, int
  FunctionType *CudaLogTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getInt32Ty(*TheContext)},
      false // Not vararg
  );
  Function::Create(
    CudaLogTy,
    Function::ExternalLinkage,
    "logE",
    TheModule.get()
  );


  // char *
  FunctionType *softmaxTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false
  );
  Function::Create(
    softmaxTy,
    Function::ExternalLinkage,
    "softmax",
    TheModule.get()
  );

  // 
  FunctionType *onehotTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {Type::getFloatTy(*TheContext)},
      false
  );
  Function::Create(
    onehotTy,
    Function::ExternalLinkage,
    "onehot",
    TheModule.get()
  );

  // 
  FunctionType *sumTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {},
      false
  );
  Function::Create(
    sumTy,
    Function::ExternalLinkage,
    "sum",
    TheModule.get()
  );

  // 
  FunctionType *meanTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {},
      false
  );
  Function::Create(
    meanTy,
    Function::ExternalLinkage,
    "mean",
    TheModule.get()
  );

  // 
  FunctionType *maxTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {},
      false
  );
  Function::Create(
    maxTy,
    Function::ExternalLinkage,
    "max",
    TheModule.get()
  );

  
  // char *, floats, Vararg
  FunctionType *viewTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getFloatTy(*TheContext),Type::getFloatTy(*TheContext),Type::getFloatTy(*TheContext),Type::getFloatTy(*TheContext),Type::getFloatTy(*TheContext),Type::getFloatTy(*TheContext),Type::getFloatTy(*TheContext),Type::getFloatTy(*TheContext)},
      true // Vararg
  );
  Function::Create(
    viewTy,
    Function::ExternalLinkage,
    "view",
    TheModule.get()
  );
  

  //===----------------------------------------------------------------------===//
  // Loss CUDA Ops
  //===----------------------------------------------------------------------===//


  // char *, char *
  FunctionType *cross_entropyTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), PointerType::get(Type::getInt8Ty(*TheContext), 0)}, 
      false // Not vararg
  );
  Function::Create(
    cross_entropyTy,
    Function::ExternalLinkage, // Linkage (e.g., external for linking with other modules)
    "cross_entropy", // Function name
    TheModule.get() // Module to which the function belongs
  );

  //===----------------------------------------------------------------------===//
  // File Handling
  //===----------------------------------------------------------------------===//

  
  // char *
  FunctionType *load_imgTy = FunctionType::get(
      PointerType::get(Type::getFloatTy(*GlobalContext), 0),
      {PointerType::get(Type::getInt8Ty(*GlobalContext), 0)},
      false // Not vararg
  );
  Function::Create(
    load_imgTy,
    Function::ExternalLinkage,
    "load_img",
    TheModule.get()
  );
  



  // float, char *, ... 
  FunctionType *yieldTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {Type::getFloatTy(*TheContext), PointerType::get(Type::getInt8Ty(*TheContext), 0), PointerType::get(Type::getInt8Ty(*TheContext), 0),PointerType::get(Type::getInt8Ty(*TheContext), 0),PointerType::get(Type::getInt8Ty(*TheContext), 0),PointerType::get(Type::getInt8Ty(*TheContext), 0),PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      true // vararg
  );
  Function::Create(
    yieldTy,
    Function::ExternalLinkage,
    "Datasetyield",
    TheModule.get()
  );

  // float, char *, ... 
  FunctionType *init_datasetTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {Type::getFloatTy(*TheContext)},
      false
  );
  Function::Create(
    init_datasetTy,
    Function::ExternalLinkage,
    "Datasetinit_dataset",
    TheModule.get()
  );


  
  // char *
  FunctionType *Datasetgetitem_1Ty = FunctionType::get(
      //PointerType::get(Type::getFloatTy(*TheContext), 0),
      Type::getFloatTy(*TheContext),
      {Type::getFloatTy(*TheContext), PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false
  );
  Function::Create(
    Datasetgetitem_1Ty,
    Function::ExternalLinkage,
    "Datasetgetitem_1",
    TheModule.get()
  );


  // char *
  FunctionType *load_preprocess_imgTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false
  );
  Function::Create(
    load_preprocess_imgTy,
    Function::ExternalLinkage,
    "load_preprocess_img",
    TheModule.get()
  );



  //===----------------------------------------------------------------------===//
  // Str Ops
  //===----------------------------------------------------------------------===//


  // char *
  FunctionType *globTy = FunctionType::get(
      PointerType::get(Type::getInt8Ty(*TheContext), 0),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false // Not vararg
  );
  Function::Create(
    globTy,
    Function::ExternalLinkage,
    "_glob_b_",
    TheModule.get()
  );


  // char *
  FunctionType *PrintStrTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0)}, 
      false 
  );
  Function::Create(
    PrintStrTy,
    Function::ExternalLinkage, 
    "PrintStr", 
    TheModule.get() 
  );


  // char *
  FunctionType *shuffle_strTy = FunctionType::get(
      PointerType::get(Type::getInt8Ty(*TheContext), 0),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0)}, 
      false 
  );
  Function::Create(
    shuffle_strTy,
    Function::ExternalLinkage, 
    "shuffle_str", 
    TheModule.get() 
  );


  //===----------------------------------------------------------------------===//
  // Other Ops
  //===----------------------------------------------------------------------===//


  // char *, int
  FunctionType *FirstArgOnDemandTy = FunctionType::get(
      //PointerType::get(Type::getVoidTy(*TheContext), 0),
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getInt32Ty(*TheContext)},
      false // Not vararg
  );
  Function::Create(
    FirstArgOnDemandTy,
    Function::ExternalLinkage,
    "FirstArgOnDemand",
    TheModule.get()
  );
  



  // char *, int
  FunctionType *DimnishFirstArgOnDemandTy = FunctionType::get(
      //PointerType::get(Type::getVoidTy(*TheContext), 0),
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getInt32Ty(*TheContext)},
      false // Not vararg
  );
  Function::Create(
    DimnishFirstArgOnDemandTy,
    Function::ExternalLinkage,
    "DimnishFirstArgOnDemand",
    TheModule.get()
  );
  

  // char *, char *
  FunctionType * ConcatStrTy = FunctionType::get(
      //PointerType::get(Type::getVoidTy(*TheContext), 0),
      PointerType::get(Type::getInt8Ty(*TheContext), 0),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false // Not vararg
  );
  Function::Create(
    ConcatStrTy,
    Function::ExternalLinkage,
    "ConcatStr",
    TheModule.get()
  );


  // char *, char *
  FunctionType *ConcatFirstArgToVarNameTy = FunctionType::get(
      PointerType::get(Type::getInt8Ty(*TheContext), 0),
      //{PointerType::get(Type::getInt8Ty(*TheContext), 0), PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      {PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false // Not vararg
  );
  Function::Create(
    ConcatFirstArgToVarNameTy,
    Function::ExternalLinkage,
    "ConcatFirstArgToVarName",
    TheModule.get()
  );


  // char *, float
  FunctionType *StoreOnDemandTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), Type::getFloatTy(*TheContext)},
      false // Not vararg
  );
  Function::Create(
    StoreOnDemandTy,
    Function::ExternalLinkage,
    "StoreOnDemand",
    TheModule.get()
  );


    // char *, float
  FunctionType *StoreStrOnDemandTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0), PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false // Not vararg
  );
  Function::Create(
    StoreStrOnDemandTy,
    Function::ExternalLinkage,
    "StoreStrOnDemand",
    TheModule.get()
  );


  // char *
  FunctionType *LoadOnDemandTy = FunctionType::get(
      //PointerType::get(Type::getVoidTy(*TheContext), 0),
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false // Not vararg
  );
  Function::Create(
    LoadOnDemandTy,
    Function::ExternalLinkage,
    "LoadOnDemand",
    TheModule.get()
  );



  // char *
  FunctionType *StoreDimsOnDemandTy = FunctionType::get(
      //PointerType::get(Type::getVoidTy(*TheContext), 0),
      Type::getFloatTy(*TheContext),
      {Type::getFloatTy(*TheContext)},
      //{PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false // Not vararg
  );
  Function::Create(
    StoreDimsOnDemandTy,
    Function::ExternalLinkage,
    "StoreDimsOnDemand",
    TheModule.get()
  );


  // char *, int
  FunctionType *CreateTensorOnDemandTy = FunctionType::get(
      //PointerType::get(Type::getVoidTy(*TheContext), 0),
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0),
       Type::getInt32Ty(*TheContext),
       PointerType::get(Type::getInt8Ty(*TheContext), 0)},
      false // Not vararg
  );
  Function::Create(
    CreateTensorOnDemandTy,
    Function::ExternalLinkage,
    "CreateTensorOnDemand",
    TheModule.get()
  );




  // float, char *
  FunctionType *CallToStoredValuesTy = FunctionType::get(
      PointerType::get(Type::getFloatTy(*TheContext), 0),
      {Type::getFloatTy(*TheContext), PointerType::get(Type::getInt8Ty(*TheContext), 0)}, 
      false 
  );
  Function::Create(
    CallToStoredValuesTy,
    Function::ExternalLinkage, 
    "toStoredValues", 
    TheModule.get() 
  );


  // char *
  FunctionType *temporaryCudaResult_AttrTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0)}, 
      false 
  );
  Function::Create(
    temporaryCudaResult_AttrTy,
    Function::ExternalLinkage, 
    "temporaryCudaResult_Attr", 
    TheModule.get() 
  );


  // char *
  FunctionType *printTTy = FunctionType::get(
      Type::getFloatTy(*TheContext),
      {PointerType::get(Type::getInt8Ty(*TheContext), 0)}, 
      false 
  );
  Function::Create(
    printTTy,
    Function::ExternalLinkage, 
    "PrintTensor", 
    TheModule.get() 
  );
  

}

ThreadSafeModule irgenAndTakeOwnership(FunctionAST &FnAST,
                                       const std::string &Suffix) {
  if (auto *F = FnAST.codegen()) {
    F->setName(F->getName() + Suffix);
    auto TSM = ThreadSafeModule(std::move(TheModule), std::move(TheContext));
    // Start a new module.
    InitializeModule();
    return TSM;
  } else
    report_fatal_error("Não foi possível compilar a função JIT de forma lazy");
}

ThreadSafeModule irgenAndTakeOwnershipClass(ClassAST &FnAST,
                                       const std::string &Suffix) {
  if (auto *F = FnAST.codegen()) {
    F->setName(F->getName() + Suffix);
    auto TSM = ThreadSafeModule(std::move(TheModule), std::move(TheContext));
    // Start a new module.
    InitializeModule();
    return TSM;
  } else
    report_fatal_error("Não foi possível compilar a função JIT de forma lazy");
}



static void HandleClass() {
  

  ParseClass();

}

static void HandleDefinition() {
  
  if (auto FnAST = ParseDefinition()) {
    FunctionProtos[FnAST->getProto().getName()] =
      std::make_unique<PrototypeAST>(FnAST->getProto());
    ExitOnErr(TheJIT->addAST(std::move(FnAST)));
  } else {
    // Skip token for error recovery.
    getNextToken();
  }
}

static void HandleExtern() {
  if (auto ProtoAST = ParseExtern()) {
    if (auto *FnIR = ProtoAST->codegen()) {
      fprintf(stderr, "Ler extern: ");
      FnIR->print(errs());
      fprintf(stderr, "\n");
      FunctionProtos[ProtoAST->getName()] = std::move(ProtoAST);
    }
  } else {
    // Skip token for error recovery.
    getNextToken();
  }
}

static void HandleTopLevelExpression() {
  // Evaluate a top-level expression into an anonymous function.
  if (auto FnAST = ParseTopLevelExpr()) {
    if (FnAST->codegen()) {
      // Create a ResourceTracker for memory managment
      // anonymous expression -- that way we can free it after executing.
      auto RT = TheJIT->getMainJITDylib().createResourceTracker();

      auto TSM = ThreadSafeModule(std::move(TheModule), std::move(TheContext));
      ExitOnErr(TheJIT->addModule(std::move(TSM), RT));
      // Add IR module

      InitializeModule();

      // Points __anon_expr
      auto Sym = ExitOnErr(TheJIT->lookup("__anon_expr"));

      // Get the symbol's address and cast it to the right type (takes no
      // arguments, returns a float) so we can call it as a native function.
      auto *FP = Sym.getAddress().toPtr<float (*)()>();
      auto fp = FP();
      
      //std::cout << "\nResult times 5 is " << fp*5 << "\n";
      fprintf(stderr, "%.2f\n", fp);

      // Delete the anonymous expression module from the JIT.
      ExitOnErr(RT->remove());
    }
  } else {
    // Skip token for error recovery.
    getNextToken();
  }
}

/// top ::= definition | external | expression | ';'
static void MainLoop() {
  while (true) {
    //if (CurTok!=tok_space)
    //  std::cout << "MAIN LOOP, reading token: " << CurTok << "\n";
    switch (CurTok) {
    case tok_eof:
      return;
    case ';': // ignore top-level semicolons.
      getNextToken();
      break;
    case tok_space:
      getNextToken();
      break;
    case tok_tab:
      LogError("Tab inesperado encontrado\n");
      break;
    case tok_def:
      HandleDefinition();
      break;
    case tok_class:
      HandleClass();
      break;
    case tok_extern:
      HandleExtern();
      break;
    //case (tok_space || 59):
    default:
      HandleTopLevelExpression();
      break;
    }
  }
}


//===----------------------------------------------------------------------===//
// "Library" functions that can be "extern'd" from user code.
//===----------------------------------------------------------------------===//

/// putchard - putchar that takes a float and returns 0.
extern "C" float putchard(float X) {
  fputc((char)X, stderr);
  return 0;
}

/// printd - printf that takes a float prints it as "%f\n", returning 0.
extern "C" float printd(float X) {
  fprintf(stderr, "%f\n", X);
  return 0;
}

//===----------------------------------------------------------------------===//
// Main driver code.
//===----------------------------------------------------------------------===//

int main() {

  int deviceIdx = 0;
  cudaCheck(hipSetDevice(deviceIdx));
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceIdx);
  printf("Device %d: %s\n", deviceIdx, deviceProp.name);


  cublasCheck(hipblasCreate(&cublas_handle));
  cublasCheck(hipblasLtCreate(&cublaslt_handle));


  int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
  //printf("enable_tf32: %d\n", enable_tf32);
  cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
  hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
  cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
  // setup the (global) cuBLASLt workspace
  cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

  InitializeNativeTarget();
  InitializeNativeTargetAsmPrinter(); // Prepare for target hardware
  InitializeNativeTargetAsmParser();

  // Install standard binary operators.
  // 1 is lowest precedence.
  BinopPrecedence[tok_space] = 1;
  BinopPrecedence[':'] = 9;
  BinopPrecedence['='] = 4;
  BinopPrecedence['>'] = 10;
  BinopPrecedence['<'] = 10;
  BinopPrecedence['+'] = 20;
  BinopPrecedence['-'] = 20;
  BinopPrecedence['/'] = 39;
  BinopPrecedence['*'] = 40;  // highest.
  BinopPrecedence['^'] = 50;
  BinopPrecedence['@'] = 60;

  // Prime the first token.
  //fprintf(stderr, "ready> ");
  getNextToken();

  TheJIT = ExitOnErr(KaleidoscopeJIT::Create());
  InitializeModule();

  // Run the main "interpreter loop" now.
  MainLoop();

  return 0;
}


/*
//forward
hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, OC, B, C, &alpha, weight, C, inp,  C,  &beta, out,     OC);

//backward to input
hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, C, B, OC, &alpha, weight, C, dout, OC, &beta, dinp,     C)
//backward to weight
hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, C, OC, , &alpha, inp,    C, dout, OC, &beta, dweight,  C)
*/

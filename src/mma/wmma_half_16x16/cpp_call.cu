#include "../include.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>  // if using bfloat16

#include "wmma_blocking.h"
#include "cpp_call.h"

template<int WMMA_T, int WX, int WY>
void launch_kernel(Wmma_Grid grid, const float* x, const float* w, float* o,
                   int B, int C, int OC, hipStream_t stream) {
    wmma_blocking<WMMA_T, WX, WY, 32><<<grid.g, grid.w, grid.smem, stream>>>(
        x, w, o, B, C, OC, grid.bx, grid.by, grid.wx, grid.wy,
        grid.bx_per_w, grid.by_per_w,
        grid.bx_per_wx);
}




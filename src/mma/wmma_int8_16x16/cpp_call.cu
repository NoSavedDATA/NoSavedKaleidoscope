#include "../include.h"

#include "wmma_blocking_i8.h"
#include "cpp_call.h"

template<int WMMA_T, int WX, int WY>
void launch_kernel_i8(Wmma_Grid grid, const int8_t* x, const int8_t* w, float* o,
                   int B, int C, int OC, hipStream_t stream) {
    wmma_blocking_i8<WMMA_T, WX, WY, 128><<<grid.g, grid.w, grid.smem, stream>>>( // 128 because float has 4B and int8 has 1B
        x, w, o, B, C, OC, grid.bx, grid.by, grid.wx, grid.wy,
        grid.bx_per_w, grid.by_per_w,
        grid.bx_per_wx);
}




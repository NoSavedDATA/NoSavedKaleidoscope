
#include<string>
#include<vector>
#include<map>
#include<cstring>
#include<random>
#include<thread>
#include<cstdarg>

#include "../backprop/include.h"
#include "../common/include.h"
#include "../compiler_frontend/logging.h"
#include "../cuda_kernels/calculate_grids.h"
#include "../cuda_kernels/elementwise_kernels_inline.cu"
#include "../mangler/scope_struct.h"
#include "../tensor/include.h"
#include "include.h"




extern "C" DT_tensor *tensor_Create(Scope_Struct *scope_struct, char *tensor_name, char *scopeless_name, DT_tensor *init_val, DT_list *notes_vector)
{
  
  // if (notes_vector->data->size()>0)
  // {

  
    int thread_id = scope_struct->thread_id;
    // std::cout << "CREATING TENSOR " << tensor_name << " AT THREAD: " << thread_id << "\n";

    DT_tensor *tensor;


    std::vector<float> dims;
    char *init = "xavu";
    bool is_weight = false;
    for (int i=0; i<notes_vector->data->size(); i++)
    {
      if(notes_vector->data_types->at(i)=="float")
        dims.push_back(notes_vector->get<float>(i));
      if(notes_vector->data_types->at(i)=="str")
      {
        std::cout << "get char" << ".\n";
        char *note = notes_vector->get<char *>(i);
        if (std::strcmp(note,"param") == 0)
          is_weight = true;
        else
          init = note; 
        std::cout << "got char" << ".\n";
      }
    }

    
    int product = DimsProd(dims);

    float *tensor_ptr;
    float *tensor_cpu;

    if (init_val==nullptr)
    {

      if(product>0)
      {
        if (std::strcmp(init, "randu") == 0)
          tensor_cpu = make_random_float_uniform(product);
        if (std::strcmp(init, "zeros") == 0)
          tensor_cpu = make_zeros_float(product);
        if (std::strcmp(init, "ones") == 0)
          tensor_cpu = make_ones_float(product);
        if (std::strcmp(init, "normal") == 0)
          tensor_cpu = make_normal(product);
        if (std::strcmp(init, "xavu") == 0)
          tensor_cpu = make_xavier_uniform_float(product, dims[dims.size()-1], dims[dims.size()-2]);
        if (std::strcmp(init, "xavu_relu") == 0)
          tensor_cpu = make_xavier_uniform_float_relu(product, dims[dims.size()-1], dims[dims.size()-2]);
        if (std::strcmp(init, "xavu_tanh") == 0)
          tensor_cpu = make_xavier_uniform_float_tanh(product, dims[dims.size()-1], dims[dims.size()-2]);
        if (std::strcmp(init, "he_normal_relu") == 0)
          tensor_cpu = make_he_normal_float_relu(product, dims[dims.size()-1]);
        if (std::strcmp(init, "init_gpt") == 0)
          tensor_cpu = make_gpt_init(product);
        if (std::strcmp(init, "int") == 0)
          tensor_cpu = make_random_int(product, 10);
        if (std::strcmp(init, "arange") == 0)
          tensor_cpu = make_arange(product);
        if (std::strcmp(init, "binary") == 0)
          tensor_cpu = make_random_int(product, 1);

        cudaCheck(hipGetLastError());
        std::string _name = "create tensor ";
        _name = _name + tensor_name;
        tensor_ptr = get_from_pool(thread_id, product, _name);
        //std::cout << "cpy of: " << tensor_name << "\n";

        hipStream_t stream = ThreadsStream[thread_id];
        cudaCheck(hipMemcpyAsync(tensor_ptr, tensor_cpu, product*sizeof(float), hipMemcpyHostToDevice, stream));
        //hipStreamSynchronize(stream);
        delete[] tensor_cpu;
      }
    } else {
      int grid_size, block_size;
      CalculateGridAndBlockSizes(product, grid_size, block_size);
      hipStream_t stream = ThreadsStream[thread_id];
      copy_tensor_kernel<<<grid_size, block_size, 0, stream>>>(tensor_ptr, init_val->tensor_ptr, product);
    }
    
    

    tensor = createTensor(tensor_ptr, dims, product, true, tensor_name);
    tensor->scopeless_name = scopeless_name;
    if(is_weight)
      tensor->SetIsWeight();
    tensor->op = create_tensor_op;

   

    // }

  if(NamedTensorsT.count(tensor_name)>0)
  {
    DT_tensor *tensor_to_clean = NamedTensorsT[tensor_name];

    // if (tensor_to_clean->name=="batch_acc")
    // if (tensor_to_clean->name=="batch_acc"||tensor_to_clean->name=="y")
    // {
      // std::cout << "0000000000000000000000000000000000000CLEANING " << tensor_name << ".\n";
      move_to_pool(thread_id, tensor_to_clean->dims_prod, tensor_to_clean->tensor_ptr, "tensor_Create tensor substitution of " + tensor_to_clean->name + ".");
    // }
    // delete tensor_to_clean;
  }
    
  NamedTensorsT[tensor_name] = tensor;
  


  return tensor;
}








extern "C" DT_tensor *tensor_Load(Scope_Struct *scope_struct, char *tensor_name) {

  
  // std::cout << "\n\nLOAD TENSOR: " << tensor_name <<  "\n";
  DT_tensor *ret = NamedTensorsT[tensor_name];

  if(scope_struct->is_at_return && (nn_mode==eval_mode||scope_struct->thread_id!=0))
    ret->leaf = false;
  //std::cout << "return load." << "\n";

  return ret;
}


//todo: copy tensor
extern "C" DT_tensor *tensor_Copy(Scope_Struct *scope_struct, DT_tensor *tensor){

  int thread_id = scope_struct->thread_id;

  std::string tensor_name = tensor->name;
  
  std::string arg_tensor_name = "list_" + tensor_name;
  

  std::vector<float> dims = tensor->dims;
  int dims_prod = tensor->dims_prod;

  float *arg_tensor, *tensor_ptr;

  tensor_ptr = tensor->tensor_ptr;

  std::string where_from = "arg tensor of ";
  where_from = where_from + tensor_name;
  arg_tensor = get_from_pool(thread_id, dims_prod, where_from);
  
  
  if (dims_prod!=0)
  {
    int grid_size, block_size, shared_mem_size; 
    std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(tensor->dims_prod);
    grid_size = grid_block_mem_sizes[0];
    block_size = grid_block_mem_sizes[1];

    tensor->Sync();

    hipStream_t stream = ThreadsStream[thread_id];
    copy_tensor_kernel<<<grid_size,block_size,0,stream>>>(arg_tensor, tensor_ptr, dims_prod);
  }
  

  DT_tensor *new_tensor = createTensor(arg_tensor, dims, dims_prod, true, arg_tensor_name, tensor->hip_stream, tensor->loader);
  new_tensor->scopeless_name = tensor->scopeless_name;
  new_tensor->from_grad_or_load = tensor->from_grad_or_load;

  
  if(nn_mode==eval_mode)//
    to_free_tensor_forward(tensor, scope_struct->scope);//
  else
    to_free_tensor(tensor);
  // std::cout << "DT_tensor copied" << ".\n";

  return new_tensor;
}





inline DT_tensor *store_intermediate_result_tensor(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  // RHS does not need to be saved. So we just move the pointer to LHS
  // if(nn_mode==eval_mode||thread_id!=0)
  // {
  //   if(tensor->from_grad_or_load) //if(DoesTreeContainWeight(tensor)>0)
  //     ForwardCleanupToPool(tensor, scope);
  //   ForwardCleanupToPool(stored_tensor, scope);
  // }
  // else {
  DT_tensor *attr_tensor;
  if (has_grad==0)
      tensor->op = detach_op;
  attr_tensor = createBackward(stored_tensor->scopeless_name, tensor);
  todo_backward_tensors.push_back(attr_tensor);
  // } 

  std::string scopeless_name = stored_tensor->scopeless_name;
  stored_tensor = createTensor(tensor->tensor_ptr, tensor->dims, tensor->dims_prod, true, tensor_name, tensor->hip_stream, tensor->loader);
  stored_tensor->from_grad_or_load = tensor->from_grad_or_load;
  stored_tensor->scopeless_name = scopeless_name;

  return stored_tensor;
}


inline void clean_tensor(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  if (nn_mode==eval_mode||stored_tensor->thread_id!=0)
    CleanTreeNow(stored_tensor->thread_id, stored_tensor, stored_tensor->name);
  // Else, save the tensor for the backrpop.
}

inline DT_tensor *change_tensor_dims(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  stored_tensor->tensor_ptr = get_from_pool(thread_id, tensor->dims_prod, "z=x");
  stored_tensor->dims = tensor->dims;
  stored_tensor->dims_prod = tensor->dims_prod;
  
  return stored_tensor;
}



inline DT_tensor *sync_and_copy_tensors(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(tensor->dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];

  stored_tensor->Sync();
  tensor->Sync();
  
  hipStream_t stream = ThreadsStream[thread_id];
  copy_tensor_kernel<<<grid_size,block_size,0,stream>>>(stored_tensor->tensor_ptr, tensor->tensor_ptr, tensor->dims_prod);
  return stored_tensor;
}

inline DT_tensor *store_leaf_backward(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  if(nn_mode==training_mode&&thread_id==0)
  {
    DT_tensor *attribution_tensor;
  
    if (has_grad==0)
      tensor->op = detach_op;  
    attribution_tensor = createBackward(stored_tensor->scopeless_name, tensor);
    todo_backward_tensors.push_back(attribution_tensor);

    std::string scopeless_name = stored_tensor->scopeless_name;
    stored_tensor = createTensor(stored_tensor->tensor_ptr, tensor->dims, tensor->dims_prod, true, tensor_name, stored_tensor->hip_stream, stored_tensor->loader);
    
    stored_tensor->from_grad_or_load = tensor->from_grad_or_load;
    stored_tensor->scopeless_name = scopeless_name;
  }

  return stored_tensor;
}

extern "C" float tensor_Store(char *tensor_name, DT_tensor *tensor, Scope_Struct *scope_struct)
{
  // std::cout << "tensor_Store execution" << ".\n";


  char *scope = scope_struct->scope;
  int thread_id= scope_struct->thread_id;
  int has_grad = scope_struct->has_grad;


  DT_tensor *stored_tensor = NamedTensorsT[tensor_name];
  stored_tensor->is_last_version = false;
  
  // View op
  if (tensor->view_of == tensor_name)
  {
    stored_tensor->dims = tensor->dims;
    delete tensor;
  }
  // Non-leaf RHS.
  // Free current and point to operation result
  else if (tensor->name==""||!tensor->leaf) 
  {
    clean_tensor(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope); // Remove current if evaluating
    stored_tensor = store_intermediate_result_tensor(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope);
  }
  else {
   
    // Is Leaf
    if(tensor->op==tensor_leaf||tensor->op==create_tensor_op||nn_mode==eval_mode||thread_id!=0)
    {
      clean_tensor(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope); // Remove current if evaluating
      stored_tensor = change_tensor_dims(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope);
      stored_tensor = sync_and_copy_tensors(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope);
      stored_tensor = store_leaf_backward(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope);       
    } 
  }

  stored_tensor->thread_id = thread_id;
  stored_tensor->is_last_version = true;
  NamedTensorsT[tensor_name] = stored_tensor;
  cudaCheck(hipGetLastError());
  return 0;
}



void tensor_Clean_Up(void *data_ptr) {
  // std::cout << "tensor_Clean_Up" << ".\n";
}

extern "C" float tensor_opa(Scope_Struct *scope_struct, void *data_ptr) {
  std::cout << "opa tensor"  << ".\n";
  PrintDims(static_cast<DT_tensor *>(data_ptr)->dims);
  return 25;
}



extern "C" DT_tensor *gpu(Scope_Struct *scope_struct, DT_tensor *tensor, DT_tensor *pinned_tensor)
{
  //std::cout << "\nGpu transfer for: " << tensor.name << " on worker " << idx << "\n";
  int thread_id = scope_struct->thread_id; 
  float *tensor_ptr, *tensor_cpu;

  
  tensor_cpu = pinned_tensor->cpu_tensor_ptr;
  std::vector<float> dims = pinned_tensor->dims;
  float dims_prod = pinned_tensor->dims_prod;
  



  
  if (tensor->dims_prod==dims_prod)
    tensor_ptr = tensor->tensor_ptr;
  else
    tensor_ptr = get_from_pool(thread_id, dims_prod, "gpu");
  
  //tensor_ptr = get_from_pool(dims_prod, "gpu");


  
  Loader *loader=nullptr;
  hipStream_t hip_stream = createCudaStream();
  hipMemcpyAsync(tensor_ptr, tensor_cpu, dims_prod * sizeof(float), hipMemcpyHostToDevice, hip_stream);
  //hipMemcpy(tensor_ptr, tensor_cpu, dims_prod * sizeof(float), hipMemcpyHostToDevice);
  pinned_tensor->hip_stream = hip_stream;
  



  if (nn_mode==eval_mode)
  {

  } else {
    
    DT_tensor *attr_tensor;
    attr_tensor = createTensor(tensor_ptr, dims, dims_prod, true, "");
    attr_tensor->op = gpu_op;
    todo_backward_tensors.push_back(attr_tensor); // pass to gc
    
  }

  tensor->AttrTensor(tensor_ptr, dims, dims_prod, hip_stream, loader);
  tensor->from_grad_or_load = true;

  return 0;
}



extern "C" float tensor_gpuw(Scope_Struct *scope_struct, DT_tensor *tensor, DT_tensor *pinned_tensor, float idx)
{
  int thread_id = scope_struct->thread_id;

  // std::cout << "\nGpu transfer for: " << tensor->name << " on worker " << idx << " and thread id: " << thread_id << "\n";

  float *tensor_ptr, *tensor_cpu;

  
  
  std::vector<float> dims, batchless_dims;
  dims = pinned_tensor->dims;
  

  batchless_dims = BatchLessDims(dims);
  float batchless_dims_prod = (float)DimsProd(batchless_dims);


  tensor_cpu = pinned_tensor->cpu_tensor_ptr + static_cast<int>(idx*batchless_dims_prod);

  
  if (tensor->dims_prod==batchless_dims_prod)
    tensor_ptr = tensor->tensor_ptr;
  else
    tensor_ptr = get_from_pool(thread_id, batchless_dims_prod, "gpuw");
  
  //tensor_ptr = get_from_pool(batchless_dims_prod, "gpuw");


  
  Loader *loader=nullptr;
  hipStream_t hip_stream = nullptr;
  
  if (batchless_dims_prod<2000){
    hipMemcpy(tensor_ptr, tensor_cpu, batchless_dims_prod * sizeof(float), hipMemcpyHostToDevice);
  }
  else// if (batchless_dims_prod<1000)
  {
    hip_stream = createCudaStream();
    hipMemcpyAsync(tensor_ptr, tensor_cpu, batchless_dims_prod * sizeof(float), hipMemcpyHostToDevice, hip_stream);
    //hipMemcpy(tensor_ptr, tensor_cpu, batchless_dims_prod * sizeof(float), hipMemcpyHostToDevice);
    pinned_tensor->hip_stream = hip_stream;
  }
  /*
  else
  {
    //hip_stream = AllocateStream(0);
    //hipMemcpyAsync(tensor_ptr, tensor_cpu, batchless_dims_prod * sizeof(float), hipMemcpyHostToDevice, hip_stream);
    loader = new Loader();
    loader->Load(tensor_ptr, tensor_cpu, batchless_dims_prod);
  }
  */



  if (nn_mode==eval_mode||thread_id!=0)
  {

  } else {
    
    DT_tensor *attr_tensor;
    attr_tensor = createTensor(tensor_ptr, batchless_dims, batchless_dims_prod, true, "");
    attr_tensor->op = gpu_op;
    todo_backward_tensors.push_back(attr_tensor); // pass to gc
    
  }

  tensor->AttrTensor(tensor_ptr, batchless_dims, batchless_dims_prod, hip_stream, loader);
  tensor->from_grad_or_load = true;
  tensor->leaf=true;


  return 0;
}


extern "C" float cpu(Scope_Struct *scope_struct, DT_tensor *tensor)
{

  int thread_id = scope_struct->thread_id; 

  float *tensor_ptr, *tensor_cpu;
  tensor_ptr = tensor->tensor_ptr;
  tensor_cpu = tensor->cpu_tensor_ptr;

  hipStream_t stream = ThreadsStream[thread_id];
  hipStreamSynchronize(stream);

  if (tensor_ptr==nullptr)
    LogErrorS("Cannot load tensor to cpu from an null tensor.");

  if (tensor_cpu!=nullptr)
    cudaCheck(hipFree(tensor_cpu));

  float dims_prod = tensor->dims_prod;



  hipHostMalloc(&tensor_cpu, dims_prod*sizeof(float));
  hipMemcpy(tensor_cpu, tensor_ptr, dims_prod*sizeof(float), hipMemcpyDeviceToHost);

  tensor->cpu_tensor_ptr = tensor_cpu;


  return 0;
}

extern "C" float cpu_idx(Scope_Struct *scope_struct, DT_tensor *tensor, float idx)
{

  float *tensor_cpu;
  tensor_cpu = tensor->cpu_tensor_ptr;


  if (tensor_cpu==nullptr)
    LogErrorS("Cannot idx a null cpu tensor.");

  float dims_prod = tensor->dims_prod;
  if (idx>dims_prod)
    LogErrorS("Idx higher than dims prod at cpu_idx().");

  

  return tensor_cpu[(int)idx];
}


extern "C" DT_tensor *randu_like(Scope_Struct *scope_struct, DT_tensor tensor)
{
  int thread_id = scope_struct->thread_id;

  float dims_prod = tensor.dims_prod;

  float *tensor_ptr, *tensor_cpu;

  tensor_cpu = make_random_float_uniform(dims_prod);

  hipStream_t stream = ThreadsStream[thread_id];
  hipMalloc(&tensor_ptr, dims_prod*sizeof(float));
  hipMemcpyAsync(tensor_ptr, tensor_cpu, dims_prod*sizeof(float), hipMemcpyHostToDevice, stream);
  delete[] tensor_cpu;

  DT_tensor *new_tensor = createTensor(tensor_ptr, tensor.dims, dims_prod, false, "");
  new_tensor->op = randu_like_op;
  return new_tensor;
}



void copyChunk(float* d_data, const float* h_data, int offset, float size, hipStream_t stream) {
  hipMemcpyAsync(d_data + offset, h_data + offset, size*sizeof(float), hipMemcpyHostToDevice, stream);
}


extern "C" float write_zerosw(DT_tensor *tensor, float worker_idx)
{
  std::vector<float> dims = tensor->dims;

  std::vector<float> workerless_dims = BatchLessDims(dims);
  int workerless_dims_prod = DimsProd(workerless_dims);

  int idx_offset = (int) (workerless_dims_prod*worker_idx);

  for(int i=0; i<workerless_dims_prod; i++)
    tensor->cpu_tensor_ptr[i+idx_offset] = 0.0f;
  
  return 0;
}


extern "C" DT_tensor *tensor_view(Scope_Struct *scope_struct, DT_tensor *tensor, float first_dim, ...)
{

  //std::cout << "Executing: " << tensor.name << "." << "view" << "\n";
   
  std::vector<float> new_dims, new_dims_no_minus, current_dims;
  bool has_minus = false;
  current_dims = tensor->dims;

  
  va_list args;
  va_start(args, first_dim);

  if (first_dim!=-1)
    new_dims_no_minus.push_back(first_dim);
  else
    has_minus=true;
  
  
  new_dims.push_back(first_dim);

  for (int i=0; i<10; i++)
  {
    if (i==9)
    {
      LogErrorS("A tensor with 10 dimensions??? (view)");
      return 0;
    }

    float dim = va_arg(args, float);
    if (dim==TERMINATE_VARARG)
      break;
    new_dims.push_back(dim);

    if (dim!=-1)
      new_dims_no_minus.push_back(dim);
    else
      has_minus=true;
  }
  va_end(args);


  


  int current_dims_prod = DimsProd(current_dims);
  int new_dims_prod = DimsProd(new_dims);


  if (has_minus)
  {
    float hidden_dim = (float)current_dims_prod / (float)DimsProd(new_dims_no_minus);

    if ((float)((int)hidden_dim) != hidden_dim)
    {
      LogErrorS("Automatic view dimension calculus resulted on a non-integer dimension.");
      PrintDims(current_dims);
      std::cout << "Current dims product: " << current_dims_prod  << ".\n";
      PrintDims(new_dims);
      std::cout << "New dims product: " << std::to_string(DimsProd(new_dims_no_minus))  << ".\n";
      return 0;
    }
    
    for (int i=0; i<new_dims.size(); i++)
      if (new_dims[i]==-1)
        new_dims[i] = hidden_dim;
    
  } else {
    if (current_dims_prod != new_dims_prod)
    {
      LogErrorS("Incompatible view dimensions.");
      PrintDims(current_dims);
      std::cout << "Current dims product: " << current_dims_prod  << ".\n";
      PrintDims(new_dims);
      std::cout << "New dims product: " << new_dims_prod  << ".\n";
      return 0;
    }
  }

  

  DT_tensor *new_tensor = createTensor(tensor->tensor_ptr, new_dims, DimsProd(new_dims), false, "");
  new_tensor->view_of = tensor->name;
  new_tensor->op=view_op;
  return new_tensor;
}



extern "C" DT_tensor *NewVecToTensor(Scope_Struct *scope_struct, float first_dim, ...)
{
  std::vector<float> values;

  int thread_id = scope_struct->thread_id;
  
  va_list args;
  va_start(args, first_dim);


  values.push_back(first_dim);

  for (int i=0; i<10; i++)
  {
    if (i==9)
    {
      LogErrorS("Tried to create a tensor from brackets with more than 10 positions. This is not yet supported");
      return nullptr;
    }

    float dim = va_arg(args, float);
    if (dim==TERMINATE_VARARG)
      break;
    values.push_back(dim);


  }
  va_end(args);


  float dims_prod = values.size();

  float *tensor_ptr, *tensor_cpu;
  tensor_cpu = values.data();

  tensor_ptr = get_from_pool(thread_id, dims_prod, "tensor from brackets");
  hipMemcpy(tensor_ptr, tensor_cpu, dims_prod*sizeof(float), hipMemcpyHostToDevice);
  

  DT_tensor *new_tensor = createTensor(tensor_ptr, {dims_prod}, dims_prod, true, "");
  new_tensor->op=create_tensor_from_brackets_op;
  return new_tensor;
}




extern "C" float tensor_CalculateIdx(char *tensor_name, float first_idx, ...) {
  
  // std::cout << "pinned_tensor_CalculateIdx of " << tensor_name << "\n";

  DT_tensor *tensor = NamedTensorsT[tensor_name];

  std::vector<float> idxs, new_dims_no_minus, dims;
  int current_dims_prod;
  bool has_minus = false;
  dims = tensor->dims;

  int idx_at = 0;

  
  va_list args;
  va_start(args, first_idx);

  if (first_idx!=-1)
    new_dims_no_minus.push_back(first_idx);
  else
    has_minus=true;
  
    
  idxs.push_back(first_idx);

  dims = RemoveFirstDim(dims);
  
  current_dims_prod = DimsProd(dims);

  idx_at += (int)(current_dims_prod*first_idx);



  //std::cout << "Get idx of " << tensor_name << "\nCalculateIdxOffset pushing dim: " << first_idx << "\n";

  for (int i=0; i<10; i++)
  {
    if (i==9)
    {
      LogErrorS("A tensor with 10 dimensions??? (calc idx)");
      return 0;
    }

    float idx = va_arg(args, float);
    if (idx==TERMINATE_VARARG)
      break;

    idxs.push_back(idx);
    
    dims = RemoveFirstDim(dims);
    
    current_dims_prod = DimsProd(dims);

    idx_at += (int)(current_dims_prod*idx);

    //std::cout << "CalculateIdxOffset pushing dim: " << idx << "\n";
    

    if (idx!=-1)
      new_dims_no_minus.push_back(idx);
    else
      has_minus=true;
  }
  va_end(args);



  return idx_at;
}




extern "C" DT_tensor *zeros_like(Scope_Struct *scope_struct, DT_tensor *tensor) {

  // DT_tensor *zeros_tensor = new DT_tensor();

  int thread_id = scope_struct->thread_id;
  float *tensor_ptr = tensor->tensor_ptr;
  std::vector<float> dims = tensor->dims;
  float dims_prod = tensor->dims_prod;

  int grid_size, block_size; 
  CalculateGridAndBlockSizes(dims_prod, grid_size, block_size);
  

  float *y = get_from_pool(thread_id, dims_prod, "relu");


  tensor->Sync();
  hipStream_t stream = ThreadsStream[thread_id];
  set_to_zero_kernel<<<grid_size, block_size, 0, stream>>>(y, dims_prod);

  return customOpTensor(y, dims, DimsProd(dims), "set_to_zero", "", tensor);
}
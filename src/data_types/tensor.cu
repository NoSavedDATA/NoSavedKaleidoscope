
#include<string>
#include<vector>
#include<map>
#include<cstring>
#include<random>
#include<thread>
#include<cstdarg>

#include "../backprop/include.h"
#include "../common/include.h"
#include "../compiler_frontend/logging.h"
#include "../cuda_kernels/calculate_grids.h"
#include "../cuda_kernels/elementwise_kernels_inline.cu"
#include "../mangler/scope_struct.h"
#include "../nsk_cuda/pool/include.h"
#include "../tensor/include.h"
#include "include.h"




extern "C" DT_tensor *tensor_Create(Scope_Struct *scope_struct, char *tensor_name, char *scopeless_name, DT_tensor *init_val, DT_list *notes_vector)
{
  
  // if (notes_vector->data->size()>0)
  // {

  
    int thread_id = scope_struct->thread_id;
    // std::cout << "CREATING TENSOR " << tensor_name << " AT THREAD: " << thread_id << "\n";

    DT_tensor *tensor;


    std::vector<int> dims;
    char *init = "xavu";
    bool is_weight = false;
    for (int i=0; i<notes_vector->data->size(); i++)
    {
      if(notes_vector->data_types->at(i)=="int")
        dims.push_back(notes_vector->get<int>(i));
      if(notes_vector->data_types->at(i)=="str")
      {
        std::cout << "get char" << ".\n";
        char *note = notes_vector->get<char *>(i);
        if (std::strcmp(note,"param") == 0)
          is_weight = true;
        else
          init = note; 
        std::cout << "got char" << ".\n";
      }
    }

    
    int product = DimsProd(dims);

    float *tensor_ptr;
    float *tensor_cpu;

    if (init_val==nullptr)
    {

      if(product>0)
      {
        if (std::strcmp(init, "randu") == 0)
          tensor_cpu = make_random_float_uniform(product);
        if (std::strcmp(init, "zeros") == 0)
          tensor_cpu = make_zeros_float(product);
        if (std::strcmp(init, "ones") == 0)
          tensor_cpu = make_ones_float(product);
        if (std::strcmp(init, "normal") == 0)
          tensor_cpu = make_normal(product);
        if (std::strcmp(init, "xavu") == 0)
          tensor_cpu = make_xavier_uniform_float(product, dims[dims.size()-1], dims[dims.size()-2]);
        if (std::strcmp(init, "fixed8i") == 0)
          tensor_cpu = make_xavier_uniform_float_fixed(product, dims[dims.size()-1], dims[dims.size()-2], 8);
        if (std::strcmp(init, "fixed42i") == 0)
          tensor_cpu = make_xavier_uniform_float_fixed(product, dims[dims.size()-1], dims[dims.size()-2], 42);
        if (std::strcmp(init, "xavu_relu") == 0)
          tensor_cpu = make_xavier_uniform_float_relu(product, dims[dims.size()-1], dims[dims.size()-2]);
        if (std::strcmp(init, "xavu_tanh") == 0)
          tensor_cpu = make_xavier_uniform_float_tanh(product, dims[dims.size()-1], dims[dims.size()-2]);
        if (std::strcmp(init, "he_normal_relu") == 0)
          tensor_cpu = make_he_normal_float_relu(product, dims[dims.size()-1]);
        if (std::strcmp(init, "init_gpt") == 0)
          tensor_cpu = make_gpt_init(product);
        if (std::strcmp(init, "ints") == 0)
          tensor_cpu = make_random_int(product, 10);
        if (std::strcmp(init, "arange") == 0)
          tensor_cpu = make_arange(product);
        if (std::strcmp(init, "binary") == 0)
          tensor_cpu = make_random_int(product, 1);

        cudaCheck(hipGetLastError());
        std::string _name = "create tensor ";
        _name = _name + tensor_name;
        tensor_ptr = get_from_pool(thread_id, product, _name);
        //std::cout << "cpy of: " << tensor_name << "\n";

        hipStream_t stream = ThreadsStream[thread_id];
        cudaCheck(hipMemcpyAsync(tensor_ptr, tensor_cpu, product*sizeof(float), hipMemcpyHostToDevice, stream));
        //hipStreamSynchronize(stream);
        delete[] tensor_cpu;
      }
    } else {
      int grid_size, block_size;
      CalculateGridAndBlockSizes(product, grid_size, block_size);
      hipStream_t stream = ThreadsStream[thread_id];
      copy_tensor_kernel<<<grid_size, block_size, 0, stream>>>(tensor_ptr, init_val->tensor_ptr, product);
    }
    
    

    tensor = createTensor(tensor_ptr, dims, product, true, tensor_name);
    tensor->scopeless_name = scopeless_name;
    if(is_weight)
      tensor->SetIsWeight();
    tensor->op = create_tensor_op;

   

    // }

  if(NamedTensorsT.count(tensor_name)>0)
  {
    DT_tensor *tensor_to_clean = NamedTensorsT[tensor_name];

    // if (tensor_to_clean->name=="batch_acc")
    // if (tensor_to_clean->name=="batch_acc"||tensor_to_clean->name=="y")
    // {
      // std::cout << "0000000000000000000000000000000000000CLEANING " << tensor_name << ".\n";
      move_to_pool(thread_id, tensor_to_clean->dims_prod, tensor_to_clean->tensor_ptr, "tensor_Create tensor substitution of " + tensor_to_clean->name + ".");
    // }
    // delete tensor_to_clean;
  }
    


  // NamedTensorsT[tensor_name] = tensor;
  


  return tensor;
}








extern "C" DT_tensor *tensor_Load(Scope_Struct *scope_struct, char *tensor_name) {

  
  // std::cout << "\n\nLOAD TENSOR: " << tensor_name <<  "\n";
  DT_tensor *ret = NamedTensorsT[tensor_name];

  if(scope_struct->is_at_return && (nn_mode==eval_mode||scope_struct->thread_id!=0))
    ret->leaf = false; // Marks to clean

  return ret;
}


extern "C" DT_tensor *tensor_Copy(Scope_Struct *scope_struct, DT_tensor *tensor) {
  std::cout << "-------**tensor_Copy" <<  ".\n";

  int thread_id = scope_struct->thread_id;

  std::string tensor_name = tensor->name;
  
  // std::string arg_tensor_name = "list_" + tensor_name;
  std::string arg_tensor_name = tensor_name;
  

  std::vector<int> dims = tensor->dims;
  int dims_prod = tensor->dims_prod;

  float *arg_tensor, *tensor_ptr;

  tensor_ptr = tensor->tensor_ptr;

  std::string where_from = "arg tensor of ";
  where_from = where_from + tensor_name;
  arg_tensor = get_from_pool(thread_id, dims_prod, where_from);
  
  
  if (dims_prod!=0)
  {
    int grid_size, block_size;
    CalculateGridAndBlockSizes(tensor->dims_prod, grid_size, block_size);

    tensor->Sync();

    hipStream_t stream = ThreadsStream[thread_id];
    copy_tensor_kernel<<<grid_size,block_size,0,stream>>>(arg_tensor, tensor_ptr, dims_prod);
  }
  

  DT_tensor *new_tensor = createTensor(arg_tensor, dims, dims_prod, true, arg_tensor_name, tensor->hip_stream, tensor->loader);
  new_tensor->scopeless_name = tensor->scopeless_name;

  
  if(nn_mode==eval_mode)//
    to_free_tensor_forward(tensor, scope_struct->scope);//
  else
    to_free_tensor(tensor);
  // std::cout << "DT_tensor copied" << ".\n";

  return new_tensor;
}





inline void create_backward_tensor(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  // RHS does not need to be saved. So we just move the pointer to LHS
  // if(nn_mode==eval_mode||thread_id!=0)
  // {
  //   if(tensor->from_grad_or_load) //if(DoesTreeContainWeight(tensor)>0)
  //     ForwardCleanupToPool(tensor, scope);
  //   ForwardCleanupToPool(stored_tensor, scope);
  // }
  // else {

  DT_tensor *attr_tensor;
  if (has_grad==0)
      tensor->op = detach_op;
  attr_tensor = createBackward(stored_tensor->scopeless_name, tensor);
  todo_backward_tensors.push_back(attr_tensor);
  // } 
}


inline void clean_tensor(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  if (nn_mode==eval_mode||stored_tensor->thread_id!=0)
    CleanTreeNow(stored_tensor->thread_id, stored_tensor, stored_tensor->name);
  // Else, save the tensor for the backrpop.
}

inline DT_tensor *change_tensor_dims(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  stored_tensor->tensor_ptr = get_from_pool(thread_id, tensor->dims_prod, "z=x");
  stored_tensor->dims = tensor->dims;
  stored_tensor->dims_prod = tensor->dims_prod;
  
  return stored_tensor;
}



inline DT_tensor *sync_and_copy_tensors(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  int grid_size, block_size; 
  CalculateGridAndBlockSizes(tensor->dims_prod, grid_size, block_size);

  stored_tensor->Sync();
  tensor->Sync();
  
  hipStream_t stream = ThreadsStream[thread_id];
  copy_tensor_kernel<<<grid_size,block_size,0,stream>>>(stored_tensor->tensor_ptr, tensor->tensor_ptr, tensor->dims_prod);
  return stored_tensor;
}

inline DT_tensor *store_leaf_backward(DT_tensor *stored_tensor, DT_tensor *tensor, char *tensor_name, int thread_id, int has_grad, char *scope) {
  if(nn_mode==training_mode&&thread_id==0)
  {
    DT_tensor *attribution_tensor;
  
    if (has_grad==0)
      tensor->op = detach_op;  
    attribution_tensor = createBackward(stored_tensor->scopeless_name, tensor);
    todo_backward_tensors.push_back(attribution_tensor);

    std::string scopeless_name = stored_tensor->scopeless_name;
    stored_tensor = createTensor(stored_tensor->tensor_ptr, tensor->dims, tensor->dims_prod, true, tensor_name, stored_tensor->hip_stream, stored_tensor->loader);
    
    stored_tensor->scopeless_name = scopeless_name;
  }

  return stored_tensor;
}

extern "C" float tensor_StoreTrigger(char *tensor_name, DT_tensor *stored_tensor, DT_tensor *tensor, Scope_Struct *scope_struct)
{
  // std::cout << "tensor_Store execution for " << tensor_name << ".\n";

  std::string scopeless_name = stored_tensor->scopeless_name;



  char *scope = scope_struct->scope;
  int thread_id= scope_struct->thread_id;
  int has_grad = scope_struct->has_grad;


  // DT_tensor *stored_tensor = NamedTensorsT[tensor_name];
  stored_tensor->is_last_version = false;
  
  // View op
  if (tensor->view_of == tensor_name)
  {
    delete stored_tensor;
  }
  // Non-leaf RHS.
  // Free current and point to operation result
  else if (tensor->name==""||!tensor->leaf) 
  {
    clean_tensor(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope); // Remove current if evaluating
    create_backward_tensor(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope);
  }
  else {

   
    // Is Leaf
    if(tensor->op==tensor_leaf||tensor->op==create_tensor_op||nn_mode==eval_mode||thread_id!=0)
    {
      clean_tensor(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope); // Remove current if evaluating
      // stored_tensor = store_leaf_backward(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope);       
      create_backward_tensor(stored_tensor, tensor, tensor_name, thread_id, has_grad, scope);
    } 
  }


  tensor->name = tensor_name;
  tensor->scopeless_name = scopeless_name;


  tensor->thread_id = thread_id;
  tensor->is_last_version = true;
  
  // NamedTensorsT[tensor_name] = stored_tensor;
  cudaCheck(hipGetLastError());
  return 0;
}



void tensor_Clean_Up(void *data_ptr) {
  // std::cout << "tensor_Clean_Up" << ".\n";
}




extern "C" DT_tensor *gpu(Scope_Struct *scope_struct, DT_tensor *tensor, DT_tensor *pinned_tensor)
{
  //std::cout << "\nGpu transfer for: " << tensor.name << " on worker " << idx << "\n";
  int thread_id = scope_struct->thread_id; 
  float *tensor_ptr, *tensor_cpu;

  
  tensor_cpu = pinned_tensor->cpu_tensor_ptr;
  std::vector<int> dims = pinned_tensor->dims;
  float dims_prod = pinned_tensor->dims_prod;
  



  
  if (tensor->dims_prod==dims_prod)
    tensor_ptr = tensor->tensor_ptr;
  else
    tensor_ptr = get_from_pool(thread_id, dims_prod, "gpu");
  
  //tensor_ptr = get_from_pool(dims_prod, "gpu");


  
  Loader *loader=nullptr;
  hipStream_t hip_stream = createCudaStream();
  hipMemcpyAsync(tensor_ptr, tensor_cpu, dims_prod * sizeof(float), hipMemcpyHostToDevice, hip_stream);
  //hipMemcpy(tensor_ptr, tensor_cpu, dims_prod * sizeof(float), hipMemcpyHostToDevice);
  pinned_tensor->hip_stream = hip_stream;
  



  if (nn_mode==eval_mode)
  {

  } else {
    
    DT_tensor *attr_tensor;
    attr_tensor = createTensor(tensor_ptr, dims, dims_prod, true, "");
    attr_tensor->op = gpu_op;
    todo_backward_tensors.push_back(attr_tensor); // pass to gc
    
  }

  tensor->AttrTensor(tensor_ptr, dims, dims_prod, hip_stream, loader);

  return 0;
}



extern "C" float tensor_gpuw(Scope_Struct *scope_struct, DT_tensor *tensor, DT_tensor *pinned_tensor, int idx)
{
  int thread_id = scope_struct->thread_id;

  // std::cout << "\nGpu transfer for: " << tensor->name << " on worker " << idx << " and thread id: " << thread_id << "\n";

  float *tensor_ptr, *tensor_cpu;

  
  
  std::vector<int> dims, batchless_dims;
  dims = pinned_tensor->dims;
  

  batchless_dims = BatchLessDims(dims);
  float batchless_dims_prod = (float)DimsProd(batchless_dims);


  tensor_cpu = pinned_tensor->cpu_tensor_ptr + static_cast<int>(idx*batchless_dims_prod);

  
  if (tensor->dims_prod==batchless_dims_prod)
    tensor_ptr = tensor->tensor_ptr;
  else
    tensor_ptr = get_from_pool(thread_id, batchless_dims_prod, "gpuw");
  
  //tensor_ptr = get_from_pool(batchless_dims_prod, "gpuw");


  
  Loader *loader=nullptr;
  hipStream_t hip_stream = nullptr;
  
  if (batchless_dims_prod<2000){
    hipMemcpy(tensor_ptr, tensor_cpu, batchless_dims_prod * sizeof(float), hipMemcpyHostToDevice);
  }
  else// if (batchless_dims_prod<1000)
  {
    hip_stream = createCudaStream();
    // hipMemcpyAsync(tensor_ptr, tensor_cpu, batchless_dims_prod * sizeof(float), hipMemcpyHostToDevice, hip_stream);
    hipMemcpy(tensor_ptr, tensor_cpu, batchless_dims_prod * sizeof(float), hipMemcpyHostToDevice);
    pinned_tensor->hip_stream = hip_stream;
  }
  /*
  else
  {
    //hip_stream = AllocateStream(0);
    //hipMemcpyAsync(tensor_ptr, tensor_cpu, batchless_dims_prod * sizeof(float), hipMemcpyHostToDevice, hip_stream);
    loader = new Loader();
    loader->Load(tensor_ptr, tensor_cpu, batchless_dims_prod);
  }
  */



  if (nn_mode==eval_mode||thread_id!=0)
  {

  } else {
    
    DT_tensor *attr_tensor;
    attr_tensor = createTensor(tensor_ptr, batchless_dims, batchless_dims_prod, true, "");
    attr_tensor->op = gpu_op;
    todo_backward_tensors.push_back(attr_tensor); // pass to gc
    
  }

  tensor->AttrTensor(tensor_ptr, batchless_dims, batchless_dims_prod, hip_stream, loader);
  tensor->leaf=true;


  return 0;
}


extern "C" float cpu(Scope_Struct *scope_struct, DT_tensor *tensor)
{

  int thread_id = scope_struct->thread_id; 

  float *tensor_ptr, *tensor_cpu;
  tensor_ptr = tensor->tensor_ptr;
  tensor_cpu = tensor->cpu_tensor_ptr;

  hipStream_t stream = ThreadsStream[thread_id];
  hipStreamSynchronize(stream);

  if (tensor_ptr==nullptr)
    LogErrorS("Cannot load tensor to cpu from an null tensor.");

  if (tensor_cpu!=nullptr)
    cudaCheck(hipFree(tensor_cpu));

  float dims_prod = tensor->dims_prod;



  hipHostMalloc(&tensor_cpu, round_to_nearest_pow2(dims_prod)*sizeof(float));
  hipMemcpy(tensor_cpu, tensor_ptr, dims_prod*sizeof(float), hipMemcpyDeviceToHost);

  tensor->cpu_tensor_ptr = tensor_cpu;


  return 0;
}

extern "C" float cpu_idx(Scope_Struct *scope_struct, DT_tensor *tensor, float idx)
{

  float *tensor_cpu;
  tensor_cpu = tensor->cpu_tensor_ptr;


  if (tensor_cpu==nullptr)
    LogErrorS("Cannot idx a null cpu tensor.");

  float dims_prod = tensor->dims_prod;
  if (idx>dims_prod)
    LogErrorS("Idx higher than dims prod at cpu_idx().");

  

  return tensor_cpu[(int)idx];
}


extern "C" DT_tensor *randu_like(Scope_Struct *scope_struct, DT_tensor tensor)
{
  int thread_id = scope_struct->thread_id;

  float dims_prod = tensor.dims_prod;

  float *tensor_ptr, *tensor_cpu;

  tensor_cpu = make_random_float_uniform(dims_prod);

  hipStream_t stream = ThreadsStream[thread_id];
  hipMalloc(&tensor_ptr, round_to_nearest_pow2(dims_prod)*sizeof(float));
  hipMemcpyAsync(tensor_ptr, tensor_cpu, dims_prod*sizeof(float), hipMemcpyHostToDevice, stream);
  delete[] tensor_cpu;

  DT_tensor *new_tensor = createTensor(tensor_ptr, tensor.dims, dims_prod, false, "");
  new_tensor->op = randu_like_op;
  return new_tensor;
}



void copyChunk(float* d_data, const float* h_data, int offset, float size, hipStream_t stream) {
  hipMemcpyAsync(d_data + offset, h_data + offset, size*sizeof(float), hipMemcpyHostToDevice, stream);
}


extern "C" float write_zerosw(Scope_Struct *scope_struct, DT_tensor *tensor, int worker_idx)
{
  std::vector<int> dims = tensor->dims;

  std::vector<int> workerless_dims = BatchLessDims(dims);
  int workerless_dims_prod = DimsProd(workerless_dims);

  int idx_offset =  workerless_dims_prod*worker_idx;

  for(int i=0; i<workerless_dims_prod; i++)
    tensor->cpu_tensor_ptr[i+idx_offset] = 0.0f;
  
  return 0;
}




extern "C" DT_tensor *tensor_view(Scope_Struct *scope_struct, DT_tensor *tensor, int first_dim, ...)
{

  // std::cout << "Executing: " << tensor->name << "." << "view" << "\n";
   
  std::vector<int> new_dims, new_dims_no_minus, current_dims;
  bool has_minus = false;
  current_dims = tensor->dims;

  
  va_list args;
  va_start(args, first_dim);


  if (first_dim!=-1)
    new_dims_no_minus.push_back(first_dim);
  else
    has_minus=true;
  
  
  new_dims.push_back(first_dim);

  for (int i=0; i<10; i++)
  {
    if (i==9)
    {
      LogErrorS("A tensor with 10 dimensions??? (view)");
      return 0;
    }

    int dim = va_arg(args, int);
    if (dim==TERMINATE_VARARG)
      break;
    new_dims.push_back(dim);

    if (dim!=-1)
      new_dims_no_minus.push_back(dim);
    else
      has_minus=true;
  }
  va_end(args);


  


  int current_dims_prod = DimsProd(current_dims);
  int new_dims_prod = DimsProd(new_dims);


  if (has_minus)
  {
    float hidden_dim = (float)current_dims_prod / (float)DimsProd(new_dims_no_minus);

    if ((float)((int)hidden_dim) != hidden_dim)
    {
      LogErrorS("Automatic view dimension calculus resulted on a non-integer dimension.");
      PrintDims(current_dims);
      std::cout << "Current dims product: " << current_dims_prod  << ".\n";
      PrintDims(new_dims);
      std::cout << "New dims product: " << std::to_string(DimsProd(new_dims_no_minus))  << ".\n";
      return 0;
    }
    
    for (int i=0; i<new_dims.size(); i++)
      if (new_dims[i]==-1)
        new_dims[i] = hidden_dim;
    
  } else {
    if (current_dims_prod != new_dims_prod)
    {
      LogErrorS("Incompatible view dimensions.");
      PrintDims(current_dims);
      std::cout << "Current dims product: " << current_dims_prod  << ".\n";
      PrintDims(new_dims);
      std::cout << "New dims product: " << new_dims_prod  << ".\n";
      return 0;
    }
  }

  

  DT_tensor *new_tensor = createTensor(tensor->tensor_ptr, new_dims, DimsProd(new_dims), false, "");
  new_tensor->view_of = tensor->name;
  new_tensor->op=view_op;
  return new_tensor;
}








extern "C" int tensor_CalculateIdx(char *tensor_name, int first_idx, ...) {
  
  // std::cout << "pinned_tensor_CalculateIdx of " << tensor_name << "\n";

  DT_tensor *tensor = NamedTensorsT[tensor_name];

  std::vector<int> idxs, new_dims_no_minus, dims;
  int current_dims_prod;
  bool has_minus = false;
  dims = tensor->dims;

  int idx_at = 0;

  
  va_list args;
  va_start(args, first_idx);

  if (first_idx!=-1)
    new_dims_no_minus.push_back(first_idx);
  else
    has_minus=true;
  
    
  idxs.push_back(first_idx);

  dims = RemoveFirstDim(dims);
  
  current_dims_prod = DimsProd(dims);

  idx_at += (int)(current_dims_prod*first_idx);



  //std::cout << "Get idx of " << tensor_name << "\nCalculateIdxOffset pushing dim: " << first_idx << "\n";

  for (int i=0; i<10; i++)
  {
    if (i==9)
    {
      LogErrorS("A tensor with 10 dimensions??? (calc idx)");
      return 0;
    }

    int idx = va_arg(args, int);
    if (idx==TERMINATE_VARARG)
      break;

    idxs.push_back(idx);
    
    dims = RemoveFirstDim(dims);
    
    current_dims_prod = DimsProd(dims);

    idx_at += current_dims_prod*idx;

    //std::cout << "CalculateIdxOffset pushing dim: " << idx << "\n";
    

    if (idx!=-1)
      new_dims_no_minus.push_back(idx);
    else
      has_minus=true;
  }
  va_end(args);



  return idx_at;
}




extern "C" DT_tensor *zeros_like(Scope_Struct *scope_struct, DT_tensor *tensor) {

  // DT_tensor *zeros_tensor = new DT_tensor();

  int thread_id = scope_struct->thread_id;
  float *tensor_ptr = tensor->tensor_ptr;
  std::vector<int> dims = tensor->dims;
  int dims_prod = tensor->dims_prod;

  int grid_size, block_size; 
  CalculateGridAndBlockSizes(dims_prod, grid_size, block_size);
  

  float *y = get_from_pool(thread_id, dims_prod, "relu");


  tensor->Sync();
  hipStream_t stream = ThreadsStream[thread_id];
  set_to_zero_kernel<<<grid_size, block_size, 0, stream>>>(y, dims_prod);

  return customOpTensor(y, dims, DimsProd(dims), "set_to_zero", nullptr, tensor);
}






extern "C" void *tensor_CopyArg(Scope_Struct *scope_struct, DT_tensor *tensor, char *new_tensor_name)
{
  std::cout << "CopyArgTensor " << new_tensor_name << ".\n";

  char *scope = scope_struct->scope;
  int thread_id = scope_struct->thread_id;
  std::string tensor_name = tensor->name;

  
  
  std::string arg_tensor_name = scope;
  arg_tensor_name = arg_tensor_name + new_tensor_name;
  

  std::vector<int> dims = tensor->dims;
  int dims_prod = tensor->dims_prod;

  float *tensor_ptr = tensor->tensor_ptr;

  

  DT_tensor *new_tensor = createTensor(tensor_ptr, dims, dims_prod, true, arg_tensor_name, tensor->hip_stream, tensor->loader);
  new_tensor->scopeless_name = tensor->scopeless_name;


  return new_tensor;
}










extern "C" float tensor_print(Scope_Struct *scope_struct, DT_tensor *tensor) {
  PrintTensor(scope_struct, tensor);
  return 0;
}









#include<string>
#include<map>
#include<vector>
#include<iostream>
#include<algorithm>
#include<random>
#include<thread>

#include <hip/hip_fp16.h>

#include "../common/include.h"
#include "tensor_struct.h"








void Tensor::NewNullTensor()
{
  tensor_ptr = nullptr;
  dims = {0};
  dims_prod = 0;
  cpu_tensor_ptr = nullptr;
  L_Node=nullptr;
  R_Node=nullptr;
  dy=nullptr;
  visited=false;
  weight=false;
  from_grad_or_load=false;
  hip_stream = nullptr;
  loader = nullptr;
  from_cudnn = "";
  is_pinned=false;
  thread_id = 0;
  scalar=1;
  Sparse_Idx_Tensor=nullptr;
}

void Tensor::NewTensor(float *new_tensor_ptr, std::vector<float> new_dims, float new_dims_prod,
              bool new_is_leaf, std::string new_name, CudaStreams *_cuda_stream, Loader *_loader){
  tensor_ptr = new_tensor_ptr;
  dims = new_dims;
  dims_prod = new_dims_prod;
  leaf = new_is_leaf;
  name = new_name;
  cpu_tensor_ptr = nullptr;
  op=leaf;
  L_Node=nullptr;
  R_Node=nullptr;
  dy=nullptr;
  visited=false;
  weight=false;
  op=tensor_leaf;
  from_grad_or_load=false;
  hip_stream = _cuda_stream;
  loader = _loader;
  from_cudnn = "";
  is_pinned=false;
  thread_id = 0;
  scalar=1;
  Sparse_Idx_Tensor=nullptr;
}
void Tensor::NewTensor(half *new_tensor_ptr, std::vector<float> new_dims, float new_dims_prod,
              bool new_is_leaf, std::string new_name, CudaStreams *_cuda_stream, Loader *_loader){
  half_ptr = new_tensor_ptr;
  dims = new_dims;
  dims_prod = new_dims_prod;
  leaf = new_is_leaf;
  name = new_name;
  cpu_tensor_ptr = nullptr;
  op=leaf;
  L_Node=nullptr;
  R_Node=nullptr;
  dy=nullptr;
  visited=false;
  weight=false;
  op=tensor_leaf;
  from_grad_or_load=false;
  hip_stream = _cuda_stream;
  loader = _loader;
  from_cudnn = "";
  is_pinned=false;
  thread_id = 0;
  scalar=1;
  Sparse_Idx_Tensor=nullptr;
}

void Tensor::NewPinned(float *new_tensor_ptr, float *new_cpu_tensor_ptr,
              std::vector<float> new_dims, float new_dims_prod,
              bool new_is_leaf, std::string new_name){
  tensor_ptr = new_tensor_ptr;
  cpu_tensor_ptr = new_cpu_tensor_ptr;
  dims = new_dims;
  dims_prod = new_dims_prod;
  leaf = new_is_leaf;
  name = new_name;
  weight=false;
  from_grad_or_load=true;
  is_pinned=true;
  thread_id = 0;
  Sparse_Idx_Tensor=nullptr;
}

void Tensor::AttrTensor(float *new_tensor_ptr, std::vector<float> new_dims, float new_dims_prod, CudaStreams *_cuda_stream, Loader *_loader){
  tensor_ptr = new_tensor_ptr;
  dims = new_dims;
  dims_prod = new_dims_prod;
  hip_stream = _cuda_stream;
  loader = _loader;
  is_pinned=false;
}


void Tensor::AttrNodes(Tensor *new_L_Tensor, Tensor *new_R_Tensor, int op_type)
{
  L_Node = new_L_Tensor;
  R_Node = new_R_Tensor;
  op = op_type;
  leaf=false;
  visited=false;
  dy=nullptr;
  weight=false;
  from_grad_or_load = ((from_grad_or_load||new_L_Tensor->from_grad_or_load||new_R_Tensor->from_grad_or_load)&&!in_int(op, gradless_ops));
  is_pinned=false;
}

void Tensor::AttrLNode(Tensor *new_L_Tensor, int op_type)
{
  L_Node = new_L_Tensor;
  R_Node=nullptr;
  op = op_type;
  leaf=false;
  visited=false;
  dy=nullptr;
  weight=false;
  from_grad_or_load = ((from_grad_or_load||new_L_Tensor->from_grad_or_load)&&!in_int(op, gradless_ops));
  is_pinned=false;
}

void Tensor::AttributionBackwardNode(std::string _name, Tensor *new_R_Tensor)
{
  name = _name;
  R_Node = new_R_Tensor;
  op = attribution;
  leaf=false;
  visited=false;
  
  L_Node=nullptr;
  dy=nullptr;
  weight=false;
  is_pinned=false;
}
void Tensor::SetIsWeight()
{
  weight=true;
  from_grad_or_load=true;
  is_pinned=false;
}
void Tensor::SetBias(float *b, int b_size)
{
  this->b=b;
  this->b_size=b_size;
  leaf=true;
  is_pinned=false;
}
void Tensor::Sync()
{
  if(loader!=nullptr)
  {
    loader->Sync();
    delete loader;
    loader=nullptr;
  }
  if(hip_stream!=nullptr)
  {
    SynchronizeStream(hip_stream);
    hip_stream=nullptr;
  }
}



Tensor *createTensor(float* tensor_ptr, const std::vector<float>& dims, float kDataLen,
                     bool is_leaf, std::string name, CudaStreams *_cuda_stream, Loader *_loader) {
    Tensor *new_tensor = new Tensor();
    new_tensor->NewTensor(tensor_ptr, dims, kDataLen, is_leaf, name, _cuda_stream, _loader);
    return new_tensor;
}
Tensor *createTensorHalf(half* tensor_ptr, const std::vector<float>& dims, float kDataLen,
                     bool is_leaf, std::string name, CudaStreams *_cuda_stream, Loader *_loader) {
    Tensor *new_tensor = new Tensor();
    new_tensor->NewTensor(tensor_ptr, dims, kDataLen, is_leaf, name, _cuda_stream, _loader);
    return new_tensor;
}

Tensor *createPinned(float* tensor_ptr, float *tensor_cpu, const std::vector<float>& dims, float kDataLen,
                     std::string name) {
    Tensor *new_tensor = new Tensor();
    new_tensor->NewPinned(tensor_ptr, tensor_cpu, dims, kDataLen, true, name);
    return new_tensor;
}
Tensor *createBackward(std::string name, Tensor *tensor) {
    Tensor *new_tensor = new Tensor();
    new_tensor->AttributionBackwardNode(name, tensor);
    return new_tensor;
}
Tensor *wrapTensorWithDetached(Tensor* tensor) {
    /*
    Tensor *new_tensor = new Tensor();

    new_tensor->NewNullTensor();
    new_tensor->AttrLNode(tensor, detach_op);
    new_tensor->tensor_ptr = tensor->tensor_ptr;
    new_tensor->dims_prod = tensor->dims_prod;
    new_tensor->dims = tensor->dims;
    
    return new_tensor;
    */
    
    tensor->op = detach_op;
    return tensor;
}


bool in_tensor_ptr_vec(Tensor *value, const std::vector<Tensor *>& list) {
    return std::find(list.begin(), list.end(), value) != list.end();
}

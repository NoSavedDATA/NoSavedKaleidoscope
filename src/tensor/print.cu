#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <iomanip>
#include <iostream>
#include <limits>
#include <numbers> 
#include <memory> 
#include <cstring> 


#include "../common/cu_commons.h"
#include "../mangler/scope_struct.h"
#include "include.h"


extern "C" float PrintTensor(Scope_Struct *scope_struct, DT_tensor *tensor){
  int thread_id = scope_struct->thread_id;
  std::string tensorName = tensor->name;
  std::cout << "Printing tensor " << tensorName << " at stream " << thread_id << "\n";



  int arr_size = tensor->dims_prod;
  float *tensor_cpu = new float[arr_size];

  
  std::vector<int> dims = tensor->dims;
  
  
  hipStream_t stream = ThreadsStream[thread_id];
  tensor->Sync();
  hipStreamSynchronize(stream);
  hipDeviceSynchronize();
  cudaCheck(hipMemcpy(tensor_cpu, tensor->tensor_ptr, arr_size*sizeof(float), hipMemcpyDeviceToHost));


  std::cout << "\nTensor \033[95m" << tensorName << "\033[0m:\n\n";
  PrintDims(dims);
  std::cout << "\n";
  std::vector<int> ends;


  for (int i = 0; i < dims.size(); i++) {
    int prod=1;
    for (int j = 0; j <= i; j++)
      prod = prod*dims[dims.size()-1-j];
    ends.push_back(prod);
  }


  int line = 1;
  bool line_changed = true;

  
  //if (arr_size>2000)
  //  arr_size = 2000;

  for (int i = 0; i < arr_size; i++) {

    int to_prints = 0;

    for (int e = 0; e < ends.size(); e++)
    {
      if (fmod((arr_size-i),(int)ends[e]) == 0.0f)
        to_prints+=1;
    }

    if(to_prints>0)
    {
      for (int j=0; j<(dims.size()-to_prints); j++)
        std::cout << " ";
        
      for (int j=0; j<to_prints; j++)
        std::cout << "[";
    }
    

    //std::cout << "LAST SIZE " << dims[dims.size()-1] << " Mod: " << fmod(i, 1+dims[dims.size()-1]) << "\n";
    int precision;
    if (tensor_cpu[i]>=0)
      precision=4;
    else
      precision=3;
    std::cout << std::fixed  << std::setprecision(precision) << tensor_cpu[i];


    for (int e = 0; e < ends.size(); e++)
      if (fmod((i+1), ends[e]) == 0.0f)
        std::cout << "],";
    

    if (i!=(arr_size-1))
    {
      if (fmod(i+1, dims[dims.size()-1]) == 0.0f)
      {
        line+=1;
        line_changed=true;
        std::cout << "\n";
      }
      else
        std::cout << ",  ";
    }

    if(fmod(i+1, ends[1]) == 0.0f)
      std::cout << "\n";


  }
  
  std::cout << "\n";
  PrintDims(dims);
  std::cout << "\n\n";

  delete[] tensor_cpu;

  return 0;
}





extern "C" float PrintTensorF(const float *cuda_tensor, int d1, int d2){

  std::vector<int> dims;
  dims.push_back(d1);
  dims.push_back(d2);

  int arr_size = DimsProd(dims);


  float *tensor = new float[arr_size];
  //std::cout << "Printing DT_tensor " << arr_size << "\n";
  
  hipDeviceSynchronize();
  cudaCheck(hipMemcpy(tensor, cuda_tensor, arr_size*sizeof(float), hipMemcpyDeviceToHost));


  
  std::cout << "\n";
  PrintDims(dims);
  std::vector<float> ends;


  for (int i = 0; i < dims.size(); i++) {
    int prod=1;
    for (int j = 0; j <= i; j++)
      prod = prod*dims[dims.size()-1-j];
    ends.push_back(prod);
  }


  int line = 1;
  bool line_changed = true;
  for (int i = 0; i < arr_size; i++) {

    int to_prints = 0;

    for (int e = 0; e < ends.size(); e++)
    {
      if (fmod((arr_size-i),(int)ends[e]) == 0.0f)
        to_prints+=1;
    }

    if(to_prints>0)
    {
      for (int j=0; j<(dims.size()-to_prints); j++)
        std::cout << " ";
        
      for (int j=0; j<to_prints; j++)
        std::cout << "[";
    }
    

    //std::cout << "LAST SIZE " << dims[dims.size()-1] << " Mod: " << fmod(i, 1+dims[dims.size()-1]) << "\n";
    int precision;
    if (tensor[i]>=0)
      precision=4;
    else
      precision=3;
    std::cout << std::fixed  << std::setprecision(precision) << tensor[i];


    for (int e = 0; e < ends.size(); e++)
      if (fmod((i+1),(int)ends[e]) == 0.0f)
        std::cout << "],";
    

    if (i!=(arr_size-1))
    {
      if (fmod(i+1, dims[dims.size()-1]) == 0.0f)
      {
        line+=1;
        line_changed=true;
        std::cout << "\n";
      }
      else
        std::cout << ",  ";
    }

    if(fmod(i+1, ends[1]) == 0.0f)
      std::cout << "\n";


  }
  std::cout << "\n";
  
  delete[] tensor;

  return 0;
}




extern "C" float PrintTensorI8(const int8_t *cuda_tensor, int d1, int d2){

  std::vector<int> dims;
  dims.push_back(d1);
  dims.push_back(d2);

  int arr_size = DimsProd(dims);


  int8_t *tensor = new int8_t[arr_size];
  //std::cout << "Printing DT_tensor " << arr_size << "\n";
  
  hipDeviceSynchronize();
  cudaCheck(hipMemcpy(tensor, cuda_tensor, arr_size*sizeof(int8_t), hipMemcpyDeviceToHost));


  
  std::cout << "\n";
  PrintDims(dims);
  std::vector<int> ends;


  for (int i = 0; i < dims.size(); i++) {
    int prod=1;
    for (int j = 0; j <= i; j++)
      prod = prod*dims[dims.size()-1-j];
    ends.push_back(prod);
  }


  int line = 1;
  bool line_changed = true;
  for (int i = 0; i < arr_size; i++) {

    int to_prints = 0;

    for (int e = 0; e < ends.size(); e++)
    {
      if (fmod((arr_size-i),(int)ends[e]) == 0)
        to_prints+=1;
    }

    if(to_prints>0)
    {
      for (int j=0; j<(dims.size()-to_prints); j++)
        std::cout << " ";
        
      for (int j=0; j<to_prints; j++)
        std::cout << "[";
    }
    

    std::cout << (int)tensor[i];


    for (int e = 0; e < ends.size(); e++)
      if (fmod((i+1),(int)ends[e]) == 0)
        std::cout << "],";
    

    if (i!=(arr_size-1))
    {
      if (fmod(i+1, dims[dims.size()-1]) == 0)
      {
        line+=1;
        line_changed=true;
        std::cout << "\n";
      }
      else
        std::cout << ",  ";
    }

    if(fmod(i+1, ends[1]) == 0)
      std::cout << "\n";


  }
  std::cout << "\n";
  
  delete[] tensor;

  return 0;
}
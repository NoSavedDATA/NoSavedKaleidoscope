#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_fp16.h>
#include <hipDNN.h>
#include <mma.h>

#include <string>
#include <vector>


#include "../../../backprop/include.h"
#include "../../../common/cu_commons.h"
#include "../../../cuda_kernels/handles.h"
#include "../../../cuda_kernels/elementwise_kernels_inline.cu"
#include "../../../tensor/include.h"
#include "class.h"
#include "kernels.h"


DT_EmbeddingLn::DT_EmbeddingLn(int V, int C, int OC, std::string Init, std::string Name)
    : V(V), C(C), OC(OC), Init(Init), Name(Name) {
    // C == num_codebooks
    B = 0;

    float *w_cpu, *book_cpu;
        
    //w_cpu = make_xavier_uniform_float(OC*C, OC,  C);
    // w_cpu = make_normal(OC*C);
    book_cpu = make_embedding_uniform(V*C);
    w_cpu = make_xavier_uniform_float(OC*C, OC, C);



    
    
    Book = get_from_pool(0, V*C, "Embedding Book");
    W = get_from_pool(0, OC*C, "Embedding W");

    hipMemcpy(Book, book_cpu, V*C*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(W, w_cpu, OC*C*sizeof(float), hipMemcpyHostToDevice);

    DT_tensor *tensor_Book = createTensor(Book, {V, OC}, V*C, true, Name+"_Book");
    DT_tensor *tensor_W = createTensor(W, {C, OC}, OC*C, true, Name+"_W");
    
    
    


    dBook = get_from_pool(0, V*C, "embedding dW");
    dW = get_from_pool(0, OC*C, "embedding dW");
    set_to_zero_kernel<<<std::ceil((V*C)/(float)TILE_SIZE_SQ), TILE_SIZE_SQ, 0, main_stream>>>(dBook, V*C);
    set_to_zero_kernel<<<std::ceil((OC*C)/(float)TILE_SIZE_SQ), TILE_SIZE_SQ, 0, main_stream>>>(dW, OC*C);

    NamedTensorsT[Name+"_Book"] = tensor_Book;
    NamedParamGrads[Name+"_Book"] = dBook;
    NamedTensorsT[Name+"_W"] = tensor_W;
    NamedParamGrads[Name+"_W"] = dW;

    delete[] book_cpu;
    delete[] w_cpu;

    changed_descriptors=false;
}


void DT_EmbeddingLn::SetDescriptors(int B)
{
  this->B=B;
  changed_descriptors=true;
}


float *DT_EmbeddingLn::Forward(DT_tensor *tensor, int B, int thread_id)
{
  float *out = get_from_pool(thread_id, B*OC, "embedding out");


  if (this->B!=B)
    SetDescriptors(B);

  //if(thread_id==0 && nn_mode==training_mode)
  //  NamedTensorsT[Name]->Sparse_Idx_Tensor = tensor;


  int b = B;
  while (b>1 && std::ceil((b*OC)/TILE_SIZE_SQ)>128)
    b-=1;
  int batches_per_block = std::ceil(B/(float)b);



  dim3 block_size(TILE_SIZE, TILE_SIZE);
  dim3 grid_size(std::ceil(OC/(float)TILE_SIZE), std::ceil(b/(float)TILE_SIZE));
  //std::cout << "blocks: " << (grid_size.x*grid_size.y) << ", b " << b << ", B " << B << ", OC " << OC << ", TILE_SIZE " << TILE_SIZE << "\n";
  hipStream_t stream = ThreadsStream[thread_id];
  embeddingln_forward_kernel<<<grid_size, block_size, 0, stream>>>(tensor->tensor_ptr, W, out, TILE_SIZE, B, batches_per_block, C, OC);

  return out;
}



void DT_EmbeddingLn::SetBackwardDescriptors()
{
}

void DT_EmbeddingLn::Backward(float *x, float *dy)
{
  /*
  if(changed_descriptors)
    SetBackwardDescriptors();
  //dW = dy;
  copy_tensor_kernel<<<std::ceil((B*OC)/(float)THREADS_PER_BLOCK), THREADS_PER_BLOCK, 0, main_stream>>>(dW, dy, B*C);
  */

  

 
  dim3 block_size(TILE_SIZE, TILE_SIZE);
  dim3 grid_size(std::ceil((float)OC/(float)TILE_SIZE), std::ceil((float)B/(float)TILE_SIZE));
  embeddingln_backward_kernel<<<grid_size, block_size, 0, main_stream>>>(x, dW, dy, TILE_SIZE, B, C, OC);
}
#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_fp16.h>
#include <hipDNN.h>
#include <mma.h>

#include <string>
#include <vector>


#include "../../../backprop/include.h"
#include "../../../common/cu_commons.h"
#include "../../../cuda_kernels/elementwise_kernels_inline.cu"
#include "../../../cuda_kernels/handles.h"
#include "../../../tensor/include.h"
#include "class.h"


Conv2dCPP::Conv2dCPP(int C, int OC, int ks, int stride, int padding, std::string Init, std::vector<std::string> Notes, std::string Name)
    : C(C), OC(OC), ks(ks), stride(stride), padding(padding), Init(Init), Notes(Notes), Name(Name) {
    NamedTensorsT[Name+"W"] = new DT_tensor();
    d_filter=nullptr;
    d_workspace=nullptr;
    d_workspace_w_back=nullptr;
    d_workspace_y_back=nullptr;
    workspace_size=0;
    workspace_size_w_back=0;
    workspace_size_y_back=0;
}


void Conv2dCPP::SetDescriptors(int H, int W, int B, DT_tensor *tensor)
{
  this->H = H;
  this->W = W;
  this->B = B;


  //std::cout << "\nConv2d Set Descriptors\nC: " << C << " OC " << OC << " ks " << ks << " stride " << stride << " padding " << padding << " H " << H << " W " << W << "\n";


  out_H = std::floor((H - ks + 2 * padding) / stride) + 1;
  out_W = std::floor((W - ks + 2 * padding) / stride) + 1;
  //std::cout << "Out H: " << out_H << " out W: " << out_W << "\n";


  /*
  switch(tensor->op)
  {
    case conv2d:
      input_desc = NamedConv2d[tensor->from_cudnn]->output_desc;
      break;
    case bn2drelu:
      input_desc = NamedBN2dRelu[tensor->from_cudnn]->output_desc;
      break;
    case cudnn_relu_op:
      input_desc = NamedRelu[tensor->from_cudnn]->output_desc;
      break;
    case batchnorm2d:
      input_desc = NamedBatchNorm2d[tensor->from_cudnn]->output_desc;
      break;
    case maxpool2d:
      input_desc = NamedMaxPool2d[tensor->from_cudnn]->output_desc;
      break;
    default:
      // Initialize input tensor descriptor
      checkCUDNN(hipdnnCreateTensorDescriptor(&input_desc));
      checkCUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W));
      break;
  }*/

  checkCUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W));
  
  // Initialize filter descriptor
  hipdnnFilterDescriptor_t filter_desc;
  checkCUDNN(hipdnnCreateFilterDescriptor(&filter_desc));
  checkCUDNN(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OC, C, ks, ks));
  this->filter_desc = filter_desc;

  // Initialize convolution descriptor
  hipdnnConvolutionDescriptor_t conv_desc;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(conv_desc, padding, padding, stride, stride, 1, 1,
                                           HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));
  this->conv_desc = conv_desc;

  // Initialize output tensor descriptor
  hipdnnTensorDescriptor_t output_desc;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, OC, out_H, out_W));
  this->output_desc = output_desc;

  
  int requested_algo_count;
  int algo_count;




  // Forward
  checkCUDNN(cudnnGetConvolutionForwardAlgorithmMaxCount(cudnn, &requested_algo_count));
  std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(requested_algo_count);
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(
        cudnn,
        input_desc,
        filter_desc,
        conv_desc,
        output_desc,
        requested_algo_count,
        &algo_count,
        perf_results.data()
  ));

  this->fwd_algo = perf_results.front().algo;


  
  if (d_workspace!=nullptr)
    move_to_pool_pow2(0, workspace_size, d_workspace, "d workspace");
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn,
        input_desc,
        filter_desc,
        conv_desc,
        output_desc,
        fwd_algo,
        &workspace_size
  ));
  d_workspace = get_from_pool_pow2(0, workspace_size, "d workspace");
  
  




  // Backward to input
  checkCUDNN(cudnnGetConvolutionBackwardDataAlgorithmMaxCount(cudnn, &requested_algo_count));
  std::vector<hipdnnConvolutionBwdDataAlgoPerf_t> perf_results_back_y(requested_algo_count);
  checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithm(
        cudnn,
        filter_desc,
        output_desc,
        conv_desc,
        input_desc,
        requested_algo_count,
        &algo_count,
        perf_results_back_y.data()
  ));

  y_bwd_algo = perf_results_back_y.front().algo;

  
  if(d_workspace_y_back!=nullptr)
    move_to_pool_pow2(0, workspace_size_y_back, d_workspace_y_back, "d workspace y back");
  checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
        cudnn,
        filter_desc,
        output_desc,
        conv_desc,
        input_desc,
        y_bwd_algo,
        &workspace_size_y_back
  ));

  d_workspace_y_back = get_from_pool_pow2(0, workspace_size_y_back, "d workspace y back");
  




  // Backward to weight
  checkCUDNN(cudnnGetConvolutionBackwardFilterAlgorithmMaxCount(cudnn, &requested_algo_count));
  std::vector<hipdnnConvolutionBwdFilterAlgoPerf_t> perf_results_back_w(requested_algo_count);
  checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithm(
        cudnn,
        input_desc,
        output_desc,
        conv_desc,
        filter_desc,
        requested_algo_count,
        &algo_count,
        perf_results_back_w.data()
  ));

  w_bwd_algo = perf_results_back_w.front().algo;

  
  
  if (d_workspace_w_back!=nullptr)
    move_to_pool_pow2(0, workspace_size_w_back, d_workspace_w_back, "conv d workspace w back");
  checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
        cudnn,
        input_desc,
        output_desc,
        conv_desc,
        filter_desc,
        w_bwd_algo,
        &workspace_size_w_back
  ));
  d_workspace_w_back = get_from_pool_pow2(0, workspace_size_w_back, "conv d workspace w back");
  
  
}





void Conv2dCPP::InitFilters()
{
  std::vector<float> h_filter;
  float *filter;
  for (std::size_t idx = 0; idx < C * OC; ++idx) {

    if (Init=="xavu_relu")
      filter = make_xavier_uniform_float_relu(ks*ks, ks*ks*C, ks*ks*OC);
    if (Init == "xavu_tanh")
      filter = make_xavier_uniform_float_tanh(ks*ks, ks*ks*C, ks*ks*OC);
    if (Init=="he_normal_relu")
      filter = make_he_normal_float_relu(ks*ks, ks*ks*C);
    if (Init == "init_gpt")
      filter = make_gpt_init(ks*ks);
    if (Init=="xavu")
      filter = make_xavier_uniform_float(ks*ks, ks*ks*C, ks*ks*OC);
    if (Init=="zeros")
      filter = make_zeros_float(ks*ks);
    if (Init=="ones")
      filter = make_ones_float(ks*ks);
    if (Init=="randu")
      filter = make_random_float_uniform(ks*ks);


    for (int i=0; i < ks*ks; i++)
      h_filter.emplace_back(filter[i]);

    delete[] filter;
    //for (const auto& val : filter) 
    //  h_filter.emplace_back(val);
  }
    
  float* d_filter = nullptr;
  const std::size_t filter_size = h_filter.size();
  cudaCheck(hipMalloc(&d_filter, filter_size * sizeof(float)));

  cudaCheck(hipMemcpy(d_filter, h_filter.data(), filter_size * sizeof(float), hipMemcpyDefault));
  this->d_filter = d_filter;
  

  std::vector<float> kernel_dims = {(float)OC, (float)C, (float)ks, (float)ks}; 

  DT_tensor *tensor_W = createTensor(d_filter, kernel_dims, DimsProd(kernel_dims), true, Name+"W");
  tensor_W->SetIsWeight();

  NamedTensorsT[Name+"W"] = tensor_W;
}




void Conv2dCPP::FirstBackward()
{
  if (first_backward)
  {
    dW = get_from_pool(0, OC*C*ks*ks, "conv2d gradient");
    set_to_zero_kernel<<<std::ceil((OC*C*ks*ks)/(float)TILE_SIZE_SQ), TILE_SIZE_SQ, 0, main_stream->stream>>>(dW, OC*C*ks*ks);

    NamedParamGrads[Name+"W"] = dW;
    first_backward = false;
  }
  
}


float *Conv2dCPP::Forward(DT_tensor *tensor, int H, int W, int B, int thread_id)
{
  // Initialize descriptors.
  //std::cout << "\nConv2d Forward with H: " << H << " W: " << W << "\n";

  if (H != this->H || W != this->W || B != this->B)
    this->SetDescriptors(H, W, B, tensor);

  // Initialize weights.
  if (d_filter==nullptr)
    this->InitFilters();
  

  // Forward
  float *d_output = get_from_pool(thread_id, B * out_H * out_W * OC, "conv2d");




  constexpr float one = 1.0f;
  constexpr float zero = 0.0f;

  
 

  checkCUDNN(hipdnnConvolutionForward(
        cudnn,
        &one,
        input_desc,
        tensor->tensor_ptr,
        filter_desc,
        d_filter,
        conv_desc,
        fwd_algo,
        d_workspace,
        workspace_size,
        &zero,
        output_desc,
        d_output
    ));
  



  return d_output;
}


void Conv2dCPP::Backward(float *tensor, float *dx, float *dy)
{
  //std::cout << "\nConv2d Backward with H: " << H << " W: " << W << "\n";


  constexpr float one = 1.0f;
  constexpr float zero = 0.0f;
  
  FirstBackward();

  // Backward to input
  checkCUDNN(hipdnnConvolutionBackwardData(
    cudnn,
    &one,
    filter_desc, // input tensor descriptor
    d_filter,
    output_desc, // output grad tensor descriptor
    dy,
    conv_desc, // convolution descriptor
    y_bwd_algo, //Obtained with getConvolutionBackwardDataAlgorithm
    d_workspace_y_back, 
    workspace_size_y_back, //Obtained with getConvolutionBackwardDataWorkspaceSize
    &zero,
    input_desc, // filter descriptor
    dx
  ));



  // Backward to weight
  checkCUDNN(hipdnnConvolutionBackwardFilter(
    cudnn,
    &one,
    input_desc, // input tensor descriptor
    tensor,
    output_desc, // output grad tensor descriptor
    dy,
    conv_desc, // convolution descriptor
    w_bwd_algo, //Obtained with getConvolutionBackwardFilterAlgorithm
    d_workspace_w_back, 
    workspace_size_w_back, //Obtained with getConvolutionBackwardFilterWorkspaceSize
    &one,
    filter_desc, // filter descriptor
    dW
  ));

  

  /*
  std::cout << "d_w is:\n";
  PrintTensorF(dW, C*OC, ks*ks);
  std::cout << "\n";
  */

}
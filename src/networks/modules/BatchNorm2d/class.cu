#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_fp16.h>
#include <hipDNN.h>
#include <mma.h>

#include <string>
#include <vector>


#include "../../../backprop/include.h"
#include "../../../common/cu_commons.h"
#include "../../../cuda_kernels/calculate_grids.h"
#include "../../../cuda_kernels/elementwise_kernels_inline.cu"
#include "../../../cuda_kernels/handles.h"
#include "../../../tensor/include.h"
#include "class.h"




BatchNorm2dCPP::BatchNorm2dCPP(int C, std::string Name)
    : C(C), Name(Name) {
  // NamedTensorsT[Name+"W"] = new DT_tensor();
  // NamedTensorsT[Name+"B"] = new DT_tensor();
}



void BatchNorm2dCPP::SetDescriptors(int H, int W, int B, DT_tensor *tensor)
{
  this->H = H;
  this->W = W;
  this->B = B;

  /*
  switch(tensor->op)
  {
    case conv2d:
      input_desc = NamedConv2d[tensor->from_cudnn]->output_desc;
      break;
    case bn2drelu:
      input_desc = NamedBN2dRelu[tensor->from_cudnn]->output_desc;
      break;
    case cudnn_relu_op:
      input_desc = NamedRelu[tensor->from_cudnn]->output_desc;
      break;
    case batchnorm2d:
      input_desc = NamedBatchNorm2d[tensor->from_cudnn]->output_desc;
      break;
    case maxpool2d:
      input_desc = NamedMaxPool2d[tensor->from_cudnn]->output_desc;
      break;
    default:
      checkCUDNN(hipdnnCreateTensorDescriptor(&input_desc));
      checkCUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W));
      break;
  }*/
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W));
  
  
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W));
  
  checkCUDNN(hipdnnCreateTensorDescriptor(&scale_bias_mean_var_desc));
  //checkCUDNN(hipdnnSetTensor4dDescriptor(scale_bias_mean_var_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, C, 1, 1));
  checkCUDNN(hipdnnDeriveBNTensorDescriptor(scale_bias_mean_var_desc, input_desc, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT));
}

void BatchNorm2dCPP::InitMovingAverages()
{
  float *aux;

  aux = make_ones_float(C);
  cudaCheck(hipMalloc(&scale, C*sizeof(float)));
  cudaCheck(hipMemcpy(scale, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;
  
  aux = make_zeros_float(C);
  cudaCheck(hipMalloc(&bias, C*sizeof(float)));
  cudaCheck(hipMemcpy(bias, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;
  

  aux = make_zeros_float(C);
  cudaCheck(hipMalloc(&running_mean, C*sizeof(float)));
  cudaCheck(hipMemcpy(running_mean, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;

  aux = make_zeros_float(C);
  cudaCheck(hipMalloc(&saved_mean, C*sizeof(float)));
  cudaCheck(hipMemcpy(saved_mean, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;
  
  
  aux = make_ones_float(C);
  cudaCheck(hipMalloc(&running_var, C*sizeof(float)));
  cudaCheck(hipMemcpy(running_var, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;

  aux = make_ones_float(C);
  cudaCheck(hipMalloc(&saved_var, C*sizeof(float)));
  cudaCheck(hipMemcpy(saved_var, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;


  DT_tensor *scale_tensor, *bias_tensor;
  scale_tensor = new DT_tensor();
  scale_tensor->NewTensor(scale, {(float)C}, C, true, Name);

  bias_tensor = new DT_tensor();
  bias_tensor->NewTensor(bias, {(float)C}, C, true, Name);

  NamedTensorsT[Name+"W"] = scale_tensor;
  NamedTensorsT[Name+"B"] = bias_tensor;
}

float *BatchNorm2dCPP::Forward(DT_tensor *tensor, int H, int W, int B, int C, int thread_id)
{

  if (H != this->H || W != this->W || B != this->B)
    this->SetDescriptors(H, W, B, tensor);

  // Initialize weights.
  if (scale==nullptr)
    this->InitMovingAverages();


  // Forward
  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(B*C);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  
  

  float *output = get_from_pool(thread_id, B * H * W * C, "batchnorm2d");
  //set_to_one_kernel<<<grid_size, block_size>>>(output, B * H * W * C);
  
  
  constexpr float one = 1.0f;
  constexpr float zero = 0.0f;
  float gamma = 0.9f;
  float eps = 0.00001f;

  

  if(nn_mode==training_mode)
  {
    checkCUDNN(hipdnnBatchNormalizationForwardTraining(
      cudnn,
      HIPDNN_BATCHNORM_SPATIAL_PERSISTENT,
      &one,
      &zero,
      input_desc,
      tensor->tensor_ptr,
      output_desc,
      output,
      scale_bias_mean_var_desc,
      scale,
      bias,
      gamma,
      running_mean,
      running_var,
      eps,
      saved_mean,
      saved_var
    ));
  }
  else
  {
    checkCUDNN(hipdnnDeriveBNTensorDescriptor(scale_bias_mean_var_desc, input_desc, HIPDNN_BATCHNORM_SPATIAL));
    checkCUDNN(hipdnnBatchNormalizationForwardInference(
      cudnn,
      HIPDNN_BATCHNORM_SPATIAL,
      &one,
      &zero,
      input_desc,
      tensor->tensor_ptr,
      output_desc,
      output,
      scale_bias_mean_var_desc,
      scale,
      bias,
      running_mean,
      running_var,
      eps
    ));
  }
  
  return output;
}



void BatchNorm2dCPP::FirstBackward() {
  if (first_backward) {

    dW = get_from_pool(0, C, "BatchNorm2d dW");
    dB = get_from_pool(0, C, "BatchNorm2d dB");

    
    set_to_zero_kernel<<<std::ceil(C/(float)TILE_SIZE_SQ), TILE_SIZE_SQ, 0, main_stream>>>(dW, C);
    set_to_zero_kernel<<<std::ceil(C/(float)TILE_SIZE_SQ), TILE_SIZE_SQ, 0, main_stream>>>(dB, C);

    NamedParamGrads[Name+"W"] = dW;
    NamedParamGrads[Name+"B"] = dB;

    first_backward=false;
  }
}




void BatchNorm2dCPP::Backward(float *tensor, float *dx, float *dy)
{
  constexpr float one = 1.0f;
  constexpr float zero = 0.0f;
  float eps = 0.00001f;
  
  FirstBackward();
  

  checkCUDNN(hipdnnBatchNormalizationBackward(
    cudnn,
    HIPDNN_BATCHNORM_SPATIAL_PERSISTENT,
    &one,
    &zero,
    &one,
    &one,
    input_desc,
    tensor,
    output_desc,
    dy,
    input_desc,
    dx,
    scale_bias_mean_var_desc,
    scale,
    dW,
    dB,
    eps,
    saved_mean,
    saved_var
  ));
}

#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_fp16.h>
#include <hipDNN.h>
#include <mma.h>

#include <string>
#include <vector>


#include "../../../common/cu_commons.h"
#include "../../../cuda_kernels/calculate_grids.h"
#include "../../../cuda_kernels/handles.h"
#include "../../../tensor/include.h"
#include "class.h"




BatchNorm2dCPP::BatchNorm2dCPP(int C, std::string Name)
    : C(C), Name(Name) {
  NamedTensorsT[Name] = new Tensor();
  NamedTensorsT[Name+"_bias"] = new Tensor();
}



void BatchNorm2dCPP::SetDescriptors(int H, int W, int B, Tensor *tensor)
{
  this->H = H;
  this->W = W;
  this->B = B;

  /*
  switch(tensor->op)
  {
    case conv2d:
      input_desc = NamedConv2d[tensor->from_cudnn]->output_desc;
      break;
    case bn2drelu:
      input_desc = NamedBN2dRelu[tensor->from_cudnn]->output_desc;
      break;
    case cudnn_relu_op:
      input_desc = NamedRelu[tensor->from_cudnn]->output_desc;
      break;
    case batchnorm2d:
      input_desc = NamedBatchNorm2d[tensor->from_cudnn]->output_desc;
      break;
    case maxpool2d:
      input_desc = NamedMaxPool2d[tensor->from_cudnn]->output_desc;
      break;
    default:
      checkCUDNN(hipdnnCreateTensorDescriptor(&input_desc));
      checkCUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W));
      break;
  }*/
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W));
  
  
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W));
  
  checkCUDNN(hipdnnCreateTensorDescriptor(&scale_bias_mean_var_desc));
  //checkCUDNN(hipdnnSetTensor4dDescriptor(scale_bias_mean_var_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, C, 1, 1));
  checkCUDNN(hipdnnDeriveBNTensorDescriptor(scale_bias_mean_var_desc, input_desc, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT));
}

void BatchNorm2dCPP::InitMovingAverages()
{
  float *aux;

  aux = make_ones_float(C);
  cudaCheck(hipMalloc(&scale, C*sizeof(float)));
  cudaCheck(hipMemcpy(scale, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;
  
  aux = make_zeros_float(C);
  cudaCheck(hipMalloc(&bias, C*sizeof(float)));
  cudaCheck(hipMemcpy(bias, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;
  

  aux = make_zeros_float(C);
  cudaCheck(hipMalloc(&running_mean, C*sizeof(float)));
  cudaCheck(hipMemcpy(running_mean, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;

  aux = make_zeros_float(C);
  cudaCheck(hipMalloc(&saved_mean, C*sizeof(float)));
  cudaCheck(hipMemcpy(saved_mean, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;
  
  
  aux = make_ones_float(C);
  cudaCheck(hipMalloc(&running_var, C*sizeof(float)));
  cudaCheck(hipMemcpy(running_var, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;

  aux = make_ones_float(C);
  cudaCheck(hipMalloc(&saved_var, C*sizeof(float)));
  cudaCheck(hipMemcpy(saved_var, aux, C*sizeof(float), hipMemcpyHostToDevice));
  delete[] aux;
}

float *BatchNorm2dCPP::Forward(Tensor *tensor, int H, int W, int B, int C, int thread_id)
{

  if (H != this->H || W != this->W || B != this->B)
    this->SetDescriptors(H, W, B, tensor);

  // Initialize weights.
  if (scale==nullptr)
    this->InitMovingAverages();


  // Forward
  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(B*C);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  
  

  float *output = get_from_pool(thread_id, B * H * W * C, "batchnorm2d");
  //set_to_one_kernel<<<grid_size, block_size>>>(output, B * H * W * C);
  
  
  constexpr float one = 1.0f;
  constexpr float zero = 0.0f;
  float gamma = 0.9f;
  float eps = 0.00001f;

  

  if(nn_mode==training_mode)
  {
    checkCUDNN(hipdnnBatchNormalizationForwardTraining(
      cudnn,
      HIPDNN_BATCHNORM_SPATIAL_PERSISTENT,
      &one,
      &zero,
      input_desc,
      tensor->tensor_ptr,
      output_desc,
      output,
      scale_bias_mean_var_desc,
      scale,
      bias,
      gamma,
      running_mean,
      running_var,
      eps,
      saved_mean,
      saved_var
    ));
  }
  else
  {
    checkCUDNN(hipdnnDeriveBNTensorDescriptor(scale_bias_mean_var_desc, input_desc, HIPDNN_BATCHNORM_SPATIAL));
    checkCUDNN(hipdnnBatchNormalizationForwardInference(
      cudnn,
      HIPDNN_BATCHNORM_SPATIAL,
      &one,
      &zero,
      input_desc,
      tensor->tensor_ptr,
      output_desc,
      output,
      scale_bias_mean_var_desc,
      scale,
      bias,
      running_mean,
      running_var,
      eps
    ));
  }
  
  return output;
}


void BatchNorm2dCPP::Backward(float *tensor, float *dx, float *dw, float *db, float *dy)
{
  constexpr float one = 1.0f;
  constexpr float zero = 0.0f;
  float eps = 0.00001f;
  
  
  checkCUDNN(hipdnnBatchNormalizationBackward(
    cudnn,
    HIPDNN_BATCHNORM_SPATIAL_PERSISTENT,
    &one,
    &zero,
    &one,
    &one,
    input_desc,
    tensor,
    output_desc,
    dy,
    input_desc,
    dx,
    scale_bias_mean_var_desc,
    scale,
    dw,
    db,
    eps,
    saved_mean,
    saved_var
  ));
}
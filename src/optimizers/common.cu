#include "../backprop/include.h"
#include "../cuda_kernels/calculate_grids.h"
#include "../cuda_kernels/elementwise_kernels_inline.cu"
#include "../cuda_threads/include.h"

#include "interface.h"



std::unique_ptr<Optimizer> optimize(std::unique_ptr<Optimizer> optimizer)
{
  int num_streams = NamedParamGrads.size();

  std::vector<hipStream_t> streams(num_streams);

  for (int i = 0; i < num_streams; ++i)
  {

    hipStreamCreate(&streams[i]);
    //StreamAwaitStreamB(streams[i], main_stream->stream);
  }

  hipStreamSynchronize(main_stream->stream);

  int i=0;
  for (auto& pair : NamedParamGrads)
  {
    std::string param_name = pair.first;
    //std::cout << "Optimizing " << param_name << "\n";

    if (param_name!="none")
    {
      float *grad = pair.second;
      Tensor *tensor = NamedTensorsT[param_name];
      
      //std::cout << "param dims: "  << "\n";
      //PrintDims(tensor->dims);
      optimizer->init_states(param_name, tensor->dims_prod);

      if (tensor->Sparse_Idx_Tensor!=nullptr)
      {
        //std::cout << "Tensor " << param_name << " has a sparse gradient "<< "\n";
        Tensor *idx_tensor = tensor->Sparse_Idx_Tensor;

        optimizer->sparse_step(tensor->tensor_ptr, grad, idx_tensor->tensor_ptr,
                               idx_tensor->dims, tensor->dims, param_name, streams[i]);

        move_to_pool(0, idx_tensor->dims_prod, idx_tensor->tensor_ptr, "sparse grad idxs");
        delete idx_tensor;
      } else
        optimizer->step(tensor->tensor_ptr, grad, tensor->dims, param_name, streams[i]);

      int grid_size, block_size; 
      std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(tensor->dims_prod);
      grid_size = grid_block_mem_sizes[0];
      block_size = grid_block_mem_sizes[1];

      set_to_zero_kernel<<<grid_size, block_size, 0, streams[i]>>>(grad, tensor->dims_prod);
    }
    i+=1;
  }
  optimizer->count_step();

  
  for (int i = 0; i < num_streams; ++i)
  {
    hipStreamSynchronize(streams[i]);
    //StreamAwaitStreamB(main_stream->stream, streams[i]);
  }
  for (int i = 0; i < num_streams; ++i)
    hipStreamDestroy(streams[i]);

  hipStreamSynchronize(main_stream->stream);

  return std::move(optimizer);
}


std::unique_ptr<Optimizer> optimizer = nullptr;


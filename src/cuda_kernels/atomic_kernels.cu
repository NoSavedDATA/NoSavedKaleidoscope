
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <mma.h>


__device__ __forceinline__ float atomicMul(float* address, float val) {
    int *addr_as_int = (int *)address;
    int old = *addr_as_int, assumed;
    do {
        assumed = old;
        old = atomicCAS(addr_as_int, assumed,
                        __float_as_int(val * __int_as_float(assumed)));
    } while (assumed != old);
    return __int_as_float(old);
}
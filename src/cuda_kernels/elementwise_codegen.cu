#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#include "../tensor/include.h"
#include "../cuda_threads/include.h"
#include "include.h"



extern "C" void *logE(int thread_id, Tensor tensor) {
  //std::cout << "logE of: " << tensor.name << "\n";

  float * device_x = tensor.tensor_ptr;
  std::vector<float> dims = tensor.dims;
  int kDataLen = tensor.dims_prod;


  float* device_y = get_from_pool(thread_id, kDataLen, "scalar sub");


  int grid_size, block_size;
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(kDataLen);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];

  tensor.Sync();
  hipStream_t stream = ThreadsStream[thread_id];
  vec_log<<<grid_size, block_size, 0, stream>>>(device_x, device_y, kDataLen);

  Tensor *new_tensor = createTensor(device_y, dims, kDataLen, false, "");
  return new_tensor;
}

extern "C" void *logE2(int thread_id, Tensor tensor) {
  std::cout << "logE2 of: " << tensor.name << "\n";

  float * device_x = tensor.tensor_ptr;
  std::vector<float> dims = tensor.dims;
  int kDataLen = tensor.dims_prod;


  float* device_y = get_from_pool(thread_id, kDataLen, "scalar sub");


  int grid_size, block_size;
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(kDataLen);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];

  tensor.Sync();
  hipStream_t stream = ThreadsStream[thread_id];
  vec_log2<<<grid_size, block_size, 0, stream>>>(device_x, device_y, kDataLen);

  Tensor *new_tensor = createTensor(device_y, dims, kDataLen, false, "");
  return new_tensor;
}



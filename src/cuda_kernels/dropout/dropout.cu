#include <vector>
#include <hip/hip_runtime.h>

#include "../../codegen/random.h"
#include "../../cuda_threads/include.h"
#include "../../nsk_cuda/pool/include.h"
#include "../../tensor/include.h"
#include "../include.h"


// extern "C" DT_tensor *dropout(int thread_id, DT_tensor *tensor, float rate)
// {
//   if (nn_mode==training_mode&&thread_id==0)
//   {
//     float dims_prod = tensor->dims_prod;

//     int grid_size, block_size;
//     std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
//     grid_size = grid_block_mem_sizes[0];
//     block_size = grid_block_mem_sizes[1];

//     float *dropout_ptr = get_from_pool(thread_id, dims_prod, "dropout forward");
//     float *device_y = get_from_pool(thread_id, dims_prod, "dropout forward output");

//     float scale = 1 / (1-rate);
    
//     unsigned long long seed = get_int_seed();

//     dropout_mask_kernel<<<grid_size, block_size, 0, main_stream>>>(device_y, dropout_ptr, tensor->tensor_ptr, rate, scale, dims_prod, seed);
    
//     DT_tensor *dropout_tensor = createTensor(dropout_ptr, tensor->dims, dims_prod, true, "");
//     dropout_tensor->scopeless_name="";

//     DT_tensor *new_tensor = createTensor(device_y, tensor->dims, dims_prod, false, "");
//     new_tensor->AttrNodes(tensor, dropout_tensor, dropout_op);
//     return new_tensor;
//   }
//   return tensor;
// }


// void dropout_backward(float *dx, float *mask, float *dy, float dims_prod)
// {
//   int grid_size, block_size;
//   std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
//   grid_size = grid_block_mem_sizes[0];
//   block_size = grid_block_mem_sizes[1];

//   dropout_backward_kernel<<<grid_size, block_size, 0, main_stream>>>(dx, mask, dy, dims_prod);
// }
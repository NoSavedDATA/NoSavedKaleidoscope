#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <iostream>

#include "../../mangler/scope_struct.h"
#include "../../tensor/include.h"
#include "../elementwise_kernels_inline.cu"
#include "../calculate_grids.h"
#include "activation_kernels.h"




extern "C" void *relu(Scope_Struct *scope_struct, Tensor *tensor)
{
  //std::cout << "RELU THREAD IS: " << thread_id << "\n";
  int thread_id = scope_struct->thread_id;
  float *tensor_ptr = tensor->tensor_ptr;
  std::vector<float> dims = tensor->dims;
  std::vector<float> linear_layer_dims = format_LinearLayer_Dims(dims);
  float dims_prod = tensor->dims_prod;

  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  

  float *y = get_from_pool(thread_id, dims_prod, "relu");

  tensor->Sync();
  hipStream_t stream = ThreadsStream[thread_id];
  relu_forward<<<grid_size, block_size, 0, stream>>>(tensor_ptr, y, dims_prod);



  Tensor *new_tensor = createTensor(y, dims, DimsProd(dims), false, "");
  new_tensor->AttrLNode(tensor, relu_op);
  return new_tensor;
}


void relu_backward(float* inp, float dims_prod, float* dinp, float* dout) {

  
  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];

  relu_backward1<<<grid_size, block_size, 0, main_stream->stream>>>(inp, dinp, dout, dims_prod);
  
}


void gelu_backward(const float* inp, float dims_prod, float* dinp, const float* dout) {

  
  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  shared_mem_size = grid_block_mem_sizes[2];

  gelu_backward1<<<grid_size, block_size, 0, main_stream->stream>>>(dinp, inp, dout, dims_prod);
  
}

extern "C" void *gelu(Scope_Struct *scope_struct, Tensor *tensor)
{
  int thread_id = scope_struct->thread_id;
  float *tensor_ptr = tensor->tensor_ptr;
  std::vector<float> dims = tensor->dims;

  std::cout << "GELU AT THREAD " << thread_id << "\n";
  

  float dims_prod = DimsProd(dims);

  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  shared_mem_size = grid_block_mem_sizes[2];

  std::vector<float> linear_layer_dims = format_LinearLayer_Dims(dims);
  
  float *y = get_from_pool(thread_id, dims_prod,"gelu");

  tensor->Sync();
  hipStream_t stream = ThreadsStream[thread_id];
  gelu_forward_kernel1<<<grid_size, block_size, 0, stream>>>(tensor_ptr, y, dims_prod);
  

  
  int is_forward_func=1;
  

  Tensor *new_tensor = createTensor(y, dims, DimsProd(dims), false, "");
  new_tensor->AttrLNode(tensor, gelu_op);
  return new_tensor;
}

void sigmoid_backward(const float* out, float dims_prod, float* dinp, const float* dout) {
  
  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  shared_mem_size = grid_block_mem_sizes[2];

  sigmoid_backward_kernel<<<grid_size, block_size, 0, main_stream->stream>>>(dinp, out, dout, dims_prod);
  
}

extern "C" void *sigmoid(Scope_Struct *scope_struct, Tensor *tensor)
{
  int thread_id = scope_struct->thread_id;
  float *tensor_ptr = tensor->tensor_ptr;
  std::vector<float> dims = tensor->dims;
  

  float dims_prod = DimsProd(dims);

  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  shared_mem_size = grid_block_mem_sizes[2];

  std::vector<float> linear_layer_dims = format_LinearLayer_Dims(dims);
  
  float *y = get_from_pool(thread_id, dims_prod, "sigmoid");  
  
  tensor->Sync();
  hipStream_t stream = ThreadsStream[thread_id];
  sigmoid_forward_kernel<<<grid_size, block_size, 0, stream>>>(tensor_ptr, y, dims_prod);
  

  
  int is_forward_func=1;


  Tensor *new_tensor = createTensor(y, dims, DimsProd(dims), false, "");
  new_tensor->AttrLNode(tensor, sigmoid_op);
  return new_tensor;
}


void tanh_backward(const float* out, float dims_prod, float* dinp, const float* dout) {
  
  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  

  tanh_backward_kernel<<<grid_size, block_size, 0, main_stream->stream>>>(dinp, out, dout, dims_prod);
  
}

extern "C" void *_tanh(Scope_Struct *scope_struct, Tensor *tensor)
{
  int thread_id = scope_struct->thread_id;
  float *tensor_ptr = tensor->tensor_ptr;
  std::vector<float> dims = tensor->dims;
  

  float dims_prod = DimsProd(dims);

  int grid_size, block_size, shared_mem_size; 
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(dims_prod);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];
  shared_mem_size = grid_block_mem_sizes[2];

  std::vector<float> linear_layer_dims = format_LinearLayer_Dims(dims);
  
  float *y = get_from_pool(thread_id, dims_prod, "tanh");

  tensor->Sync();
  hipStream_t stream = ThreadsStream[thread_id];
  tanh_forward_kernel<<<grid_size, block_size, 0, stream>>>(tensor_ptr, y, dims_prod);
    
  int is_forward_func=1;

  //std::cout << "tanh tensor attribution from " << tensor->name<<"/"<<tensor->scopeless_name << "\n";

  Tensor *new_tensor = createTensor(y, dims, DimsProd(dims), false, "");
  new_tensor->AttrLNode(tensor, tanh_op);
  return new_tensor;
}




extern "C" void *softmax(Scope_Struct *scope_struct, Tensor *tensor)
{
  int thread_id = scope_struct->thread_id;
  float *tensor_ptr = tensor->tensor_ptr;
  std::vector<float> dims = tensor->dims;
  
  dims =  format_LinearLayer_Dims(dims);

  int B = dims[0];
  int C = dims[1];


  int grid_size, block_size, shared_mem_size;
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(B*C);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];


  tensor->Sync();
  float *probs = get_from_pool(thread_id, B*C, "softmax");
  hipStream_t stream = ThreadsStream[thread_id];
  set_to_zero_kernel<<<grid_size, block_size, 0, stream>>>(probs, B*C);


  
  /*
  grid_block_mem_sizes = CalculateGridAndBlockSizes(B*C);
  grid_size  = B;
  block_size = grid_block_mem_sizes[1];

  shared_mem_size = 2 * block_size / 32 * sizeof(float);
  softmax_forward_kernel4<<<grid_size, block_size, shared_mem_size, stream>>>(tensor_ptr, probs, B, C);
  */
 
 
  grid_block_mem_sizes = CalculateSimpleWarpGridAndBlockSizes(B);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];

  online_softmax<<<grid_size, block_size, 0, stream>>>(tensor_ptr, probs, B, C);



  Tensor *new_tensor = createTensor(probs, tensor->dims, tensor->dims_prod, false, "");
  new_tensor->op=softmax_op;
  return new_tensor;
}
#pragma once

#include <iostream>

#include "../../backprop/include.h"
#include "../../mangler/scope_struct.h"
#include "../../tensor/include.h"
#include "../activation_functions/include.h"
#include "../calculate_grids.h"
#include "../elementwise_kernels_inline.cu"
#include "classification_kernels.h"


void CrossEntropyBackward(DT_tensor *L_tensor, DT_tensor *R_tensor,
                          float *dloss,
                          float scale)
{
  
  /*
  int grid_size = B;
  int block_size = 32;
  size_t shared_mem_size = 2 * block_size / 32 * sizeof(float);
  */

  float *y_hat = L_tensor->tensor_ptr;
  float *y = R_tensor->tensor_ptr;
  std::vector<int> BC = format_LinearLayer_Dims(L_tensor->dims);
  float B  = BC[0];
  float C  = BC[1];
  

  float *probs = get_from_pool(0, B*C,"ce probs");

  //int grid_size, block_size;
  //size_t shared_mem_size;
  

  int grid_size, block_size, shared_mem_size;
  CalculateGridAndBlockSizes(B*C, grid_size, block_size);
  set_to_zero_kernel<<<grid_size, block_size, 0, main_stream>>>(probs, B*C);
  
  
  /*
  grid_block_mem_sizes = CalculateGridAndBlockSizes(B*32*C);
  grid_size  = B*32;
  block_size = grid_block_mem_sizes[1];
  
  online_softmax<<<grid_size, block_size, 0, main_stream>>>(y_hat, probs, B, C);
  */
  
  
  
  
  
  std::vector<int> grid_block_mem_sizes;
  grid_block_mem_sizes = CalculateGridAndBlockSizes(B*C);
  grid_size  = B;
  block_size = grid_block_mem_sizes[1];
  shared_mem_size = 2 * block_size / 32 * sizeof(float);

  softmax_forward_kernel4<<<grid_size, block_size, shared_mem_size, main_stream>>>(y_hat, probs, B, C);
  
  


  
  CalculateGridAndBlockSizes(B*C, grid_size, block_size);

  
  crossentropy_softmax_backward_kernel1<<<grid_size, block_size, 0, main_stream>>>(dloss, probs, y, B, C, scale);
  move_to_pool(0, B*C, probs,"ce probs");

  
}



extern "C" float cross_entropy(Scope_Struct *scope_struct, DT_tensor *y_hat, DT_tensor *y, float scale)
{
  // std::cout << "Cross entropy with scale " << scale << ".\n";

  DT_tensor *loss_tensor = new DT_tensor();
  // std::cout << "Cross entropy got last version? " << y_hat->is_last_version << "/" << y->is_last_version << ".\n";

  loss_tensor->AttrNodes(y_hat, y, cross_entropy_op);
  loss_tensor->scalar = scale;


  todo_backward_tensors.push_back(loss_tensor);

  

  return 0;
}


void CrossEntropyIdxBackward(DT_tensor *L_tensor, DT_tensor *R_tensor, 
                          float *dloss,
                          float scale)
{
  float *y_hat = L_tensor->tensor_ptr;
  float *y = R_tensor->tensor_ptr; 
  std::vector<int> BC = format_LinearLayer_Dims(L_tensor->dims);
  float B  = BC[0];
  float C  = BC[1];
  
  float *probs = get_from_pool(0, B*C,"ce probs");

  int grid_size, block_size, shared_mem_size;
  std::vector<int> grid_block_mem_sizes = CalculateGridAndBlockSizes(B*C);
  grid_size  = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];

  set_to_zero_kernel<<<grid_size, block_size, 0, main_stream>>>(probs, B*C);


  

  /*
  grid_block_mem_sizes = CalculateGridAndBlockSizes(B*C);
  grid_size  = B;
  block_size = grid_block_mem_sizes[1];
  shared_mem_size = 2 * block_size / 32 * sizeof(float);

  softmax_forward_kernel4<<<grid_size, block_size, shared_mem_size, main_stream>>>(y_hat, probs, B, C);
  */
  grid_block_mem_sizes = CalculateSimpleWarpGridAndBlockSizes(B);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];

  online_softmax<<<grid_size, block_size, 0, main_stream>>>(y_hat, probs, B, C);
  
  


  
  grid_block_mem_sizes = CalculateGridAndBlockSizes(B*C);
  grid_size = grid_block_mem_sizes[0];
  block_size = grid_block_mem_sizes[1];

  
  crossentropy_idx_backward_kernel<<<grid_size, block_size, 0, main_stream>>>(dloss, probs, y, B, C, scale);
  move_to_pool(0, B*C, probs,"ce probs");
}



extern "C" float cross_entropy_idx(Scope_Struct *scope_struct, DT_tensor *y_hat, DT_tensor *y, float scale)
{
  
  DT_tensor *loss_tensor = new DT_tensor();


  loss_tensor->AttrNodes(y_hat, y, cross_entropy_idx_op);
  loss_tensor->scalar = scale;


  todo_backward_tensors.push_back(loss_tensor);

  

  return 0;
}